#include "hip/hip_runtime.h"
// REMEMBER TO INCREASE c_code_cache_version when changing this file
//
//TODO detect SHARED_SIZE dynamically
#define SHARED_SIZE (16*1024)

enum { ConvMode_FULL, ConvMode_VALID };
PyObject * PyGpuArray_Conv(PyGpuArrayObject *img, PyGpuArrayObject * kern,
                           PyGpuArrayObject * out, const int mode,
                           const size_t subsample_rows,
                           const size_t subsample_cols,
                           const int version, const int verbose);

/*
 * version: -1, autodetect, >=0 a specific version to use.
 *          If it can't be executed, we revert to the reference implementation
 */
int
PyGpuArray_conv_valid(const PyGpuArrayObject *img,
                      const PyGpuArrayObject * kern,
                      PyGpuArrayObject * out, size_t subsample_rows,
                      size_t subsample_cols,
                      int version = -1, int verbose=0,
                      int max_threads_dim0 = 512)
{
    int work_complete = 0;
    const int shared_avail = SHARED_SIZE-150;//144 is the biggest static shared size used with compiling this file.
    if (PyGpuArray_NDIM(img) != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required img of 4D");
        return -1;
    }
    if (PyGpuArray_NDIM(kern) != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required kern of 4D");
        return -1;
    }
    if (PyGpuArray_NDIM(out) != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required out of 4D");
        return -1;
    }
    
    if (verbose>1)
    {
        fprintf(stderr,
                "INFO: Running conv_valid version=%d,"
                " MACRO kern_width=%d with inputs:\n",
                version, THEANO_KERN_WID);
        fprintf(stderr,
                "INFO:   img  dim: %llu %llu %llu %llu  "
                "img  stride: %lld %lld %lld %lld\n",
                (unsigned long long)PyGpuArray_DIMS(img)[0],
                (unsigned long long)PyGpuArray_DIMS(img)[1],
                (unsigned long long)PyGpuArray_DIMS(img)[2],
                (unsigned long long)PyGpuArray_DIMS(img)[3],
                (long long)PyGpuArray_STRIDES(img)[0]/4,
                (long long)PyGpuArray_STRIDES(img)[1]/4,
                (long long)PyGpuArray_STRIDES(img)[2]/4,
                (long long)PyGpuArray_STRIDES(img)[3]/4);
        fprintf(stderr,
                "INFO:   kern dim: %llu %llu %llu %llu  "
                "kern stride: %lld %lld %lld %lld\n",
                (unsigned long long)PyGpuArray_DIMS(kern)[0],
                (unsigned long long)PyGpuArray_DIMS(kern)[1],
                (unsigned long long)PyGpuArray_DIMS(kern)[2],
                (unsigned long long)PyGpuArray_DIMS(kern)[3],
                (long long)PyGpuArray_STRIDES(kern)[0]/4,
                (long long)PyGpuArray_STRIDES(kern)[1]/4,
                (long long)PyGpuArray_STRIDES(kern)[2]/4,
                (long long)PyGpuArray_STRIDES(kern)[3]/4);
        fprintf(stderr,
                "INFO:   out dim: %llu %llu %llu %llu  "
                "out stride: %lld %lld %lld %lld\n",
                (unsigned long long)PyGpuArray_DIMS(out)[0],
                (unsigned long long)PyGpuArray_DIMS(out)[1],
                (unsigned long long)PyGpuArray_DIMS(out)[2],
                (unsigned long long)PyGpuArray_DIMS(out)[3],
                (long long)PyGpuArray_STRIDES(out)[0]/4,
                (long long)PyGpuArray_STRIDES(out)[1]/4,
                (long long)PyGpuArray_STRIDES(out)[2]/4,
                (long long)PyGpuArray_STRIDES(out)[3]/4);
        fprintf(stderr,
                "INFO:   subsample_rows=%llu, subsample_cols=%llu\n",
                (unsigned long long)subsample_rows,
                (unsigned long long)subsample_cols);
    }

    //Check the output size is valid
    if (!(PyGpuArray_DIMS(out)[2] == ceil_intdiv(PyGpuArray_DIMS(img)[2]- PyGpuArray_DIMS(kern)[2] + 1, subsample_rows) ||
          PyGpuArray_DIMS(out)[3] == ceil_intdiv(PyGpuArray_DIMS(img)[3]- PyGpuArray_DIMS(kern)[3] + 1, subsample_cols) ||
          PyGpuArray_DIMS(out)[0] == PyGpuArray_DIMS(img)[0] ||
          PyGpuArray_DIMS(out)[1] == PyGpuArray_DIMS(kern)[0] ||
          PyGpuArray_DIMS(img)[1] == PyGpuArray_DIMS(kern)[1])) {
        PyErr_SetString(PyExc_ValueError, "GpuConv: sizes don't match");
        return -1;
    }

    // we now search through a few implementations until one applies to our arguments.

    //TODO: make separate version as if all fill this is slower.
    //TODO: Make a switch with power of 2 max size as template
    //TODO: make a parameter the number of division
    //TODO: Should we make them in separate grid block instead?
 
    const int stack_len = PyGpuArray_DIMS(img)[1];
    const int nstack=PyGpuArray_DIMS(kern)[1];
    const int nbatch=PyGpuArray_DIMS(img)[0];
    const int nkern=PyGpuArray_DIMS(kern)[0];
    const int img_wid=PyGpuArray_DIMS(img)[3];
    const int img_len=PyGpuArray_DIMS(img)[2];
    const int kern_wid=PyGpuArray_DIMS(kern)[3];
    const int kern_len=PyGpuArray_DIMS(kern)[2];
    const int out_wid=PyGpuArray_DIMS(out)[3];
    const int out_len=PyGpuArray_DIMS(out)[2];

    const int img_stride_col= PyGpuArray_STRIDES(img)[3]/4;
    const int img_stride_row=PyGpuArray_STRIDES(img)[2]/4;
    const int img_stride_stack= PyGpuArray_STRIDES(img)[1]/4;
    const int img_stride_batch=PyGpuArray_STRIDES(img)[0]/4;
    const int kern_stride_col= PyGpuArray_STRIDES(kern)[3]/4;
    const int kern_stride_row=PyGpuArray_STRIDES(kern)[2]/4;
    const int kern_stride_stack= PyGpuArray_STRIDES(kern)[1]/4;
    const int kern_stride_nkern=PyGpuArray_STRIDES(kern)[0]/4;
    const int out_stride_col = PyGpuArray_STRIDES(out)[3]/4;
    const int out_stride_row = PyGpuArray_STRIDES(out)[2]/4;
    const int out_stride_nkern = PyGpuArray_STRIDES(out)[1]/4;
    const int out_stride_batch = PyGpuArray_STRIDES(out)[0]/4;

    const int img_size=img_len*img_wid;
    const int kern_size=kern_len*kern_wid;
    const int out_size=out_len*out_wid;
    const int img_size_byte = img_size*sizeof(float);
    const int kern_size_byte = kern_size*sizeof(float);
    const int out_size_byte = out_size*sizeof(float);
    if (!((THEANO_KERN_WID == PyGpuArray_DIMS(kern)[3]) || (THEANO_KERN_WID==0))){
     PyErr_Format(PyExc_ValueError, "ERROR: This GpuConv code was compiled for"
                   " %d kernel columns, but the kernel we received had %llu columns!",
                   THEANO_KERN_WID, (unsigned long long)PyGpuArray_DIMS(kern)[3]);
      return -1;
    }

    bool subsample = subsample_rows!=1 || subsample_cols!=1;
    bool img_contiguous = img->ga.flags & GA_C_CONTIGUOUS;
    bool kern_contiguous = kern->ga.flags & GA_C_CONTIGUOUS;
    bool out_contiguous = out->ga.flags & GA_C_CONTIGUOUS;
    bool c_contiguous = img_contiguous &&  kern_contiguous && out_contiguous;

    bool img_contiguous_2d = (img_stride_col == 1) && (img_stride_row==img_wid);
    bool kern_contiguous_2d = (kern_stride_col == 1) && (kern_stride_row==kern_wid);

    //if the lower 2 dims are c_contiguous but flipped, unflipping the
    // stride and not flipping the kernel in shared memroy
    //allow to use a version that use less registers(so is faster)
    //the unflipped version of variable have the original value when
    //we don't need to unflip it, but have the new value when we unflip it.
    bool kern_flipped=true;
    bool kern_contiguous_2d_unflipped = kern_contiguous_2d;
    if(kern_stride_col==-1 && kern_stride_row==-kern_wid){
      //the last two dimensions are c_contiguous but flipped!
      kern_flipped=false;
      kern_contiguous_2d_unflipped = true;
    }

    //if we remove the restriction
    //img_size_byte+kern_size_byte>8*1024, we can enter in condition where
    //we will lower the occupency due to shared memory and/or registers.
    if ((version == -1) &&
        (out_size<64 || img_size_byte+kern_size_byte>8*1024) &&
        out_size<=256){
      //condition for exec 
      if(!subsample &&
        out_contiguous &&
        out_size<=max_threads_dim0 &&//Maximum of X threads by block
         std::max(int(img_size_byte+2*kern_wid*sizeof(float)), out_size_byte*2)<shared_avail && //their is only 16k of shared memory and if we can't have the output at least twice in shared mem, we won't have any reduce!
        !work_complete)
        version = 7; //conv_patch_stack_reduce, switch to version 8/13 automatically if needed.
    }

    if (!subsample && c_contiguous &&
        (version==0||version==2||version==-1) &&
        out_wid<=max_threads_dim0 &&//Maximum of X threads for block.x
        nstack == 1 &&// don't implement the stack in the kernel.
        img_size_byte+kern_size_byte<shared_avail && //their is only 16k of shared memory
        !work_complete) //conv_patch
    {
        int nb_split=1;//The number of split (i.e. the number of output pixel each thread compute.)
        if(version==2 && out_len>1)nb_split++;//to force the use of split=true when testing.
        //we pass by ceil_intdiv in case the out_len is not a multiple of nb_split, we want nb_split the number of iteration.
        while (ceil_intdiv(out_len,nb_split)*out_wid>max_threads_dim0)
            nb_split++;
        size_t threads_per_block[3] = {(size_t)out_wid,
                               ceil_intdiv((size_t)out_len,(size_t)nb_split),
                               (size_t)1};
        size_t n_blocks[3] = {(size_t)nbatch, (size_t)nkern, (size_t)1};

        size_t shmem_sz = (img_size + kern_size)*sizeof(float);

        GpuKernel *k = NULL;
        if(threads_per_block[1]==out_len) k=&conv_patch_2_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else k=&conv_patch_3_node_<<<<HASH_PLACEHOLDER>>>>_0;

        void *kernel_params[] = {(void *)img->ga.data, (void *)&img->ga.offset,
                                 (void *)kern->ga.data, (void *)&kern->ga.offset,
                                 (void *)out->ga.data, (void *)&out->ga.offset,
                                 (void *)&img_len, (void *)&img_wid,
                                 (void *)&kern_len, (void *)&kern_wid,
                                 (void *)&nkern, (void *)&nstack};
        int err = GpuKernel_call(k, 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

        if (err == GA_NO_ERROR)
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: used 'conv_patch' version %s nb_split=%d\n",
                      threads_per_block[1]==out_len ? "no split": "split", nb_split);
            work_complete = true;
        }
        else
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: impl 'conv_patch' failed (%s),"
                      " trying next implementation\n",
                      GpuKernel_error(k, err));
        }
    }

    if (out_contiguous &&
        (version==1||version==3||version==11||version==12||version==-1) &&
        (version!=1 || out_size<=max_threads_dim0) &&//Maximum of X threads by block.x
        out_wid<=max_threads_dim0 &&//Maximum of X threads by block.x
        img_size_byte+kern_wid*sizeof(float)<shared_avail && //their is only 16k of shared memory
        !work_complete) //conv_patch_stack
    {
      //version 1 is without split and preload the full kernel
      //version 3 is with split and preload the full kernel
      //version 11 is without split and load only 1 kernel row at a time.
      //version 12 is with split and load only 1 kernel row at a time.
        int nb_split=1;//The number of split (i.e. the number of output pixel each thread compute.)
        if((version==3||version==12) && out_len>1)nb_split++;//to force the use of split=true when testing.
        //we pass by ceil_intdiv in case the out_len is not a multiple of nb_split, we want nb_split the number of iteration.
        while (ceil_intdiv(out_len,nb_split)*out_wid>max_threads_dim0) nb_split++;
        size_t threads_per_block[3] = {(size_t)out_wid,
                               (size_t)ceil_intdiv(out_len,nb_split),
                               (size_t)1};

        bool preload_full_kernel = (img_size_byte + kern_size_byte) <shared_avail;
        if(version==11 || version==12) preload_full_kernel=false;
        size_t n_blocks[3] = {(size_t)nbatch, (size_t)nkern, (size_t)1};
        size_t shmem_sz = (img_size + (preload_full_kernel?kern_size:kern_wid))*sizeof(float);

        GpuKernel *k = NULL;
        if(!preload_full_kernel && nb_split==1 && !img_contiguous_2d && !kern_contiguous_2d && !subsample){ k=&conv_patch_stack_64_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split==1 && !img_contiguous_2d && !kern_contiguous_2d && subsample){ k=&conv_patch_stack_65_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split==1 && !img_contiguous_2d && !kern_contiguous_2d && !subsample){ k=&conv_patch_stack_66_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split==1 && !img_contiguous_2d && !kern_contiguous_2d && subsample){ k=&conv_patch_stack_67_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split!=1 && !img_contiguous_2d && !kern_contiguous_2d && !subsample){ k=&conv_patch_stack_68_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split!=1 && !img_contiguous_2d && !kern_contiguous_2d && subsample){ k=&conv_patch_stack_69_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split!=1 && !img_contiguous_2d && !kern_contiguous_2d && !subsample){ k=&conv_patch_stack_70_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split!=1 && !img_contiguous_2d && !kern_contiguous_2d && subsample){ k=&conv_patch_stack_71_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split==1 && !img_contiguous_2d && kern_contiguous_2d && !subsample){ k=&conv_patch_stack_72_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split==1 && !img_contiguous_2d && kern_contiguous_2d && subsample){ k=&conv_patch_stack_73_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split==1 && !img_contiguous_2d && kern_contiguous_2d && !subsample){ k=&conv_patch_stack_74_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split==1 && !img_contiguous_2d && kern_contiguous_2d && subsample){ k=&conv_patch_stack_75_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split!=1 && !img_contiguous_2d && kern_contiguous_2d && !subsample){ k=&conv_patch_stack_76_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split!=1 && !img_contiguous_2d && kern_contiguous_2d && subsample){ k=&conv_patch_stack_77_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split!=1 && !img_contiguous_2d && kern_contiguous_2d && !subsample){ k=&conv_patch_stack_78_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split!=1 && !img_contiguous_2d && kern_contiguous_2d && subsample){ k=&conv_patch_stack_79_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split==1 && img_contiguous_2d && !kern_contiguous_2d && !subsample){ k=&conv_patch_stack_80_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split==1 && img_contiguous_2d && !kern_contiguous_2d && subsample){ k=&conv_patch_stack_81_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split==1 && img_contiguous_2d && !kern_contiguous_2d && !subsample){ k=&conv_patch_stack_82_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split==1 && img_contiguous_2d && !kern_contiguous_2d && subsample){ k=&conv_patch_stack_83_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split!=1 && img_contiguous_2d && !kern_contiguous_2d && !subsample){ k=&conv_patch_stack_84_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split!=1 && img_contiguous_2d && !kern_contiguous_2d && subsample){ k=&conv_patch_stack_85_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split!=1 && img_contiguous_2d && !kern_contiguous_2d && !subsample){ k=&conv_patch_stack_86_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split!=1 && img_contiguous_2d && !kern_contiguous_2d && subsample){ k=&conv_patch_stack_87_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split==1 && img_contiguous_2d && kern_contiguous_2d && !subsample){ k=&conv_patch_stack_88_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split==1 && img_contiguous_2d && kern_contiguous_2d && subsample){ k=&conv_patch_stack_89_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split==1 && img_contiguous_2d && kern_contiguous_2d && !subsample){ k=&conv_patch_stack_90_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split==1 && img_contiguous_2d && kern_contiguous_2d && subsample){ k=&conv_patch_stack_91_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split!=1 && img_contiguous_2d && kern_contiguous_2d && !subsample){ k=&conv_patch_stack_92_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(!preload_full_kernel && nb_split!=1 && img_contiguous_2d && kern_contiguous_2d && subsample){ k=&conv_patch_stack_93_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split!=1 && img_contiguous_2d && kern_contiguous_2d && !subsample){ k=&conv_patch_stack_94_node_<<<<HASH_PLACEHOLDER>>>>_0;}
        else if(preload_full_kernel && nb_split!=1 && img_contiguous_2d && kern_contiguous_2d && subsample){ k=&conv_patch_stack_95_node_<<<<HASH_PLACEHOLDER>>>>_0;}

        void *kernel_params[] = {(void *)img->ga.data, (void *)&img->ga.offset,
                                 (void *)kern->ga.data, (void *)&kern->ga.offset,
                                 (void *)out->ga.data, (void *)&out->ga.offset,
                                 (void *)&img_len, (void *)&img_wid,
                                 (void *)&kern_len, (void *)&kern_wid,
                                 (void *)&out_len, (void *)&out_wid,
                                 (void *)&nkern, (void *)&nstack,
                                 (void *)&img_stride_col, (void *)&img_stride_row,
                                 (void *)&img_stride_stack, (void *)&img_stride_batch,
                                 (void *)&kern_stride_col, (void *)&kern_stride_row,
                                 (void *)&kern_stride_stack, (void *)&kern_stride_nkern,
                                 (void *)&subsample_rows, (void *)&subsample_cols};

        int err = GpuKernel_call(k, 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

        if (err == GA_NO_ERROR)
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: used 'conv_patch_stack' version with nb_split=%i"
                      " and preload_full_kernel=%i,"
                      " subsample_rows=%llu, subsample_cols=%llu\n",
                      nb_split, preload_full_kernel,
                      (unsigned long long)subsample_rows,
                      (unsigned long long)subsample_cols);
            work_complete = true;
        }
        else
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: impl 'conv_patch_stack' failed (%s),"
                      " trying next implementation\n",
                      GpuKernel_error(k, err));
        }
    }

    if (!subsample && out_contiguous &&
        (version==4||version==-1) &&
        out_wid<=max_threads_dim0 &&//Maximum of X threads by block.x
        nstack == 1 &&// don't implement the stack in the kernel.
        kern_len*img_wid*sizeof(float)+kern_size_byte<shared_avail &&//their is only 16k of shared memory
        !work_complete) //conv_rows

    {
        size_t threads_per_block[3] = {(size_t)out_wid, (size_t)1, (size_t)1};
        size_t n_blocks[3] = {(size_t)out_len, (size_t)nbatch*nkern, (size_t)1};
        size_t shmem_sz = (kern_len*img_wid + kern_size)*sizeof(float);

        GpuKernel *k = NULL;
        if(!img_contiguous_2d || !kern_contiguous_2d) k=&conv_rows_0_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else k=&conv_rows_1_node_<<<<HASH_PLACEHOLDER>>>>_0;

        void *kernel_params[] = {
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&nkern, (void *)&nstack,
            (void *)&img_stride_col, (void *)&img_stride_row,
            (void *)&img_stride_stack, (void *)&img_stride_batch,
            (void *)&kern_stride_col, (void *)&kern_stride_row,
            (void *)&kern_stride_stack, (void *)&kern_stride_nkern};
        int err = GpuKernel_call(k, 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

        if (err == GA_NO_ERROR)
        {
            work_complete = true;
            if (verbose)
              fprintf(stderr, "INFO: used 'conv_rows' version\n");
        }
        else
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: impl 'conv_rows' failed (%s),"
                      " trying next implementation\n",
                      GpuKernel_error(k, err));
        }
    }
    if (!subsample && out_contiguous &&
        (version==5||version==-1) &&
        out_wid<=max_threads_dim0 &&//Maximum of X threads by block.x
        img_wid*kern_len*sizeof(float)+kern_size_byte<shared_avail && //their is only 16k of shared memory
        !work_complete) //conv_rows_stack

    {
        int nb_row=1;
        //TODO:if not c_contiguous, lower max_thread as we use 22
        //registers by thread and we won't execute 2 block in one MP.
        for(int i=2;i<=out_len;i++){
          if((i)*out_wid<=max_threads_dim0 && ((kern_len+i)*img_wid + kern_size)*sizeof(float)<shared_avail)
            nb_row=i;
        }

        size_t threads_per_block[3] = {(size_t)out_wid, (size_t)nb_row, (size_t)1};
        size_t n_blocks[3] = {(size_t)ceil_intdiv(out_len,nb_row),
                              (size_t)nbatch*nkern, (size_t)1};

        size_t shmem_sz =((kern_len+nb_row-1)*img_wid + kern_size)*sizeof(float);

        GpuKernel *k = NULL;
        if(!img_contiguous_2d || !kern_contiguous_2d) {
            k=&conv_rows_stack_0_node_<<<<HASH_PLACEHOLDER>>>>_0;
        } else {
            k=&conv_rows_stack_1_node_<<<<HASH_PLACEHOLDER>>>>_0;
        }

        void *kernel_params[] = {
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&nkern, (void *)&nstack,
            (void *)&img_stride_col, (void *)&img_stride_row,
            (void *)&img_stride_stack, (void *)&img_stride_batch,
            (void *)&kern_stride_col, (void *)&kern_stride_row,
            (void *)&kern_stride_stack, (void *)&kern_stride_nkern};
        int err = GpuKernel_call(k, 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

        if (err == GA_NO_ERROR)
        {
            work_complete = true;
            if (verbose)
              fprintf(stderr, "INFO: used 'conv_rows_stack' version\n");
        }
        else
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: impl 'conv_rows_stack' failed (%s),"
                      " trying next implementation\n",
                      GpuKernel_error(k, err));
        }
    }

    if (!subsample && out_contiguous &&
        (version==9||version==10||version==-1) &&
        out_wid<=max_threads_dim0 &&//Maximum of X threads by block.x
        (img_wid+kern_wid)*sizeof(float)<shared_avail && //their is only 16k of shared memory
        (version != 9 || (img_wid+kern_len*kern_wid)*sizeof(float)<shared_avail) && //version 9 use more memory
        !work_complete) //conv_rows_stack2

    {
      // version 9:we preload the full kernel
      // version 10: load only a few row at a time.
        int nb_row=1;
        int version_back = version;
        //TODO:if not c_contiguous, lower max_thread as we use 22 registers by thread and we won't execute 2 block in one MP.
        if(version==-1 && (img_wid+kern_len*kern_wid)*sizeof(float)<shared_avail)
          version = 9;
        else if(version==-1)version = 10;

        int k_size = kern_size;
        if(version==10)
          k_size=kern_wid;

        for(int i=2;i<=out_len;i++){
          if(i*out_wid<=max_threads_dim0 && (i*img_wid + k_size)*sizeof(float)<shared_avail)
            nb_row=i;
        }

        //to test the case when we don't have a thread by output pixel.
        if((version_back!=-1)&& nb_row>1) nb_row--;


        size_t threads_per_block[3] = {(size_t)out_wid, (size_t)nb_row, (size_t)1};
        size_t n_blocks[3] = {(size_t)ceil_intdiv(out_len,nb_row),
                              (size_t)nbatch*nkern, (size_t)1};
          
        size_t shmem_sz =((kern_len+nb_row-1)*img_wid + kern_size)*sizeof(float);

        GpuKernel *k = NULL;
        if((!img_contiguous_2d || !kern_contiguous_2d)&&version==9) k=&conv_rows_stack2_1_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(version==9) k=&conv_rows_stack2_3_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(!img_contiguous_2d || !kern_contiguous_2d) k=&conv_rows_stack2_0_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else k=&conv_rows_stack2_2_node_<<<<HASH_PLACEHOLDER>>>>_0;

        void *kernel_params[] = {
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&nkern, (void *)&nstack,
            (void *)&img_stride_col, (void *)&img_stride_row,
            (void *)&img_stride_stack, (void *)&img_stride_batch,
            (void *)&kern_stride_col, (void *)&kern_stride_row,
            (void *)&kern_stride_stack, (void *)&kern_stride_nkern};
        int err = GpuKernel_call(k, 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

        if (err == GA_NO_ERROR)
        {
            work_complete = true;
            if (verbose)
              fprintf(stderr,
                      "INFO: used 'conv_rows_stack2' version %s with"
                      " %d row(s).\n",
                      (version==9?"'load full kernel'":
                       "'load 1 kern row at a time'"),nb_row);
        }
        else
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: impl 'conv_rows_stack2' failed (%s),"
                      " trying next implementation\n",
                      GpuKernel_error(k, err));
        }
    }

    //version 8 is the same but we force the split.
    // The split is need in case we have too much threads.
    // This happen frequently if the kernel length is big.
    // Big kernel is frequent in the gradient.
    //version 8 need a minimum of kernel length as we force the split.
    //version 8 is needed to test more easily this kernel template parameter.
    //version 13 load only 1 kernel row at a time.
    if (!subsample &&
        out_contiguous &&
        out_size<=max_threads_dim0 &&//Maximum of X threads by block
        (version==7||version==8||version==13||version==-1) &&
        (version!=8||kern_len>1) && //version 8 need a minimal kernel length as big as the split.
        //version 13 need a minimal kernel length as big as the split.
        (version!=13||kern_len>1) &&
        !work_complete) //conv_patch_stack_reduce
    {
        int nb_split=1;
        int full_kern=true;

        if(version==8||version==13) nb_split++;//force the split.
        if(version==13)full_kern=false;

        //check if we can fit the full kernel in the shared memory
        if(sizeof(float)*std::max(img_size + kern_size, out_size*2) > shared_avail){
          full_kern = false;
        }

        //thread_z is going to be ceil_intdiv(kern_len, nb_split)
        // we need enough splits so that
        // a) thread_z fits in the 'z' threadIdx (i.e. is less than 64)
        // b) thread_z * out_len * out_wid fits in the thread count
        // c) the kernel doesn't need too much shared memory

        // constraint (a)
        // device 1.3 have a max of 64 thread in z
        while(ceil_intdiv(kern_len,nb_split)>64) nb_split++;

        // constraint (b)
        //  (TODO: read the number of threads per block from the device)
        while(out_size*ceil_intdiv(kern_len,nb_split)>max_threads_dim0)
            nb_split++;

        // tentative estimates (prior to contraint c)
        size_t thread_z=ceil_intdiv(kern_len,nb_split);
        size_t shmem_sz = sizeof(float)*(full_kern
                ? std::max((size_t)img_size + kern_size, out_size*thread_z)
                : std::max((size_t)img_size + thread_z*kern_wid, out_size*thread_z));

        // constraint (c)
        while ((shmem_sz >= shared_avail) && (nb_split <= kern_len)){
            //if we can't fit the kernel in shared memory, we must split it more.
            nb_split++;
            thread_z=ceil_intdiv(kern_len,nb_split);
            shmem_sz = sizeof(float)*(full_kern
                ? std::max((size_t)img_size + kern_size, out_size*thread_z)
                : std::max(img_size + thread_z*kern_wid, out_size*thread_z));
        }
        if (nb_split <= kern_len)
        {
            assert(thread_z>0);//should not happen, but in case...
            if(!full_kern) assert(thread_z!=kern_len);

            size_t threads_per_block[3] = {(size_t)out_wid,
                                   (size_t)out_len,
                                   (size_t)thread_z};
            size_t n_blocks[3] = {(size_t)nbatch, (size_t)nkern, (size_t)1};

            GpuKernel *k = NULL;

            const bool split=thread_z!=kern_len;
            const bool ccontig=img_contiguous_2d && kern_contiguous_2d_unflipped;

            //printf("kern_flipped=%d, ccontig=%d, split=%d, full_kern=%d\n",kern_flipped,ccontig,split,full_kern);
            //We will always be split when we don't load the full kernel

            /* if(!kern_flipped && !ccontig && !split && !full_kern) k=&conv_patch_stack_reduce_0_node_<<<<HASH_PLACEHOLDER>>>>_0;*/
            /*else*/ if(!kern_flipped && !ccontig && !split && full_kern) k=&conv_patch_stack_reduce_1_node_<<<<HASH_PLACEHOLDER>>>>_0;
            else if(!kern_flipped && !ccontig  && split && !full_kern) k=&conv_patch_stack_reduce_2_node_<<<<HASH_PLACEHOLDER>>>>_0;
            else if(!kern_flipped && !ccontig  && split && full_kern) k=&conv_patch_stack_reduce_3_node_<<<<HASH_PLACEHOLDER>>>>_0;
            /*else if(!kern_flipped && ccontig  && !split && !full_kern) k=&conv_patch_stack_reduce_4_node_<<<<HASH_PLACEHOLDER>>>>_0;*/
            else if(!kern_flipped && ccontig  && !split && full_kern) k=&conv_patch_stack_reduce_5_node_<<<<HASH_PLACEHOLDER>>>>_0;
            else if(!kern_flipped && ccontig  && split && !full_kern) k=&conv_patch_stack_reduce_6_node_<<<<HASH_PLACEHOLDER>>>>_0;
            else if(!kern_flipped && ccontig  && split && full_kern) k=&conv_patch_stack_reduce_7_node_<<<<HASH_PLACEHOLDER>>>>_0;
            /*else if(kern_flipped  && !ccontig && !split && !full_kern) k=&conv_patch_stack_reduce_8_node_<<<<HASH_PLACEHOLDER>>>>_0;*/
            else if(kern_flipped  && !ccontig && !split && full_kern) k=&conv_patch_stack_reduce_9_node_<<<<HASH_PLACEHOLDER>>>>_0;
            else if(kern_flipped  && !ccontig && split && !full_kern) k=&conv_patch_stack_reduce_10_node_<<<<HASH_PLACEHOLDER>>>>_0;
            else if(kern_flipped  && !ccontig && split && full_kern) k=&conv_patch_stack_reduce_11_node_<<<<HASH_PLACEHOLDER>>>>_0;
            /*else if(kern_flipped  && ccontig  && !split && !full_kern) k=&conv_patch_stack_reduce_12_node_<<<<HASH_PLACEHOLDER>>>>_0;*/
            else if(kern_flipped  && ccontig  && !split && full_kern) k=&conv_patch_stack_reduce_13_node_<<<<HASH_PLACEHOLDER>>>>_0;
            else if(kern_flipped  && ccontig  && split && !full_kern) k=&conv_patch_stack_reduce_14_node_<<<<HASH_PLACEHOLDER>>>>_0;
            else if(kern_flipped  && ccontig  && split && full_kern) k=&conv_patch_stack_reduce_15_node_<<<<HASH_PLACEHOLDER>>>>_0;

            void *kernel_params[] = {
                (void *)img->ga.data, (void *)&img->ga.offset,
                (void *)kern->ga.data, (void *)&kern->ga.offset,
                (void *)out->ga.data, (void *)&out->ga.offset,
                (void *)&img_len, (void *)&img_wid,
                (void *)&kern_len, (void *)&kern_wid,
                (void *)&nkern, (void *)&nstack,
                (void *)&img_stride_col, (void *)&img_stride_row,
                (void *)&img_stride_stack, (void *)&img_stride_batch,
                (void *)&kern_stride_col,
                (void *)&kern_stride_row,
                (void *)&kern_stride_stack, (void *)&kern_stride_nkern};
            int err = GpuKernel_call(k, 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

            if (err == GA_NO_ERROR)
            {
                if (verbose)
                    fprintf(stderr,
                            "INFO: used 'conv_patch_stack_reduce' version"
                            " kern_flipped=%i ccontig=%i nb_split=%d,"
                            " preload_full_kern=%d\n",
                            kern_flipped, ccontig, nb_split, full_kern);
                work_complete = true;
            }
            else
            {
                if (verbose)
                  fprintf(stderr,
                          "INFO: impl 'conv_patch_stack_reduce' failed (%s),"
                          " trying next implementation\n",
                          GpuKernel_error(k, err));
            }
        } // else no good nb_splits was found
    }

    if ((version==6||version==-1) &&
        kern_len<=320 &&
        !work_complete) //conv_valid_row_reduce
    {
        size_t outsize = PyGpuArray_SIZE(out);
        size_t n_blocks[3] = {std::min(outsize, (size_t)4096),
                              (size_t)1, (size_t)1};

        int block_nstack=nstack;
        //Max of 512 threads per blocks.
        //On old hardware, we have a max of 356 threads as we have only 
        //8k registers and the kernel use 23 register
        //TODO: check if we have 8k or 16k of register...
        while(block_nstack*kern_len>320)block_nstack--;
        size_t threads_per_block[3] = {(size_t)block_nstack, (size_t)kern_len, (size_t)1};

        size_t n_reduce_buf = block_nstack * kern_len * sizeof(float);
        /* initial_reduce_boundary is the greatest power of two less than n_reduce_buf/ sizeof(float)
         *
         * if n_reduce_buf == sizeof(float), then initial_reduce_boundary == 0.
         * */
        int initial_reduce_boundary = (1 << (int)(log2((double)(n_reduce_buf/sizeof(float)))));
        if (initial_reduce_boundary == (n_reduce_buf / sizeof(float)))
            initial_reduce_boundary >>= 1;

        if (n_reduce_buf == sizeof(float))
            assert (initial_reduce_boundary == 0);
        else
        {
            assert (initial_reduce_boundary * 2 >= n_reduce_buf/sizeof(float));
            assert (initial_reduce_boundary < n_reduce_buf/sizeof(float));
        }

        GpuKernel *k = NULL;
        //std::cerr << "initial_reduce_boundary " << initial_reduce_boundary << "\n";
        //std::cerr << "kerns " << nstack << " " << kern_len << "\n";
        //std::cerr << "n_reduce_buf/sizeof(float) " << n_reduce_buf / sizeof(float) << "\n";
        if(block_nstack==nstack)
          k=&conv_valid_row_reduce_0_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else
          k=&conv_valid_row_reduce_1_node_<<<<HASH_PLACEHOLDER>>>>_0;

        void *kernel_params[] = {
            (void *)&nbatch, (void *)&nkern, (void *)&stack_len,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&out_len, (void *)&out_wid,
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)&img_stride_batch, (void *)&img_stride_stack,
            (void *)&img_stride_row, (void *)&img_stride_col,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)&kern_stride_nkern, (void *)&kern_stride_stack,
            (void *)&kern_stride_row, (void *)&kern_stride_col,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&out_stride_batch, (void *)&out_stride_nkern,
            (void *)&out_stride_row, (void *)&out_stride_col,
            (void *)&subsample_rows, (void *)&subsample_cols,
            (void *)&initial_reduce_boundary};
        int err = GpuKernel_call(k, 3, threads_per_block, n_blocks, n_reduce_buf, kernel_params);

        if (err == GA_NO_ERROR)
        {
            work_complete = true;
            if (verbose)
              fprintf(stderr, "INFO: used 'conv_valid_row_reduce' version\n");
        }
        else
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: impl 'conv_valid_row_reduce' failed (%s),"
                      " trying next implementation\n",
                      GpuKernel_error(k, err));
        }
    }

    if (1 && !work_complete) //conv_reference_valid
    {
        size_t outsize = PyGpuArray_SIZE(out);
        size_t n_blocks[3] = {std::min(outsize, (size_t)4096),
                              (size_t)1, (size_t)1};
        size_t threads_per_block[3] = {std::min(ceil_intdiv(outsize, n_blocks[0]),
                                        (size_t)256),
                               (size_t)1, (size_t)1};

        if (verbose)
            fprintf(stderr, "INFO: launching conv_reference_valid\n");

        void *kernel_params[] = {
            (void *)&nbatch, (void *)&nkern, (void *)&stack_len,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&out_len, (void *)&out_wid,
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)&img_stride_batch, (void *)&img_stride_stack,
            (void *)&img_stride_row, (void *)&img_stride_col,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)&kern_stride_nkern, (void *)&kern_stride_stack,
            (void *)&kern_stride_row, (void *)&kern_stride_col,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&out_stride_batch, (void *)&out_stride_nkern,
            (void *)&out_stride_row, (void *)&out_stride_col,
            (void *)&subsample_rows, (void *)&subsample_cols};
        int err = GpuKernel_call(&conv_reference_valid_node_<<<<HASH_PLACEHOLDER>>>>_0,
                                 3, threads_per_block, n_blocks, 0, kernel_params);

        if (err == GA_NO_ERROR)
        {
            work_complete = true;
            if (verbose)
              fprintf(stderr, "INFO: used 'conv_reference_valid' version\n");
        }
        else
        {
            if (verbose)
              fprintf(stderr, "INFO: 'conv_reference_valid' failed\n");
            PyErr_Format(PyExc_RuntimeError,
                         "ERROR: all implementations failed for"
                         " PyGpuArray_conv_valid! (%s)",
                         GpuKernel_error(&conv_reference_valid_node_<<<<HASH_PLACEHOLDER>>>>_0, err));
            return -1;
        }
    }
    if (!work_complete)
    {
      PyErr_Format(PyExc_RuntimeError,
                   "ERROR: no implementation(s) worked for"
                   " PyGpuArray_conv_valid!"
                   " Version asked(%d) (-1 mean use an heuristic)",
                   version);
        return -1;
    }
    return 0;
}

int
PyGpuArray_conv_full(const PyGpuArrayObject *img, const PyGpuArrayObject * kern,
                      PyGpuArrayObject * out, size_t subsample_rows,
                      size_t subsample_cols, int version = -1, int verbose=0,
                      int max_threads_dim0=512)
{
  //144 is the biggest static shared size used with compiling this file.
    const int shared_avail = SHARED_SIZE - 150;

    int work_complete = 0;
    if (PyGpuArray_NDIM(img) != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required img of 4D");
        return -1;
    }
    if (PyGpuArray_NDIM(kern) != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required kern of 4D");
        return -1;
    }
    if (PyGpuArray_NDIM(out) != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required out of 4D");
        return -1;
    }
    // check the size of the output matrix
    assert (PyGpuArray_DIMS(out)[2] == ceil_intdiv(PyGpuArray_DIMS(img)[2] + PyGpuArray_DIMS(kern)[2] - 1, subsample_rows));
    assert (PyGpuArray_DIMS(out)[3] == ceil_intdiv(PyGpuArray_DIMS(img)[3] + PyGpuArray_DIMS(kern)[3] - 1, subsample_cols));

    assert (PyGpuArray_DIMS(out)[0] == PyGpuArray_DIMS(img)[0]);
    assert (PyGpuArray_DIMS(out)[1] == PyGpuArray_DIMS(kern)[0]);
    assert (PyGpuArray_DIMS(img)[1] == PyGpuArray_DIMS(kern)[1]);

    const int stack_len=PyGpuArray_DIMS(img)[1];
    const int nstack=PyGpuArray_DIMS(kern)[1];
    const int nbatch=PyGpuArray_DIMS(img)[0];
    const int nkern=PyGpuArray_DIMS(kern)[0];
    const int img_wid=PyGpuArray_DIMS(img)[3];
    const int img_len=PyGpuArray_DIMS(img)[2];
    const int kern_wid=PyGpuArray_DIMS(kern)[3];
    const int kern_len=PyGpuArray_DIMS(kern)[2];
    const int out_wid=PyGpuArray_DIMS(out)[3];
    const int out_len=PyGpuArray_DIMS(out)[2];

    const int img_stride_col= PyGpuArray_STRIDES(img)[3]/4;
    const int img_stride_row=PyGpuArray_STRIDES(img)[2]/4;
    const int img_stride_stack=PyGpuArray_STRIDES(img)[1]/4;
    const int img_stride_batch=PyGpuArray_STRIDES(img)[0]/4;
    const int kern_stride_col= PyGpuArray_STRIDES(kern)[3]/4;
    const int kern_stride_row=PyGpuArray_STRIDES(kern)[2]/4;
    const int kern_stride_stack= PyGpuArray_STRIDES(kern)[1]/4;
    const int kern_stride_nkern=PyGpuArray_STRIDES(kern)[0]/4;
    const int out_stride_col = PyGpuArray_STRIDES(out)[3]/4;
    const int out_stride_row = PyGpuArray_STRIDES(out)[2]/4;
    const int out_stride_nkern = PyGpuArray_STRIDES(out)[1]/4;
    const int out_stride_batch = PyGpuArray_STRIDES(out)[0]/4;

    const int img_size=img_len*img_wid;
    const int kern_size=kern_len*kern_wid;
    const int out_size=out_len*out_wid;
    const int img_size_byte = img_size*sizeof(float);
    const int kern_size_byte = kern_size*sizeof(float);
    //padded image sizes
    const int img_wid_padded=img_wid+2*kern_wid-2;
    const int img_len_padded=img_len+2*kern_len-2;
    const int img_size_padded=img_len_padded * img_wid_padded;
    const int img_size_padded_byte = img_size_padded*sizeof(float);
    
    //const int out_size_byte = out_size*sizeof(float); // unused 

    if (!((THEANO_KERN_WID == PyGpuArray_DIMS(kern)[3]) ||
          (THEANO_KERN_WID == 0))){
      PyErr_Format(PyExc_ValueError,
                   "ERROR: This GpuConv code was compiled for"
                   " %d kernel columns, but the kernel we received"
                   " had %llud columns!",
                   THEANO_KERN_WID, (unsigned long long)PyGpuArray_DIMS(kern)[3]);
      return -1;
    }
    bool subsample = subsample_rows!=1 || subsample_cols!=1;

    bool img_contiguous = img->ga.flags & GA_C_CONTIGUOUS;
    bool kern_contiguous = kern->ga.flags & GA_C_CONTIGUOUS;
    bool out_contiguous = out->ga.flags & GA_C_CONTIGUOUS;
    bool c_contiguous = img_contiguous &&  kern_contiguous && out_contiguous;

    bool img_contiguous_2d = (img_stride_col == 1) && (img_stride_row==img_wid);
    bool kern_contiguous_2d = (kern_stride_col == 1) && (kern_stride_row==kern_wid);

    bool img_batch_stack_contiguous = (img_stride_stack==img_stride_row*img_len) && (img_stride_batch==img_stride_stack*nstack);//don't support stride for nbatch and nstack

    //if the lower 2 dims are c_contiguous but flipped, unflipping the
    //stride and not flipping the kernel in shared memroy
    //allow to use a version that use less registers(so is faster)
    //the unflipped version of variable have the original value when
    //we don't need to unflip it, but have the new value when we unflip it.
    bool kern_flipped=true;
    bool kern_contiguous_2d_unflipped = kern_contiguous_2d;
    if(kern_stride_col==-1 && kern_stride_row==-kern_wid){
      //the last two dimensions are c_contiguous but flipped!
      kern_flipped=false;
      kern_contiguous_2d_unflipped = true;
    }

    if (verbose>1)
    {
        printf("INFO: Running conv_full version=%d,"
               " MACRO kern_width=%d with inputs:\n", version, THEANO_KERN_WID);
        printf("INFO:   img  dim: %llu %llu %llu %llu  "
               "img  stride: %lld %lld %lld %lld\n",
               (unsigned long long)nbatch,
               (unsigned long long)stack_len,
               (unsigned long long)img_len,
               (unsigned long long)img_wid,
               (long long)img_stride_batch,
               (long long)img_stride_stack,
               (long long)img_stride_row,
               (long long)img_stride_col);
        printf("INFO:   kern dim: %llu %llu %llu %llu  "
               "kern stride: %lld %lld %lld %lld\n",
               (unsigned long long)nkern,
               (unsigned long long)nstack,
               (unsigned long long)kern_len,
               (unsigned long long)kern_wid,
               (long long)kern_stride_nkern,
               (long long)kern_stride_stack,
               (long long)kern_stride_row,
               (long long)kern_stride_col);
        printf("INFO:   out dim: %llu %llu %llu %llu  "
               "out stride: %lld %lld %lld %lld\n",
               (unsigned long long)PyGpuArray_DIMS(out)[0],
               (unsigned long long)PyGpuArray_DIMS(out)[1],
               (unsigned long long)out_len,
               (unsigned long long)out_wid,
               (long long)out_stride_batch,
               (long long)out_stride_nkern,
               (long long)out_stride_row,
               (long long)out_stride_col);
    }

    if (!subsample &&
        out_contiguous &&
        (version==3||version==4||version==5||version==-1) &&
        out_wid<=max_threads_dim0 &&//Maximum of X threads by block.x
        (kern_len+2*kern_len-2)*img_wid_padded*sizeof(float) + kern_size_byte<shared_avail && //their is only 16k of shared memory
        !work_complete) //conv_full_patch_stack_padded
    {
      //version 3 without split
      //version 4 with split (more registers)
      //version 5 with split (more registers) low mem version(some restriction and still more register)
        int nb_split=1;//The number of split (i.e. the number of output pixel each thread compute.)
        if((version==4 || version==5) && out_len>1) nb_split++;//to force the use of split=true when testing.
        if(kern_len==1 && version==5){
          //version 5 don't support kern_len==1 as 1%0 return -1.
          version=-1;
          if(verbose)fprintf(stderr, "WARNING:conv full: Asking version 5 with kern_len==1. Combination not supported!\n");
        }
        if(img_size_padded_byte+kern_size_byte>shared_avail) version=5;

        //we pass by ceil_intdiv in case the out_len is not a multiple
        //of nb_split, we want nb_split the number of iteration.
        //Max of 16k of shared memory
        if(version==5)
          while ((((kern_len+ceil_intdiv(out_len,nb_split)-1)+2*kern_len-2)*img_wid_padded*sizeof(float) + kern_size_byte)>shared_avail) nb_split++;
        
        //327 as we use 25 register
        //version 5 will have only 1 block running at a time, so we
        //can use 32 registers per threads, but their is some other stuff that
        //for the limit to bu lower then 512.
        int max_thread = (version!=5?327:450);
        while (ceil_intdiv(out_len,nb_split)*out_wid>max_thread) nb_split++;
        if(version==-1 && out_size>max_threads_dim0)version=4;
        if(version==-1)version=3;


        if(version==-1 && nb_split>1) version=4;
        else if(version==-1) version=3;
        //force version 4 when more than 1 split are needed to always execute.
        else if(version==3 && nb_split!=1) version=4;

        assert(version!=3 || nb_split==1);
        assert(version!=5 || kern_len>1);
        assert(version!=-1);

        size_t threads_per_block[3] = {(size_t)out_wid,
                               ceil_intdiv((size_t)out_len,(size_t)nb_split),
                               (size_t)1};
        size_t n_blocks[3] = {(size_t)nbatch, (size_t)nkern, (size_t)1};

        size_t shmem_sz=img_size_padded_byte + kern_size_byte;
        if(version==5)
          shmem_sz=((kern_len+threads_per_block[1]-1)+2*kern_len-2)*img_wid_padded*sizeof(float) + kern_size_byte;

        GpuKernel *k = NULL;
        if(version==3) k=&conv_full_patch_stack_padded_0_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(version==5) k=&conv_full_patch_stack_padded_1_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(version==4) k=&conv_full_patch_stack_padded_2_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==3) k=&conv_full_patch_stack_padded_4_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==5) k=&conv_full_patch_stack_padded_5_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==4) k=&conv_full_patch_stack_padded_6_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(version==3 && kern_flipped) k=&conv_full_patch_stack_padded_8_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(version==5 && kern_flipped)k=&conv_full_patch_stack_padded_9_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(version==4 && kern_flipped)k=&conv_full_patch_stack_padded_10_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==3 && kern_flipped) k=&conv_full_patch_stack_padded_12_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==5 && kern_flipped) k=&conv_full_patch_stack_padded_13_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==4 && kern_flipped) k=&conv_full_patch_stack_padded_14_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else assert(false);

        void *kernel_params[] = {
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&nkern, (void *)&nstack,
            (void *)&img_stride_col, (void *)&img_stride_row,
            (void *)&img_stride_stack, (void *)&img_stride_batch,
            (void *)&kern_stride_col, (void *)&kern_stride_row,
            (void *)&kern_stride_stack, (void *)&kern_stride_nkern};
        int err = GpuKernel_call(k, 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

        if (err == GA_NO_ERROR)
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: used 'conv_full_patch_stack_padded'"
                      " nb_split=%d low_mem=%s\n",
                      nb_split, (version==5?"true":"false"));
            work_complete = true;
        }
        else
        {
          if (verbose)
            fprintf(stderr,
                    "INFO: impl 'conv_full_patch_stack_padded' %s %s"
                    " failed (%s), trying next implementation\n",
                    version==3?"no split": "split",
                    (version==5?"low_mem":"not_low_mem"),
                    GpuKernel_error(k, err));
        }                         
    }

    if (!subsample && c_contiguous &&
        (version==0||version==-1) &&
        out_size<=max_threads_dim0 &&//Maximum of X threads by block
        nstack == 1 &&// don't implement the stack in the kernel.
        img_size_byte+kern_size_byte<shared_avail && //their is only 16k of shared memory
        !work_complete) //conv_full_patch
    {
        size_t threads_per_block[3] = {(size_t)out_wid, (size_t)out_len, (size_t)1};
        size_t n_blocks[3] = {(size_t)nbatch, (size_t)nkern, (size_t)1};
        size_t shmem_sz = (img_size + kern_size)*sizeof(float);
        //TODO assert c_continious for img, kern and out in the 2 inner dimensions.

        void *kernel_params[] = {
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&nkern, (void *)&nstack};
        int err = GpuKernel_call(&conv_full_patch_node_<<<<HASH_PLACEHOLDER>>>>_0,
                                 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

        if (err == GA_NO_ERROR)
        {
            if (verbose) fprintf(stderr, "INFO: used 'conv_full_patch' version\n");
            work_complete = true;
        }
        else
        {
            if (verbose)
              fprintf(stderr,
                      "INFO: impl 'conv_full_patch' failed (%s),"
                      " trying next implementation\n",
                      GpuKernel_error(&conv_full_patch_node_<<<<HASH_PLACEHOLDER>>>>_0, err));
        }                         
    }
    if (false && !subsample && //disabled as test fail for this kernel
        (version==1||version==-1) &&
        out_size<=max_threads_dim0 &&//Maximum of X threads by block
        (nbatch > 20 || version==1) &&  // we only launch nbatch blocks, so make sure there is enough to be worth it, but if we specify the version, this check should not be done to allow testing.
        nstack*img_size_byte+nstack*kern_size_byte<shared_avail && //there is only 16k of shared memory
        !work_complete) //conv_full_load_everything
    {
        size_t threads_per_block[3] = {(size_t)out_wid, (size_t)out_len, (size_t)1};
        size_t n_blocks[3] = {(size_t)nbatch, (size_t)1, (size_t)1};
        size_t shmem_sz = (img_size + kern_size)*nstack*sizeof(float);
        //TODO assert c_continious for img, kern and out in the 2 inner dimensions.

        void *kernel_params[] = {
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&nkern, (void *)&nstack,
            (void *)&img_stride_col, (void *)&img_stride_row,
            (void *)&img_stride_stack, (void *)&img_stride_batch,
            (void *)&kern_stride_col, (void *)&kern_stride_row,
            (void *)&kern_stride_stack, (void *)&kern_stride_nkern};
        int err = GpuKernel_call(&conv_full_load_everything_node_<<<<HASH_PLACEHOLDER>>>>_0,
                                 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

        if (err == GA_NO_ERROR)
        {
            if (verbose) fprintf(stderr, "INFO: used 'conv_full_load_everything' version\n");
            work_complete = true;
        }
        else
        {
            if (verbose)
              fprintf(stderr, "INFO: impl 'conv_full_load_everything'"
                      " failed (%s), trying next implementation\n",
                      GpuKernel_error(&conv_full_load_everything_node_<<<<HASH_PLACEHOLDER>>>>_0, err));
        }
    }

    if (!subsample &&
        img_batch_stack_contiguous &&
        out_contiguous &&
        (version==2||version==-1) &&
        out_size<=max_threads_dim0 &&//Maximum of X threads by block
        img_size_byte+kern_size_byte<shared_avail && //their is only 16k of shared memory
        !work_complete) //conv_full_patch_stack
    {
        size_t threads_per_block[3] = {(size_t)out_wid, (size_t)out_len, (size_t)1};
        size_t n_blocks[3] = {(size_t)nbatch, (size_t)nkern, (size_t)1};
        size_t shmem_sz = (img_size + kern_size)*sizeof(float);

        GpuKernel *k = NULL;
        if(!img_contiguous_2d && !kern_contiguous_2d) k=&conv_full_patch_stack_0_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(!img_contiguous_2d && kern_contiguous_2d) k=&conv_full_patch_stack_1_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(img_contiguous_2d && !kern_contiguous_2d) k=&conv_full_patch_stack_2_node_<<<<HASH_PLACEHOLDER>>>>_0;
        else if(img_contiguous_2d && kern_contiguous_2d) k=&conv_full_patch_stack_3_node_<<<<HASH_PLACEHOLDER>>>>_0;

        void *kernel_params[] = {
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&nkern, (void *)&nstack,
            (void *)&img_stride_col, (void *)&img_stride_row,
            (void *)&kern_stride_col, (void *)&kern_stride_row,
            (void *)&kern_stride_stack, (void *)&kern_stride_nkern};
        int err = GpuKernel_call(k, 3, threads_per_block, n_blocks, shmem_sz, kernel_params);

        if (err == GA_NO_ERROR)
        {
            if (verbose)
              fprintf(stderr, "INFO: used 'conv_full_patch_stack' version\n");
            work_complete = true;
        }
        else
        {
            if (verbose)
              fprintf(stderr, "INFO: impl 'conv_full_patch_stack' failed (%s), trying next implementation\n",
                      GpuKernel_error(k, err));
        }                         
    }
    if (1 && !work_complete) //conv_reference_full
    {
        if(verbose>1) fprintf(stderr, "INFO: will start conv_reference_full\n");

        size_t outsize = PyGpuArray_SIZE(out);
        size_t n_blocks[3] = {std::min(outsize, (size_t)4096),
                              (size_t)1, (size_t)1};
        size_t threads_per_block[3] = {std::min(ceil_intdiv(outsize, n_blocks[0]),
                                        (size_t)256),
                               (size_t)1, (size_t)1};

        void *kernel_params[] = {
            (void *)&nbatch, (void *)&nkern, (void *)&stack_len,
            (void *)&img_len, (void *)&img_wid,
            (void *)&kern_len, (void *)&kern_wid,
            (void *)&out_len, (void *)&out_wid,
            (void *)img->ga.data, (void *)&img->ga.offset,
            (void *)&img_stride_batch, (void *)&img_stride_stack,
            (void *)&img_stride_row, (void *)&img_stride_col,
            (void *)kern->ga.data, (void *)&kern->ga.offset,
            (void *)&kern_stride_nkern, (void *)&kern_stride_stack,
            (void *)&kern_stride_row, (void *)&kern_stride_col,
            (void *)out->ga.data, (void *)&out->ga.offset,
            (void *)&out_stride_batch, (void *)&out_stride_nkern,
            (void *)&out_stride_row, (void *)&out_stride_col,
            (void *)&subsample_rows, (void *)&subsample_cols};
        int err = GpuKernel_call(&conv_reference_full_node_<<<<HASH_PLACEHOLDER>>>>_0,
                                 3, threads_per_block, n_blocks, 0, kernel_params);

        if (err == GA_NO_ERROR)
        {
            if (verbose)
              fprintf(stderr, "INFO: used 'conv_reference_full' version"
                      " ishp(%d, %d) kshp(%d, %d) oshp(%d, %d) nbatch=%d"
                      " nkern=%d nstack=%d subsample=%d\n",
                      img_len,img_wid, kern_len, kern_wid,
                      out_len, out_wid, nbatch, nkern, nstack, subsample);
            work_complete = true;
        }
        else
        {
          if (verbose)
            fprintf(stderr, "INFO: impl 'conv_reference_full' failed (%s),"
                    " trying next implementation\n",
                    GpuKernel_error(&conv_reference_full_node_<<<<HASH_PLACEHOLDER>>>>_0, err));
          PyErr_Format(PyExc_RuntimeError,
                       "ERROR: all implementations failed for"
                       " CudaNdarray_conv_full! (%s)",
                       GpuKernel_error(&conv_reference_full_node_<<<<HASH_PLACEHOLDER>>>>_0, err));
          return -1;
        }
    }
    return 0;
}

PyObject *
PyGpuArray_Conv(PyGpuArrayObject *img, PyGpuArrayObject * kern,
                 PyGpuArrayObject * out, const int mode,
                 const size_t subsample_rows, const size_t subsample_cols,
                 const int version, const int verbose,
                 const int max_threads_dim0 = 512
                 )
{
    // Re-use the out object if possible.  If the out object it not used, then its refcount is not modified.
    //  If the out object is re-used then it is returned, and its refcount is incremented by 1.
    //
    if (PyGpuArray_NDIM(img) != 4)
    {
      PyErr_SetString(PyExc_ValueError, "PyGpuArray 4-D tensor required");
      return NULL;
    }
    if (PyGpuArray_NDIM(kern) != 4)
    {
      PyErr_SetString(PyExc_ValueError, "PyGpuArray 4-D tensor required");
      return NULL;
    }

    size_t out_dim[4];
    out_dim[0] = PyGpuArray_DIMS(img)[0];
    out_dim[1] = PyGpuArray_DIMS(kern)[0];
    size_t logical_rows, logical_cols;
    if (mode == ConvMode_VALID)
    {
        logical_rows = PyGpuArray_DIMS(img)[2] - PyGpuArray_DIMS(kern)[2] + 1;
        logical_cols = PyGpuArray_DIMS(img)[3] - PyGpuArray_DIMS(kern)[3] + 1;
    }
    else
    {
        logical_rows = PyGpuArray_DIMS(img)[2] + PyGpuArray_DIMS(kern)[2] - 1;
        logical_cols = PyGpuArray_DIMS(img)[3] + PyGpuArray_DIMS(kern)[3] - 1;
    }
    out_dim[2] = ceil_intdiv(logical_rows, subsample_rows);
    out_dim[3] = ceil_intdiv(logical_cols, subsample_cols);

    PyGpuArrayObject * rval = NULL;

    if ( out
         && PyGpuArray_NDIM(out)==4
         && out->ga.flags & GA_C_CONTIGUOUS
         && PyGpuArray_DIMS(out)[0]==out_dim[0]
         && PyGpuArray_DIMS(out)[1]==out_dim[1]
         && PyGpuArray_DIMS(out)[2]==out_dim[2]
         && PyGpuArray_DIMS(out)[3]==out_dim[3])
    {
      rval = out;
      Py_INCREF(rval);
      if (verbose)
        fprintf(stderr,
                "INFO: Conv is reusing the 'out' argument"
                " structure.\n");
    }
    else
    {
      if (out && verbose)
        fprintf(stderr,
                "INFO: Conv is ignoring 'out' argument with wrong"
                " structure.\n");
      else if(verbose)
        fprintf(stderr,
                "INFO: Conv don't have an 'out' argument"
                " structure.\n");

      rval = pygpu_zeros(4, out_dim,
                         img->ga.typecode, GA_C_ORDER,
                         img->context, Py_None);
      //rval might be null
    }
    if ((rval==NULL)
        || ((mode==ConvMode_VALID) && PyGpuArray_conv_valid(img, kern, rval,
                                                            subsample_rows,
                                                            subsample_cols,
                                                            version, verbose,
                                                            max_threads_dim0))
        || ((mode==ConvMode_FULL) && PyGpuArray_conv_full(img, kern, rval,
                                                          subsample_rows,
                                                          subsample_cols,
                                                          version, verbose,
                                                          max_threads_dim0))
            )
    {
        // if rval is something we just allocated,
        // and there was a problem, then we have to free it.
        Py_XDECREF(rval);
        return NULL;
    }
    return (PyObject*)rval;
}
