#include "hip/hip_runtime.h"
#define _CUDA_NDARRAY_C

#include <Python.h>
#include <structmember.h>

#include <numpy/arrayobject.h>
#include <iostream>

#include "cuda_ndarray.cuh"

//If true, when there is a gpu malloc or free error, we print the size of allocated memory on the device.
#define COMPUTE_GPU_MEM_USED 0

//If true, we fill with NAN allocated device memory.
#define ALLOC_MEMSET 0

/////////////////////////
// Alloc and Free
/////////////////////////

static int g_gpu_context_active = 0;


PyObject *
CudaNdarray_Dimshuffle(PyObject* _unused, PyObject* args);

/**
 *
 * In the test program I'm using, the _outstanding_mallocs decreases with every call.
 * This suggests there are more free() calls being made than alloc(), but I can't figure out why.
 *
 */
int _outstanding_mallocs[] = {0,0};
#if COMPUTE_GPU_MEM_USED
int _allocated_size = 0;
const int TABLE_SIZE = 10000;
struct table_struct{
    void* ptr;
    int size;
};
table_struct _alloc_size_table[TABLE_SIZE];
#endif
void * device_malloc(size_t size)
{
    void * rval=NULL;
    hipError_t err = hipMalloc(&rval, size);
    if (hipSuccess != err)
    {
#if COMPUTE_GPU_MEM_USED
        fprintf(stderr, "Error allocating %li bytes of device memory (%s). new total bytes allocated: %d\n", (long)size, hipGetErrorString(err),_allocated_size);
#else
        fprintf(stderr, "Error allocating %li bytes of device memory (%s).\n", (long)size, hipGetErrorString(err));
#endif
        PyErr_Format(PyExc_MemoryError, "Error allocating %li bytes of device memory (%s).", (long)size, hipGetErrorString(err));
        return NULL;
    }
    _outstanding_mallocs[0] += (rval != NULL);
#if COMPUTE_GPU_MEM_USED
    for(int i=0;i<TABLE_SIZE;i++){
        if(NULL==_alloc_size_table[i].ptr){
            _alloc_size_table[i].ptr=rval;
            _alloc_size_table[i].size=size;
            break;
        }
    }
    _allocated_size += size;
#endif
    //fprintf(stderr, "allocated %li bytes of device memory (%s). new total bytes allocated: %d. ptr: %p\n", (long)size, hipGetErrorString(err),_allocated_size,rval);

    if(ALLOC_MEMSET){
        //We init them to nan to make sure we catch more debug case.
        hipMemset(rval, 0xFF, size);
        //printf("MEMSET\n");
    }
    return rval;
}
int device_free(void *ptr)
{

    // if there is no gpu context, the call to hipFree will fail; skip it entirely
    if(!g_gpu_context_active) {
        return 0;
    }
    hipError_t err =  hipFree(ptr);
    if (hipSuccess != err)
    {
#if COMPUTE_GPU_MEM_USED
        fprintf(stderr, "Error freeing device pointer %p (%s).%d byte already allocated\n", ptr, hipGetErrorString(err), _allocated_size);
#else
        fprintf(stderr, "Error freeing device pointer %p (%s).\n", ptr, hipGetErrorString(err));
#endif
        PyErr_Format(PyExc_MemoryError, "error freeing device pointer %p (%s)", ptr, hipGetErrorString(err));
        return -1;
    }
    _outstanding_mallocs[0] -= (ptr != NULL);
#if COMPUTE_GPU_MEM_USED
    int i=0;
    size_t total_freed = 0;
    for(;i<TABLE_SIZE;i++)
        if(_alloc_size_table[i].ptr==ptr){
            _allocated_size -= _alloc_size_table[i].size;
            total_freed += _alloc_size_table[i].size;
            _alloc_size_table[i].ptr=0;
            _alloc_size_table[i].size=0;

            break;
        }
    if(i==TABLE_SIZE)
        printf("Unallocated unknow size!\n");
    //fprintf(stderr, "freed %li bytes of device memory (%s). %d already allocated, ptr=%p\n", (long)total_freed, hipGetErrorString(err),_allocated_size,ptr);
#endif
    return 0;
}
static PyObject *
outstanding_mallocs(PyObject* self, PyObject * args)
{
    return PyInt_FromLong(_outstanding_mallocs[0]);
}

/////////////////////////
// Static helper methods
/////////////////////////

static void
CudaNdarray_null_init(CudaNdarray*self)
{
    self->base = NULL;
    self->nd = -1;
    self->host_structure = NULL;
    self->data_allocated = 0;
    self->dev_structure_fresh = 1;
    self->dev_structure = NULL;
    self->devdata = NULL;
}

static int
CudaNdarray_uninit(CudaNdarray*self)
{
    int rval = 0;
    if (self->data_allocated) {
        assert(self->devdata);
        if (device_free(self->devdata))
        {
            fprintf(stderr,
                    "!!!! error freeing device memory %p (self=%p)\n",
                    self->devdata, self);
            rval = -1;
        }
        self->devdata = NULL;
        self->data_allocated = 0;
    }
    if (self->dev_structure)
    {
        if (device_free(self->dev_structure))
        {
            fprintf(stderr,
                    "!!!! error freeing dev_structure memory %p (self=%p)\n",
                    self->dev_structure, self);
            rval = -1;
        }
        self->dev_structure = NULL;
    }
    if (self->host_structure)
    {
        free(self->host_structure);
        self->host_structure = NULL;
    }
    self->nd = -1;
    Py_XDECREF(self->base);
    self->base = NULL;
    return rval;
}


//make the rightmost coords change fastest
//TODO: why does a downward for-loop not work????
//TODO: use the log2_dims and driver code to remove / and %
//TODO: skip the last division (when d == 0)
#define decl_k_elemwise_unary_rowmajor(name, F) \
__global__ void name (unsigned int numEls,  \
        unsigned int nd, \
        const int * dim,  \
        const float * a_data, const int * a_str, \
        float * z_data, const int * z_str) \
{ \
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; \
    const unsigned int numThreads = blockDim.x * gridDim.x; \
 \
    for (unsigned int i = idx; i < numEls; i += numThreads) \
    { \
        unsigned int ii = i; \
        const float * a_i = a_data; \
        float * z_i = z_data; \
        for (unsigned int _d = 0; _d < nd; ++_d) \
        { \
            unsigned int d = nd - _d-1;  \
            /* i_d used to be unsigned, but their is a bug in nvcc 3.0. making it signed fix the bug.*/\
            int i_d = ii % dim[d]; /* i_d is our position in the d'th dimension   */ \
            ii = ii / dim[d]; \
            a_i += i_d * a_str[d]; /* increment our a and z pointers by i_d elements */ \
            z_i += i_d * z_str[d]; \
        } \
        z_i[0] = F(a_i[0]); \
    } \
}

template<typename T> __device__ T unary_copy(T a) { return a; }
decl_k_elemwise_unary_rowmajor(k_elemwise_unary_rowmajor_copy, unary_copy<float>)

template<typename T> __device__ T unary_exp(T a) { return exp(a); }
decl_k_elemwise_unary_rowmajor(k_elemwise_unary_rowmajor_exp, unary_exp<float>)

/////////////////////////////
// Satisfying reqs to be Type
/////////////////////////////

//DON'T use directly(if their is other CudaNdarray that point to it, it will cause problem)! use Py_DECREF() instead
static void
CudaNdarray_dealloc(CudaNdarray* self)
{
    if (0) std::cerr << "CudaNdarray dealloc " << self << " " << self->devdata << '\n';
    if(self->ob_refcnt>1)
      printf("WARNING:CudaNdarray_dealloc called when there is still active reference to it.\n");
    CudaNdarray_uninit(self);
    self->ob_type->tp_free((PyObject*)self);
    --_outstanding_mallocs[1];
    if (0)
    {
        fprintf(stderr, "device_malloc_counts: (device) %i (obj) %i\n",
                _outstanding_mallocs[0],
                _outstanding_mallocs[1]);
    }
}

static PyObject *
CudaNdarray_new(PyTypeObject *type, PyObject *args, PyObject *kwds)
{
    CudaNdarray *self;

    self = (CudaNdarray *)type->tp_alloc(type, 0);
    if (self != NULL)
    {
        CudaNdarray_null_init(self);
        ++_outstanding_mallocs[1];
    }
    return (PyObject *)self;
}
static int
CudaNdarray_init(CudaNdarray *self, PyObject *args, PyObject *kwds)
{
    PyObject *arr=NULL;

    if (! PyArg_ParseTuple(args, "O", &arr))
        return -1;
    if (! PyArray_Check(arr))
    {
        PyErr_SetString(PyExc_TypeError, "PyArray arg required");
        return -1;
    }
    int rval = CudaNdarray_CopyFromArray(self, (PyArrayObject*)arr);
    return rval;
}
static PyMemberDef CudaNdarray_members[] =
{
    /*
    {"first", T_OBJECT_EX, offsetof(CudaNdarray, first), 0,
     "first name"},
    {"last", T_OBJECT_EX, offsetof(CudaNdarray, last), 0,
     "last name"},
    {"number", T_INT, offsetof(CudaNdarray, number), 0,
     "noddy number"},
     */
    {NULL}  /* Sentinel */
};

PyObject * CudaNdarray_CreateArrayObj(CudaNdarray * self)
{
    int verbose = 0;
    if(self->nd>=0 && CudaNdarray_SIZE(self)==0){
        npy_intp * npydims = (npy_intp*)malloc(self->nd * sizeof(npy_intp));
        assert (npydims);
        for (int i = 0; i < self->nd; ++i) npydims[i] = (npy_intp)(CudaNdarray_HOST_DIMS(self)[i]);
        PyObject * rval = PyArray_SimpleNew(self->nd, npydims, REAL_TYPENUM);
        free(npydims);
        if (!rval){
            return NULL;
        }
        assert (PyArray_ITEMSIZE(rval) == sizeof(real));
        return rval;
    }
    if ((self->nd < 0) || (self->devdata == 0))
    {
        PyErr_SetString(PyExc_ValueError, "can't copy from un-initialized CudaNdarray");
        return NULL;
    }
    CudaNdarray * contiguous_self = NULL;
    if (CudaNdarray_is_c_contiguous(self))
    {
        contiguous_self = self;
        Py_INCREF(contiguous_self);
        if (verbose) std::cerr << "CreateArrayObj already contiguous" << contiguous_self << '\n';
    }
    else
    {
        contiguous_self = (CudaNdarray*)CudaNdarray_Copy(self);
        if (verbose) std::cerr << "CreateArrayObj created contiguous" << contiguous_self << '\n';
    }
    if (!contiguous_self)
    {
        return NULL;
    }

    npy_intp * npydims = (npy_intp*)malloc(self->nd * sizeof(npy_intp));
    assert (npydims);
    for (int i = 0; i < self->nd; ++i)
        npydims[i] = (npy_intp)(CudaNdarray_HOST_DIMS(self)[i]);
    PyObject * rval = PyArray_SimpleNew(self->nd, npydims, REAL_TYPENUM);
    free(npydims);
    if (!rval)
    {
        Py_DECREF(contiguous_self);
        return NULL;
    }

    assert (PyArray_ITEMSIZE(rval) == sizeof(real));

    hipblasGetVector(PyArray_SIZE(rval), sizeof(real),
                    contiguous_self->devdata, 1,
                    PyArray_DATA(rval), 1);
    CNDA_THREAD_SYNC;

    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "error copying data to host");
        Py_DECREF(rval);
        rval = NULL;
    }

    Py_DECREF(contiguous_self);
    return rval;
}

// TODO-- we have two functions here, ZEROS and Zeros.
// ZEROS is meant to be called just from C code (you don't need to pass it PyObject * s)
// but this naming is very weird, makes it look like a macro
// we should figure out the correct convention and change to that
PyObject* CudaNdarray_ZEROS(int n, int * dims)
{

    int total_elements = 1;
    for(int i=0;i<n;i++)
        total_elements*=dims[i];

    // total_elements now contains the size of the array, in reals
    int total_size = total_elements * sizeof(real);

    CudaNdarray* rval = (CudaNdarray*)CudaNdarray_New();
    if (!rval)
    {
        PyErr_SetString(PyExc_RuntimeError, "CudaNdarray_ZEROS: call to New failed");
        return NULL;
    }

    if (CudaNdarray_alloc_contiguous(rval, n, dims))
    {
        PyErr_SetString(PyExc_RuntimeError, "CudaNdarray_ZEROS: allocation failed.");
        Py_DECREF(rval);
        return NULL;
    }

    // Fill with zeros
    //fprintf(stdout, "Sizeof: %d\n", total_size);
    if (hipSuccess != hipMemset(rval->devdata, 0, total_size))
    {
        PyErr_Format(PyExc_MemoryError, "CudaNdarray_ZEROS: Error memsetting %d bytes of device memory.", total_size);
        Py_DECREF(rval);
        return NULL;
    }

    if (cnda_copy_structure_to_device(rval))
    {
        PyErr_SetString(PyExc_RuntimeError, "CudaNdarray_ZEROS: syncing structure to device failed");
        Py_DECREF(rval);
        return NULL;
    }
    return (PyObject*) rval;
}

// declared as a static method (hence "dummy" is not used)
// Based on _Copy and _dimshuffle
PyObject* CudaNdarray_Zeros(PyObject* dummy, PyObject* shape)
{
    if(!shape)
    {
        PyErr_SetString(PyExc_TypeError, "CudaNdarray_Zeros: function takes at least 1 argument (0 given)");
        return NULL;
    }
    if(!PySequence_Check(shape))
    {
        PyErr_SetString(PyExc_TypeError, "shape argument must be a sequence");
        return NULL;
    }

    int shplen = PySequence_Length(shape);

    if (shplen == 0)
    {
        return CudaNdarray_ZEROS(0, NULL);
    }

    int* newdims = (int *)malloc(sizeof(int) * shplen);

    if (!newdims)
    {
        PyErr_SetString(PyExc_MemoryError,
            "CudaNdarray_Zeros: Failed to allocate temporary space");
        return NULL;
    }

    // start from the end to compute strides
    for (int i = shplen-1; i >= 0; --i)
    {
        PyObject* shp_el_obj = PySequence_GetItem(shape, i);
        if(shp_el_obj == NULL)
        {
            // shouldn't happen since we checked length before...
            PyErr_SetString(PyExc_RuntimeError, "CudaNdarray_Zeros: Index out of bound in sequence");
            free(newdims);
            return NULL;
        }

        int shp_el = PyInt_AsLong(shp_el_obj);
        Py_DECREF(shp_el_obj);

        if (shp_el < 0)
        {
            PyErr_SetString(PyExc_ValueError, "CudaNdarray_Zeros: shape must contain only non-negative values for size of a dimension");
            free(newdims);
            return NULL;
        }

        newdims[i] = shp_el;
    }

    PyObject* rval = CudaNdarray_ZEROS(shplen,newdims);

    free(newdims);

    return (PyObject*)rval;
}





PyObject * CudaNdarray_Copy(const CudaNdarray * self)
{
    PyObject * rval = CudaNdarray_New();
    if ((!rval) || (-1 == self->nd))
    {
        return rval;
    }
    if (CudaNdarray_alloc_contiguous((CudaNdarray*)rval, self->nd, CudaNdarray_HOST_DIMS(self)))
    {
        Py_DECREF(rval);
        return NULL;
    }
    if (CudaNdarray_CopyFromCudaNdarray((CudaNdarray*)rval, self))
    {
        Py_DECREF(rval);
        return NULL;
    }
    return rval;
}
PyObject * CudaNdarray_DeepCopy(CudaNdarray * self, PyObject * memo)
{
    assert(PyDict_Check(memo));
    PyObject * selfkey = PyInt_FromLong((long)self);
    assert(selfkey);
    if (PyDict_Contains(memo, selfkey))
    {
        PyObject * rval = PyDict_GetItem(memo, selfkey);
        Py_DECREF(selfkey);
        Py_XINCREF(rval);
        return rval;
    }
    else
    {
        PyObject * rval = CudaNdarray_Copy(self);
        if (0) std::cerr << "DeepCopy created " << rval << " devdata " << ((CudaNdarray*)rval)->devdata << "\n";
        if (NULL == rval)
        {
            Py_DECREF(selfkey);
            return NULL;
        }
        if (PyDict_SetItem(memo, selfkey, rval))
        {
            Py_DECREF(rval);
            Py_DECREF(selfkey);
            return NULL;
        }
        Py_DECREF(selfkey);
        return rval;
    }
}
PyObject * CudaNdarray_ReduceSum(CudaNdarray * self, PyObject * py_reduce_mask)
{
    if (!PySequence_Check(py_reduce_mask))
    {
        PyErr_SetString(PyExc_TypeError, "reduce_mask must be sequence of ints");
        return NULL;
    }
    int len = PySequence_Length(py_reduce_mask);
    if (len != self->nd)
    {
        PyErr_SetString(PyExc_TypeError, "length of reduce_mask must match self->nd");
        return NULL;
    }
    CudaNdarray * self_sum = (CudaNdarray*)CudaNdarray_New();
    if (!self_sum)
    {
        return NULL;
    }
    //TODO: allocate a fixed size dimshuffle_pattern_cache on the stack,
    //      and use it if it is big enough.
    int * dimshuffle_pattern = (int*)malloc(len * 2 * sizeof(int));
    int * sum_dims = dimshuffle_pattern + len;
    int n_remaining_dims = 0;
    if (!dimshuffle_pattern)
    {
        Py_DECREF(self_sum);
        PyErr_SetString(PyExc_MemoryError, "failed to alloc internal storage");
        return NULL;
    }
    for (int i = 0; i < len; ++i)
    {
        PyObject *o_i = PySequence_GetItem(py_reduce_mask, i);
        int o_i_int = PyInt_AsLong(o_i);
        Py_XDECREF(o_i);
        if (PyErr_Occurred())
        {
            Py_DECREF(self_sum);
            free(dimshuffle_pattern);
            return NULL;
        }
        if (o_i_int) // this is a dimension over which we are reducing
        {
            sum_dims[i] = 1;
        }
        else
        {
            sum_dims[i] = CudaNdarray_HOST_DIMS(self)[i];
            dimshuffle_pattern[n_remaining_dims++] = i;
        }
    }
    if (0   || CudaNdarray_alloc_contiguous(self_sum, len, sum_dims)
            || CudaNdarray_reduce_sum(self_sum, self)
            || CudaNdarray_dimshuffle(self_sum, n_remaining_dims, dimshuffle_pattern))
    {
        Py_DECREF(self_sum);
        free(dimshuffle_pattern);
        return NULL;
    }
    free(dimshuffle_pattern);
    return (PyObject*)self_sum;
}

__global__ void k_copy_reshape_rowmajor(unsigned int numEls,
        unsigned int a_nd, const float * a_data, const int * a_dim, const int * a_str,
        unsigned int z_nd, float * z_data, const int * z_dim, const int * z_str)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads)
    {
        const float * a_i = a_data;
        unsigned int a_ii = i;
        for (unsigned int _d = 0; _d < a_nd; ++_d) //make the rightmost coords change fastest
        {
            unsigned int d = a_nd - _d-1;
            unsigned int a_i_d = a_ii % a_dim[d];
            a_ii = a_ii / a_dim[d];
            a_i += a_i_d * a_str[d];
        }
        unsigned int z_ii = i;
        float * z_i = z_data;
        for (unsigned int _d = 0; _d < z_nd; ++_d) //make the rightmost coords change fastest
        {
            unsigned int d = z_nd - _d-1;
            //i tried to make the for loop count down, but it didn't work!?
            unsigned int z_i_d = z_ii % z_dim[d];
            z_i += z_i_d * z_str[d];
            z_ii = z_ii / z_dim[d];
        }
        z_i[0] = a_i[0]; //copy one lousy float!
    }
}

// Reshape self to the new shape gived by the tuple shape.
//
// If self is c contiguous, it return a view. Otherwise it always do a copy.
// TODO: make it return a view when the strides allow it even if it is not
//       c contiguous
PyObject * CudaNdarray_Reshape(CudaNdarray * self, PyObject * shape)
{
    // check shape tuple
    unsigned int rval_nd;
    unsigned int * rval_dims;
    unsigned int rval_size = 1;

    if (PyTuple_Check(shape)){
        // copy shape to integer array
        rval_nd = PyTuple_Size(shape);
    }else if (PyInt_Check(shape)){
        rval_nd = 1;
    }else{
        PyErr_SetString(PyExc_TypeError, "shape must be tuple of integers or an integer");
        return NULL;
    }
    rval_dims = (unsigned int*)malloc(rval_nd * sizeof(int));

    if(PyTuple_Check(shape)){
        for (int i = 0; i < rval_nd; ++i)
        {
            rval_dims[i] = PyInt_AsLong(PyTuple_GetItem(shape, i)); //GetItem returns borrowed reference
            if (PyErr_Occurred()) //error in AsLong
            {
                free(rval_dims);
                return NULL;
            }
            if(rval_dims[i]<=0){
                PyErr_Format(PyExc_ValueError, "Reshape has invalid dimension %i (must be >0)",rval_dims[i]);
                free(rval_dims);
                return NULL;
            }
            rval_size = rval_size * rval_dims[i];
        }
    }else{
        rval_size = PyInt_AsLong(shape);
        rval_dims[0] = rval_size;
    }
    // calculate new size, assert same as old size
    if (rval_size != CudaNdarray_SIZE(self))
    {
        PyErr_Format(PyExc_ValueError, "size must remain unchanged, changed from %i to %i", CudaNdarray_SIZE(self), rval_size);
        free(rval_dims);
        return NULL;
    }
    if (rval_size==0)
    {
        PyObject * rval = CudaNdarray_NewDims(rval_nd, rval_dims);
        free(rval_dims);
        return rval;
    }

    if(CudaNdarray_is_c_contiguous(self))
    {
        //return a view, not a copy
        CudaNdarray * rval = (CudaNdarray * )CudaNdarray_New(rval_nd);

        if (!rval || 0 != rval->data_allocated
            ||CudaNdarray_set_device_data(rval, CudaNdarray_DEV_DATA(self), self))
        {
            Py_XDECREF(rval);
            free(rval_dims);
            return NULL;
        }
        //set dim and stride
        int size = 1;
        for (int i = rval_nd-1; i >= 0; --i)
        {
            CudaNdarray_set_stride(rval, i, (rval_dims[i] == 1) ? 0 : size);
            CudaNdarray_set_dim(rval, i, rval_dims[i]);
            size = size * rval_dims[i];
        }
        free(rval_dims);
        return (PyObject*)rval;
    }

    // allocate new space (TODO: test to see if we can re-use old one)
    CudaNdarray * rval = (CudaNdarray * )CudaNdarray_New();
    if (!rval || CudaNdarray_alloc_contiguous(rval, rval_nd, rval_dims)){
        Py_XDECREF(rval);
        free(rval_dims);
        return NULL;
    }

    // call worker routine
    unsigned int threads_per_block = std::min(rval_size, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
    unsigned int n_blocks = std::min(ceil_intdiv(rval_size,threads_per_block), (unsigned int)NUM_VECTOR_OP_BLOCKS);
    k_copy_reshape_rowmajor<<<n_blocks,threads_per_block>>>(
            rval_size,
            self->nd,
            CudaNdarray_DEV_DATA(self), CudaNdarray_DEV_DIMS(self), CudaNdarray_DEV_STRIDES(self),
            rval->nd,
            CudaNdarray_DEV_DATA(rval), CudaNdarray_DEV_DIMS(rval), CudaNdarray_DEV_STRIDES(rval));

    CNDA_THREAD_SYNC;
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        Py_DECREF(rval);
        PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "k_copy_reshape_rowmajor", hipGetErrorString(err));
        free(rval_dims);
        return NULL;
    }
    free(rval_dims);
    return (PyObject*)rval;
}
PyObject * CudaNdarray_View(CudaNdarray * self)
{
    CudaNdarray * rval = (CudaNdarray*)CudaNdarray_New(self->nd);
    if (!rval || CudaNdarray_set_device_data(rval, CudaNdarray_DEV_DATA(self), self))
    {
        Py_XDECREF(rval);
        rval = NULL;
    }
    else
    {
        for (int i = 0; i < self->nd; ++i)
        {
            CudaNdarray_set_dim(rval, i, CudaNdarray_HOST_DIMS(self)[i]);
            CudaNdarray_set_stride(rval, i, CudaNdarray_HOST_STRIDES(self)[i]);
        }
    }
    return (PyObject*)rval;
}

PyObject * CudaNdarray_SetStride(CudaNdarray * self, PyObject *args)
{
    int pos, stride;
    if (! PyArg_ParseTuple(args, "ii", &pos, &stride))
        return NULL;
    if ((pos < 0) || (pos >= self->nd))
    {
        PyErr_Format(PyExc_ValueError, "position argument out of legal range [0, %i)", self->nd);
        return NULL;
    }
    CudaNdarray_set_stride(self, pos, stride);
    if (cnda_copy_structure_to_device(self))
    {
        return NULL;
    }
    Py_INCREF(Py_None);
    return Py_None;
}
PyObject * CudaNdarray_SetShapeI(CudaNdarray * self, PyObject *args)
{
    int pos, dim;
    if (! PyArg_ParseTuple(args, "ii", &pos, &dim))
        return NULL;
    if ((pos < 0) || (pos >= self->nd))
    {
        PyErr_Format(PyExc_ValueError, "position argument out of legal range [0, %i)", self->nd);
        return NULL;
    }
    CudaNdarray_set_dim(self, pos, dim);
    if (cnda_copy_structure_to_device(self))
    {
        return NULL;
    }
    Py_INCREF(Py_None);
    return Py_None;
}

static PyObject *
CudaNdarray_exp(CudaNdarray* self)
{
    CudaNdarray * rval = (CudaNdarray *)CudaNdarray_New();
    if ((NULL == rval) || CudaNdarray_alloc_contiguous(rval, self->nd, CudaNdarray_HOST_DIMS(self)))
    {
        Py_XDECREF(rval);
        return NULL;
    }
    unsigned int size = 1;
    for (int i = 0; i < self->nd; i++)
    {
        size *= (unsigned int) CudaNdarray_HOST_DIMS(self)[i];
    }
    unsigned int threads_per_block = std::min(size, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
    unsigned int n_blocks = std::min(ceil_intdiv(size,threads_per_block), (unsigned int)NUM_VECTOR_OP_BLOCKS);
    k_elemwise_unary_rowmajor_exp<<<n_blocks,threads_per_block>>>(size, self->nd, CudaNdarray_DEV_DIMS(self),
            CudaNdarray_DEV_DATA(self), CudaNdarray_DEV_STRIDES(self),
            CudaNdarray_DEV_DATA(rval), CudaNdarray_DEV_STRIDES(rval));

    //TODO: don't do this right away, do it when we need the result
    CNDA_THREAD_SYNC;
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        Py_DECREF(rval);
        PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "kExp", hipGetErrorString(err));
        return NULL;
    }

    return (PyObject*)rval;
}
static PyMethodDef CudaNdarray_methods[] =
{
    {"__array__",
        (PyCFunction)CudaNdarray_CreateArrayObj, METH_NOARGS,
        "Copy from the device to a numpy ndarray"},
    {"__copy__",
        (PyCFunction)CudaNdarray_View, METH_NOARGS,
        "Create a shallow copy of this object. used by module copy"},
    {"__deepcopy__",
        (PyCFunction)CudaNdarray_DeepCopy, METH_O,
        "Create a copy of this object"},
    {"zeros",
        (PyCFunction)CudaNdarray_Zeros, METH_STATIC,
        "Create a new CudaNdarray with specified shape, filled with zeros."},
    {"copy",
        (PyCFunction)CudaNdarray_Copy, METH_NOARGS,
        "Create a copy of this object"},
    {"is_c_contiguous",
        (PyCFunction)CudaNdarray_IS_C_Contiguous, METH_NOARGS,
        "Return True is the object is c contiguous. False otherwise."},
    {"reduce_sum",
        (PyCFunction)CudaNdarray_ReduceSum, METH_O,
        "Reduce over the given dimensions by summation"},
    {"exp",
        (PyCFunction)CudaNdarray_exp, METH_NOARGS,
        "Return the exponential of all elements"},
    {"reshape",
        (PyCFunction)CudaNdarray_Reshape, METH_O,
        "Return a reshaped view (or copy) of this ndarray\n\
            The required argument is a tuple of integers specifying the shape of the new ndarray."},
    {"view",
        (PyCFunction)CudaNdarray_View, METH_NOARGS,
        "Return an alias of this ndarray"},
    {"_set_stride",
        (PyCFunction)CudaNdarray_SetStride, METH_VARARGS,
        "For integer arguments (i, s), set the 'i'th stride to 's'"},
    {"_set_shape_i",
        (PyCFunction)CudaNdarray_SetShapeI, METH_VARARGS,
        "For integer arguments (i, s), set the 'i'th shape to 's'"},
    {NULL, NULL, NULL, NULL}  /* Sentinel */
};


////////////////////
// Number protocol
////////////////////

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] + b[i];
    }
}

// Will be called by __add__ in Python
static PyObject *
CudaNdarray_add(PyObject* py_self, PyObject * py_other)
{
    if (! CudaNdarray_Check(py_self)) {
        PyErr_SetString(PyExc_TypeError, "need a CudaNdarray on left");
        return NULL;
    }
    if (! CudaNdarray_Check(py_other)) {
        PyErr_SetString(PyExc_TypeError, "need a CudaNdarray on right");
        return NULL;
    }
    CudaNdarray * self = (CudaNdarray *)py_self;
    CudaNdarray * other = (CudaNdarray *)py_other;
    if(!CudaNdarray_is_c_contiguous(self) || !CudaNdarray_is_c_contiguous(other)){
        PyErr_SetString(PyExc_TypeError, "We have implementet only the c_contiguous version for now.");
        return NULL;
    }

    //standard elemwise size checks
    if (self->nd != other->nd)
    {
        PyErr_SetString(PyExc_TypeError, "CudaNdarray_add: need same number of dims");
        return NULL;
    }
    //standard elemwise dim checks
    unsigned int size = 1;
    for (int i = 0; i< self->nd; ++i)
    {
        if (CudaNdarray_HOST_DIMS(self)[i] != CudaNdarray_HOST_DIMS(other)[i])
        {
            PyErr_SetString(PyExc_TypeError, "need same dimensions");
            return NULL;
        }
        size *= (unsigned int) CudaNdarray_HOST_DIMS(self)[i];
    }
    CudaNdarray * rval = (CudaNdarray *)CudaNdarray_New();
    if (!rval || CudaNdarray_alloc_contiguous(rval, self->nd, CudaNdarray_HOST_DIMS(self)))
    {
        Py_XDECREF(rval);
        return NULL;
    }

    if(CudaNdarray_SIZE((CudaNdarray *)py_self)==0 && CudaNdarray_SIZE((CudaNdarray *)py_other)==0){
      return (PyObject *) rval;
    }

    int threads_per_block = std::min(size, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
    int n_blocks = std::min(ceil_intdiv(size,(unsigned int)threads_per_block), (unsigned int)NUM_VECTOR_OP_BLOCKS);
    kAdd_contiguous<<<n_blocks,threads_per_block>>>(
            self->devdata, other->devdata, rval->devdata, size);
    CNDA_THREAD_SYNC;
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "kAdd", hipGetErrorString(err));
        Py_DECREF(rval);
        return NULL;
    }
    return (PyObject *) rval;
}

enum operator_t
{
    IADD=0,
    IDIV,
    CPY,
    N_ELEMWISE_OPS // What this mean? It is not used
};

template <int operator_num>
__global__ void k_ielem_3(const int d0, const int d1, const int d2,
        float* a, const int sA0, const int sA1, const int sA2,
        const float* b, const int sB0, const int sB1, const int sB2){
    for (int i0 = blockIdx.x; i0 < d0; i0 += gridDim.x){
        for (int i1 = blockIdx.y; i1 < d1; i1 += gridDim.y){
            for (int i2 = threadIdx.x; i2 < d2; i2 += blockDim.x){
                switch (operator_num)
                {
                  case IADD:
                    a[i0*sA0 + i1*sA1 + i2*sA2] += b[i0*sB0 + i1*sB1 + i2*sB2];
                    break;
                  case IDIV:
                    a[i0*sA0 + i1*sA1 + i2*sA2] /= b[i0*sB0 + i1*sB1 + i2*sB2];
                    break;
                  case CPY:
                    a[i0*sA0 + i1*sA1 + i2*sA2] = b[i0*sB0 + i1*sB1 + i2*sB2];
                    break;
                }
            }
        }
    }
}

template <int operator_num>
__global__ void k_ielem_4(const int d0, const int d1, const int d2, const int d3,
                         float* a, const int sA0, const int sA1,
                         const int sA2, const int sA3,
                         const float* b, const int sB0, const int sB1,
                         const int sB2, const int sB3){
    for (int i0 = blockIdx.x; i0 < d0; i0 += gridDim.x){
        for (int i1 = blockIdx.y; i1 < d1; i1 += gridDim.y){
            for (int i2 = threadIdx.x; i2 < d2; i2 += blockDim.x){
                for (int i3 = threadIdx.y; i3 < d3; i3 += blockDim.y){
                    switch (operator_num) {
                        case IADD:
                            a[i0*sA0 + i1*sA1 + i2*sA2 + i3*sA3]
                            += b[i0*sB0 + i1*sB1 + i2*sB2 + i3*sB3];
                            break;
                        case IDIV:
                            a[i0*sA0 + i1*sA1 + i2*sA2 + i3*sA3]
                            /= b[i0*sB0 + i1*sB1 + i2*sB2 + i3*sB3];
                            break;
                        case CPY:
                            a[i0*sA0 + i1*sA1 + i2*sA2 + i3*sA3]
                            = b[i0*sB0 + i1*sB1 + i2*sB2 + i3*sB3];
                            break;
                    }
                }
            }
        }
    }
}

/*
CudaNdarray_inplace_elemwise
Compute elemwise, working inplace on A.
Currently implemented A / B, A + B and A = B
(the last is not tested and not used!)

py_self - the CudaNdarray that we'll modify (A)
py_other - the other argument (B)
fct_nb - which operation to perform (operator_t)

Returns 0 on success.
Returns -1 on failure, and sets Python exception.

*/
int
CudaNdarray_inplace_elemwise(PyObject* py_self, PyObject * py_other, operator_t fct_nb)
{
    int verbose = 0;
    void (*k3)(const int, const int, const int,
                    float*, const int, const int, const int,
                    const float*, const int, const int, const int);
    void (*k4)(const int, const int, const int, const int,
                    float*, const int, const int,
                    const int, const int,
                    const float*, const int, const int,
                    const int, const int);
    switch (fct_nb)
    {
        case IADD:
            k3 = k_ielem_3<IADD>;
            k4 = k_ielem_4<IADD>;
            break;
        case IDIV:
            k3 = k_ielem_3<IDIV>;
            k4 = k_ielem_4<IDIV>;
            break;
        case CPY:
            k3 = k_ielem_3<CPY>;
            k4 = k_ielem_4<CPY>;
            break;
        default:
            assert (0);
            PyErr_Format(
                PyExc_TypeError,
                "CudaNdarray_inplace_elemwise invalid fct_nb (%i).",
                (int)fct_nb);
            return -1;
    }
    if (!CudaNdarray_Check(py_self)) {
        PyErr_SetString(
            PyExc_TypeError,
            "CudaNdarray_inplace_elemwise need a CudaNdarray on left");
        return -1;
    }
    if (!CudaNdarray_Check(py_other)) {
        PyErr_SetString(
            PyExc_TypeError,
            "CudaNdarray_inplace_elemwise need a CudaNdarray on right");
        return -1;
    }

    CudaNdarray * self = (CudaNdarray *)py_self;
    CudaNdarray * other = (CudaNdarray *)py_other;

    if (verbose)
    {
        fprintf(stderr,
            "INPLACE ADD/DIV for self->nd=%d other->nd=%d\n",
            self->nd, other->nd);
    }

    //standard elemwise size checks
    if (self->nd != other->nd)
    {
        PyErr_Format(
            PyExc_TypeError,
            "CudaNdarray_inplace_elemwise: need same number of dims. Got %d and %d",
            self->nd, other->nd);
        return -1;
    }
    //standard elemwise dim checks
    unsigned int size = 1;
    for (int i = 0; i< self->nd; ++i)
    {
        if ((CudaNdarray_HOST_DIMS(self)[i] != CudaNdarray_HOST_DIMS(other)[i])
            && (CudaNdarray_HOST_DIMS(other)[i] != 1))
        {
            PyErr_SetString(
                PyExc_ValueError,
                "CudaNdarray_inplace_elemwise need same dimensions (or broadcastable dimension)");
            return -1;
        }
        // if we're broadcasting other, then make sure it has stride 0
        assert ((CudaNdarray_HOST_DIMS(self)[i] == CudaNdarray_HOST_DIMS(other)[i])
            || (CudaNdarray_HOST_STRIDES(other)[i] == 0));
        size *= (unsigned int) CudaNdarray_HOST_DIMS(self)[i];
    }

    if (size==0)
    {
        if (CudaNdarray_SIZE((CudaNdarray *)py_other))
        {
            PyErr_SetString(
                PyExc_ValueError,
                "CudaNdarray_inplace_elemwise cannot work inplace on an un-initialized array");
            return 0;
        }
        return 0;
    }

    switch(self->nd)
    {
        case 0:
            {
                dim3 n_blocks(1, 1, 1);
                dim3 n_threads(1);
                k3<<<n_blocks, n_threads>>>(
                        1, //d0
                        1, //d1
                        1, //d2
                        CudaNdarray_DEV_DATA(self),
                        1, //strides
                        1,
                        1,
                        CudaNdarray_DEV_DATA(other),
                        1, //strides
                        1,
                        1);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if (hipSuccess != err)
                {
                    PyErr_Format(
                        PyExc_RuntimeError,
                        "Cuda error: %s: %s.\n",
                        "k3",
                        hipGetErrorString(err));
                    return -1;
                }
            }
            break;
        case 1:
            {
                dim3 n_blocks(1, 1, 1);
                dim3 n_threads(
                        std::min(
                            CudaNdarray_HOST_DIMS(self)[0],
                            NUM_VECTOR_OP_THREADS_PER_BLOCK));
                k3<<<n_blocks, n_threads>>>(
                        1, //dimensions
                        1,
                        CudaNdarray_HOST_DIMS(self)[0],
                        CudaNdarray_DEV_DATA(self),
                        1, //strides
                        1,
                        CudaNdarray_HOST_STRIDES(self)[0],
                        CudaNdarray_DEV_DATA(other),
                        1, //strides
                        1,
                        CudaNdarray_HOST_STRIDES(other)[0]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if (hipSuccess != err)
                {
                    PyErr_Format(
                        PyExc_RuntimeError,
                        "Cuda error: %s: %s.\n",
                        "k3",
                        hipGetErrorString(err));
                    return -1;
                }
            }
            break;
        case 2:
            {
                //TODO:  if both self and other are f-contiguous
                //       Then flip the block and thread dimensions
                //       to make contiguous reads & writes
                dim3 n_blocks(1,
                        std::min(
                            CudaNdarray_HOST_DIMS(self)[0],
                            NUM_VECTOR_OP_BLOCKS));
                dim3 n_threads(
                        std::min(
                            CudaNdarray_HOST_DIMS(self)[1],
                            NUM_VECTOR_OP_THREADS_PER_BLOCK));
                k3<<<n_blocks, n_threads>>>(1,
                        CudaNdarray_HOST_DIMS(self)[0],
                        CudaNdarray_HOST_DIMS(self)[1],
                        CudaNdarray_DEV_DATA(self),
                        1,
                        CudaNdarray_HOST_STRIDES(self)[0],
                        CudaNdarray_HOST_STRIDES(self)[1],
                        CudaNdarray_DEV_DATA(other),
                        1,
                        CudaNdarray_HOST_STRIDES(other)[0],
                        CudaNdarray_HOST_STRIDES(other)[1]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if (hipSuccess != err)
                {
                    PyErr_Format(
                        PyExc_RuntimeError,
                        "Cuda error: %s: %s.\n",
                        "k3",
                        hipGetErrorString(err));
                    return -1;
                }
            }
            break;
        case 3:
            {
                //TODO:  Dimshuffle so that at least one of the arrays
                //       has a contiguous dimension on the thread idx.
                dim3 n_blocks(
                        std::min(
                            CudaNdarray_HOST_DIMS(self)[0],
                            NUM_VECTOR_OP_BLOCKS),
                        CudaNdarray_HOST_DIMS(self)[1]);
                while (n_blocks.x * n_blocks.y > NUM_VECTOR_OP_BLOCKS)
                    n_blocks.y /= 2;
                dim3 n_threads(
                        std::min(
                            CudaNdarray_HOST_DIMS(self)[2],
                            NUM_VECTOR_OP_THREADS_PER_BLOCK));
                k3<<<n_blocks, n_threads>>>(
                        CudaNdarray_HOST_DIMS(self)[0],
                        CudaNdarray_HOST_DIMS(self)[1],
                        CudaNdarray_HOST_DIMS(self)[2],
                        CudaNdarray_DEV_DATA(self),
                        CudaNdarray_HOST_STRIDES(self)[0],
                        CudaNdarray_HOST_STRIDES(self)[1],
                        CudaNdarray_HOST_STRIDES(self)[2],
                        CudaNdarray_DEV_DATA(other),
                        CudaNdarray_HOST_STRIDES(other)[0],
                        CudaNdarray_HOST_STRIDES(other)[1],
                        CudaNdarray_HOST_STRIDES(other)[2]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if (hipSuccess != err)
                {
                    PyErr_Format(
                        PyExc_RuntimeError,
                        "Cuda error: %s: %s.\n",
                        "k3",
                        hipGetErrorString(err));
                    return -1;
                }
            }
            break;
        case 4:
            {
                dim3 n_blocks(
                        std::min(
                            CudaNdarray_HOST_DIMS(self)[0],
                            NUM_VECTOR_OP_BLOCKS),
                        CudaNdarray_HOST_DIMS(self)[1]
                        );
                while (n_blocks.x * n_blocks.y > NUM_VECTOR_OP_BLOCKS)
                    n_blocks.y /= 2;
                dim3 n_threads(
                        std::min(
                            CudaNdarray_HOST_DIMS(self)[2],
                            NUM_VECTOR_OP_THREADS_PER_BLOCK)
                    //TODO: DON"T YOU NEED OT PUT DIMS[3] in here???
                            );
                k4<<<n_blocks, n_threads>>>(
                        CudaNdarray_HOST_DIMS(self)[0],
                        CudaNdarray_HOST_DIMS(self)[1],
                        CudaNdarray_HOST_DIMS(self)[2],
                        CudaNdarray_HOST_DIMS(self)[3],
                        CudaNdarray_DEV_DATA(self),
                        CudaNdarray_HOST_STRIDES(self)[0],
                        CudaNdarray_HOST_STRIDES(self)[1],
                        CudaNdarray_HOST_STRIDES(self)[2],
                        CudaNdarray_HOST_STRIDES(self)[3],
                        CudaNdarray_DEV_DATA(other),
                        CudaNdarray_HOST_STRIDES(other)[0],
                        CudaNdarray_HOST_STRIDES(other)[1],
                        CudaNdarray_HOST_STRIDES(other)[2],
                        CudaNdarray_HOST_STRIDES(other)[3]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if (hipSuccess != err)
                {
                    PyErr_Format(
                        PyExc_RuntimeError,
                        "Cuda error: %s: %s.\n",
                        "k4",
                        hipGetErrorString(err));
                    return -1;
                }
            }
            break;
        case 5:
            {
                dim3 n_blocks(
                        std::min(
                            CudaNdarray_HOST_DIMS(self)[1],
                            NUM_VECTOR_OP_BLOCKS),
                        CudaNdarray_HOST_DIMS(self)[2]);
                while (n_blocks.x * n_blocks.y > NUM_VECTOR_OP_BLOCKS)
                    n_blocks.y /= 2;
                dim3 n_threads(
                        std::min(
                            CudaNdarray_HOST_DIMS(self)[3],
                            NUM_VECTOR_OP_THREADS_PER_BLOCK)
                    //TODO: DON"T YOU NEED OT PUT DIMS[3] in here???
                    );
                for (int i = 0; i < CudaNdarray_HOST_DIMS(self)[0]; ++i)
                {
                     k4<<<n_blocks, n_threads>>>(
                            CudaNdarray_HOST_DIMS(self)[1],
                            CudaNdarray_HOST_DIMS(self)[2],
                            CudaNdarray_HOST_DIMS(self)[3],
                            CudaNdarray_HOST_DIMS(self)[4],
                            CudaNdarray_DEV_DATA(self) + i * CudaNdarray_HOST_STRIDES(self)[0],
                            CudaNdarray_HOST_STRIDES(self)[1],
                            CudaNdarray_HOST_STRIDES(self)[2],
                            CudaNdarray_HOST_STRIDES(self)[3],
                            CudaNdarray_HOST_STRIDES(self)[4],
                            CudaNdarray_DEV_DATA(other) + i * CudaNdarray_HOST_STRIDES(other)[0],
                            CudaNdarray_HOST_STRIDES(other)[1],
                            CudaNdarray_HOST_STRIDES(other)[2],
                            CudaNdarray_HOST_STRIDES(other)[3],
                            CudaNdarray_HOST_STRIDES(other)[4]);
                    CNDA_THREAD_SYNC;
                    hipError_t err = hipGetLastError();
                    if( hipSuccess != err)
                    {
                        PyErr_Format(
                            PyExc_RuntimeError,
                            "Cuda error: %s: %s.\n",
                            "k4",
                            hipGetErrorString(err));
                        return -1;
                    }
                }
            }
            break;
        default:
        {
            PyErr_Format(
                PyExc_NotImplementedError,
                "inplace_elemwise w nd=%i\n",
                self->nd);
            return -1;
        }
    }
    return 0;
}

/*
 * We need this inplace Add to support IncSubTensor
 * It returns py_self on success with an additional reference. Else NULL.
 */
// Will be called by __iadd__ in Python
static PyObject *
CudaNdarray_inplace_add(PyObject* py_self, PyObject * py_other)
{
    if (CudaNdarray_inplace_elemwise(py_self, py_other, IADD))
    {
        return NULL;
    }
    Py_INCREF(py_self);
    return py_self;
}

/*
 * We need this inplace div for cuda/tests/test_basic_ops.py:test_shared_options
 * It returns py_self on success with an additional reference. Else NULL.
 */
// Will be called by __idiv__ in Python
static PyObject *
CudaNdarray_inplace_div(PyObject* py_self, PyObject * py_other)
{
    if (CudaNdarray_inplace_elemwise(py_self, py_other, IDIV))
    {
        return NULL;
    }
    Py_INCREF(py_self);
    return py_self;
}

static PyNumberMethods CudaNdarrayNumberMethods =
{
     (binaryfunc)CudaNdarray_add,  //binaryfunc nb_add;  __add__
     0,  //binaryfunc nb_subtract;      __sub__
     0,  //binaryfunc nb_multiply;      __mul__
     0,  //binaryfunc nb_divide;        __div__
     0,  //binaryfunc nb_remainder;     __mod__
     0,  //binaryfunc nb_divmod;        __divmod__
     0,  //ternaryfunc nb_power;        __pow__
     0,  //unaryfunc nb_negative;       __neg__
     0,  //unaryfunc nb_positive;       __pos__
     0,  //unaryfunc nb_absolute;       __abs__
     0,  //inquiry nb_nonzero;          __nonzero__     /* Used by PyObject_IsTrue */
     0,  //unaryfunc nb_invert;         __invert__
     0,  //binaryfunc nb_lshift;        __lshift__
     0,  //binaryfunc nb_rshift;        __rshift__
     0,  //binaryfunc nb_and;           __and__
     0,  //binaryfunc nb_xor;           __xor__
     0,  //binaryfunc nb_or;            __or__
     0,  //coercion nb_coerce;          __coerce__     /* Used by the coerce() function */
     0,  //unaryfunc nb_int;            __int__
     0,  //unaryfunc nb_long;           __long__
     0,  //unaryfunc nb_float;          __float__
     0,  //unaryfunc nb_oct;            __oct__
     0,  //unaryfunc nb_hex;            __hex__

     /* Added in release 2.0 */
     (binaryfunc)CudaNdarray_inplace_add,  //binaryfunc nb_inplace_add;  __iadd__
     0,  //binaryfunc nb_inplace_subtract;      __isub__
     0,  //binaryfunc nb_inplace_multiply;      __imul__
     (binaryfunc)CudaNdarray_inplace_div,  //binaryfunc nb_inplace_divide;        __idiv__
     0,  //binaryfunc nb_inplace_remainder;     __imod__
     0,  //ternaryfunc nb_inplace_power;        __ipow__
     0,  //binaryfunc nb_inplace_lshift;        __ilshift__
     0,  //binaryfunc nb_inplace_rshift;        __irshift__
     0,  //binaryfunc nb_inplace_and;           __iand__
     0,  //binaryfunc nb_inplace_xor;           __ixor__
     0,  //binaryfunc nb_inplace_or;            __ior__

     /* Added in release 2.2 */
     0,  //binaryfunc nb_floor_divide;          __floordiv__
     0,  //binaryfunc nb_true_divide;           __truediv__
     0,  //binaryfunc nb_inplace_floor_divide;  __ifloordiv__
     0,  //binaryfunc nb_inplace_true_divide;   __itruediv__

#if PY_MINOR_VERSION > 4
     /* Added in release 2.5 */
     0  //unaryfunc nb_index;  __index__
#endif
};


/////////////////////
// Mapping protocol
/////////////////////

// Will by called by __len__ in Python
static Py_ssize_t
CudaNdarray_len(PyObject * py_self)
{
    CudaNdarray * self = (CudaNdarray*) py_self;
    if (self->nd <= 0)
    {
        return (Py_ssize_t) 0;
    }
    else
    {
        return (Py_ssize_t) CudaNdarray_HOST_DIMS(self)[0];
    }
}

// Will by called by __getitem__ in Python
static PyObject *
CudaNdarray_Subscript(PyObject * py_self, PyObject * key)
{
    int verbose = 0;
    if (verbose) fprintf(stderr, "Subscript .... \n");
    CudaNdarray * self = (CudaNdarray*) py_self;
    PyObject * py_rval = NULL;
    CudaNdarray * rval = NULL;
    PyObject * intobj = NULL;

    //PyObject_Print(key, stderr, 0);

    if (key == Py_Ellipsis)
    {
        Py_INCREF(py_self);
        return py_self;
    }
    if ((intobj=PyNumber_Int(key))) //INDEXING BY INTEGER
    //else if (PyInt_Check(key)) //INDEXING BY INTEGER
    {
        int d_idx = PyInt_AsLong(intobj);
        Py_DECREF(intobj); intobj=NULL;
        //int d_idx = PyInt_AsLong(key);
        if (self->nd == 0)
        {
            PyErr_SetString(PyExc_IndexError, "0-d arrays can't be indexed");
            return NULL;
        }
        int d_dim = CudaNdarray_HOST_DIMS(self)[0];
        int offset = 0;

        if ((d_idx >= 0) && (d_idx < d_dim))
        {
            //normal indexing
            offset += d_idx * CudaNdarray_HOST_STRIDES(self)[0];
        }
        else if ((d_idx < 0) && (d_idx >= -d_dim))
        {
            //end-based indexing
            // d_idx is negative
            offset += (d_dim + d_idx) * CudaNdarray_HOST_STRIDES(self)[0];
        }
        else
        {
            PyErr_SetString(PyExc_IndexError, "index out of bounds");
            return NULL;
        }

        //allocate our subtensor view
        py_rval = CudaNdarray_new_nd(self->nd - 1);
        rval = (CudaNdarray*) py_rval;
        if (!rval) return NULL;
        assert (0 == rval->data_allocated);

        //initialize the view's data pointer to our own.
        if (CudaNdarray_set_device_data(rval, CudaNdarray_DEV_DATA(self) + offset, self))
        {
            Py_DECREF(rval);
            return NULL;
        }
        for (int d = 1; d < self->nd; ++d)
        {
            CudaNdarray_set_stride(rval, d-1, CudaNdarray_HOST_STRIDES(self)[d]);
            CudaNdarray_set_dim(rval, d-1, CudaNdarray_HOST_DIMS(self)[d]);
        }
    }
    else
    {
        PyErr_Clear();
    }
    if (PySlice_Check(key)) //INDEXING BY SLICE
    {
        if (self->nd == 0)
        {
            PyErr_SetString(PyExc_ValueError, "cannot slice a 0-d array");
            return NULL;
        }

        int d_dim = CudaNdarray_HOST_DIMS(self)[0];
        Py_ssize_t start, stop, step, slen;
        if (PySlice_GetIndicesEx((PySliceObject*)key, d_dim, &start, &stop, &step, &slen))
        {
            return NULL;
        }
        if (verbose)
        {
            std::cerr << "start " << start << "\n";
            std::cerr << "stop " << stop << "\n";
            std::cerr << "step " << step << "\n";
            std::cerr << "slen " << slen << "\n";
        }

        //allocate our subtensor view
        py_rval = CudaNdarray_new_nd(self->nd);
        rval = (CudaNdarray*) py_rval;
        if (!rval) return NULL;
        assert (0 == rval->data_allocated);


        //initialize the view's data pointer to our own.
        if (CudaNdarray_set_device_data(rval,
                    CudaNdarray_DEV_DATA(self) + start * CudaNdarray_HOST_STRIDES(self)[0],
                    self))
        {
            Py_DECREF(rval);
            return NULL;
        }
        //initialize dimension 0 of rval
        CudaNdarray_set_stride(rval, 0, step * CudaNdarray_HOST_STRIDES(self)[0]);
        CudaNdarray_set_dim(rval, 0, slen);
        if (verbose) std::cerr << "rval stride " << CudaNdarray_HOST_STRIDES(rval)[0] << "\n";
        // initialize dimensions > 0 of rval
        for (int d = 1; d < self->nd; ++d)
        {
            CudaNdarray_set_stride(rval, d, CudaNdarray_HOST_STRIDES(self)[d]);
            CudaNdarray_set_dim(rval, d, CudaNdarray_HOST_DIMS(self)[d]);
        }
    }
    if (PyTuple_Check(key)) //INDEXING BY TUPLE
    {
        //elements of the tuple can be either integers or slices
        //the dimensionality of the view we will return is diminished for each slice in the tuple

        if (PyTuple_Size(key) > self->nd)
        {
            PyErr_SetString(PyExc_IndexError, "index error");
            return NULL;
        }

        //calculate the number of dimensions in the return value
        int rval_nd = self->nd;
        for (int d = 0; d < PyTuple_Size(key); ++d)
        {
            //On some paltform PyInt_Check(<type 'numpy.int64'>) return true, other it return false.
            //So we use PyArray_IsAnyScalar that should covert everything.
            rval_nd -= PyArray_IsAnyScalar(PyTuple_GetItem(key, d));
        }

        //allocate our subtensor view
        py_rval = CudaNdarray_new_nd(rval_nd);
        rval = (CudaNdarray*) py_rval;
        if (!rval) return NULL;
        assert (0 == rval->data_allocated);

        //initialize the view's data pointer to our own.
        if (CudaNdarray_set_device_data(rval, CudaNdarray_DEV_DATA(self), self))
        {
            Py_DECREF(rval);
            return NULL;
        }

        // rval_d will refer to the current dimension in the rval.
        // It will not be incremented for integer keys, but will be incremented for slice
        // keys
        int rval_d = 0;

        for (int d = 0; d < self->nd; ++d)
        {
            // keys can be shorter than self->nd.
            // when that happens, it means that the remaining dimensions are "full slices"
            if (d >=PyTuple_Size(key))
            {
                CudaNdarray_set_stride(rval, rval_d, CudaNdarray_HOST_STRIDES(self)[d]);
                CudaNdarray_set_dim(rval, rval_d, CudaNdarray_HOST_DIMS(self)[d]);
                ++rval_d;
            }
            else
            {
                PyObject * key_d = PyTuple_GetItem(key, d);

                if (PySlice_Check(key_d))
                {
                    Py_ssize_t start, stop, step, slen;
                    if (PySlice_GetIndicesEx((PySliceObject*)key_d, CudaNdarray_HOST_DIMS(self)[d], &start, &stop, &step, &slen))
                    {
                        Py_DECREF(rval);
                        return NULL;
                    }
                    rval->devdata += start * CudaNdarray_HOST_STRIDES(self)[d];
                    CudaNdarray_set_stride(rval, rval_d, step * CudaNdarray_HOST_STRIDES(self)[d]);
                    CudaNdarray_set_dim(rval, rval_d, slen);
                    if (0)
                    {
                        std::cerr << "start " << start << "\n";
                        std::cerr << "stop " << stop << "\n";
                        std::cerr << "step " << step << "\n";
                        std::cerr << "slen " << slen << "\n";
                    }
                    ++rval_d;
                }
                else if ((intobj=PyNumber_Int(key_d)))
                {
                    assert(PyArray_IsAnyScalar(key_d));
                    int d_idx = PyInt_AsLong(intobj);
                    Py_DECREF(intobj);
                    intobj = NULL;
                    int d_dim = CudaNdarray_HOST_DIMS(self)[d];

                    if ((d_idx >= 0) && (d_idx < d_dim))
                    {
                        //normal indexing
                        rval->devdata += d_idx * CudaNdarray_HOST_STRIDES(self)[d];
                    }
                    else if ((d_idx < 0) && (d_idx >= -d_dim))
                    {
                        //end-based indexing
                        rval->devdata += (d_dim + d_idx) * CudaNdarray_HOST_STRIDES(self)[d];
                    }
                    else
                    {
                        PyErr_SetString(PyExc_IndexError, "index out of bounds");
                        Py_DECREF(rval);
                        return NULL;
                    }
                }
                else
                {
                    PyErr_Clear(); // clear the error set by PyNumber_Int
                    PyErr_SetString(PyExc_IndexError, "index must be either int or slice");
                    Py_DECREF(rval);
                    return NULL;
                }
            }
        }
    }
    if (py_rval)
    {
        if (verbose) fprint_CudaNdarray(stderr, self);
        if (verbose) fprint_CudaNdarray(stderr, rval);
    }
    else
    {
        PyErr_SetString(PyExc_NotImplementedError, "Unknown key type");
        return NULL;
    }
    return py_rval;
}

// Will by called by __setitem__ in Python
// See http://docs.python.org/dev/py3k/c-api/object.html#PyObject_SetItem
// Doesn't handle broadcasting, e.g. a[:] = 5
// Can only be assigned from a CudaNdarray on the right side
// Or a ndarray when the left side part is c contiguous.
// Or a python scalar with value 0 when the left side part is c contiguous.
static int
CudaNdarray_setitem(PyObject *o, PyObject  *key, PyObject  *value)
{
    int verbose = 0;
    if (verbose) fprintf(stderr, "CudaNdarray_setitem start\n");
    // We try to copy directly into this CudaNdarray from the ndarray
    CudaNdarray* rval = (CudaNdarray*)CudaNdarray_Subscript(o, key);

    if(!rval){
        // CudaNdarray_Subscript failed and set the error msg.
        Py_XDECREF(rval);
        return -1;
    }

    if(rval != (CudaNdarray*)o &&
                (rval->data_allocated ||
                 // The new array should have a base
                 !(((CudaNdarray*)rval)->base) ||
                 // If the original array has no base, the base of the new
                 // array should be the original one
                 (!((CudaNdarray*)o)->base && ((CudaNdarray*)rval)->base != o) ||
                 // Else, the two arrays should have the same base
                 (((CudaNdarray*)o)->base && ((CudaNdarray*)rval)->base != ((CudaNdarray*)o)->base)))
    {
        // This case shouldn't happen, based on what I see in Subscript
        // but just in case it happens sometime in the future

        PyErr_Format(PyExc_RuntimeError, "__getitem__ must return a CudaNdarray that refers to the original CudaNdarray, not a copy. rval.base=%p o.base=%p o=%p",
                     (((CudaNdarray*)rval)->base), ((CudaNdarray*)o)->base, o);
        Py_DECREF(rval);
        return -1;
    }

    PyObject * intobj = NULL;
    if(CudaNdarray_Check(o)  && PyArray_Check(value)){
        if (verbose) fprintf(stderr, "CudaNdarray_setitem dest is a CudaNdarray and value is a ndarray\n");
        int typenum = PyArray_TYPE(value);
        if (typenum != REAL_TYPENUM){
            PyErr_SetString(PyExc_TypeError, "CudaNdarray.__setitem__: can only copy from float32 arrays");
            Py_XDECREF(rval);
            return -1;
        }
        if(! CudaNdarray_is_c_contiguous(rval)){
            PyErr_SetString(PyExc_NotImplementedError, "CudaNdarray.__setitem__: When the new value is an ndarray the part where we copy it to must be c contiguous.");
            Py_XDECREF(rval);
            return -1;
        }
        if(rval->nd != ((PyArrayObject*)value)->nd){
            PyErr_Format(PyExc_NotImplementedError, "CudaNdarray.__setitem__: need same number of dims. destination nd=%d, source nd=%d. broadcasting implemented only for zeroing values from python scalar.",
                         rval->nd,((PyArrayObject*)value)->nd);
            Py_XDECREF(rval);
            return -1;
        }
        for(int i=0 ; i<rval->nd ; i++){
          if(CudaNdarray_HOST_DIMS(rval)[i] != ((PyArrayObject*)value)->dimensions[i]){
            PyErr_Format(PyExc_ValueError, "CudaNdarray.__setitem__: need same dimensions for dim %d, destination=%d, source=%ld",
                i,
                CudaNdarray_HOST_DIMS(rval)[i],
                (long int)(((PyArrayObject*)value)->dimensions[i]));
            Py_XDECREF(rval);
            return -1;
          }
        }
        PyArrayObject * py_v = (PyArrayObject*)PyArray_ContiguousFromAny((PyObject*)value, typenum,
                                rval->nd, rval->nd);
        hipblasSetVector(PyArray_SIZE(py_v),
                        sizeof(real),
                        PyArray_DATA(py_v), 1,
                        rval->devdata, 1);
        CNDA_THREAD_SYNC;
        Py_XDECREF(py_v);
        Py_XDECREF(rval);
        if (HIPBLAS_STATUS_SUCCESS != cublasGetError()){
          PyErr_SetString(PyExc_RuntimeError, "CudaNdarray.__setitem__: error copying ndarray data to device memory");
          return -1;
        }
        return 0;
    }
    else if ((intobj=PyNumber_Int(value)))
    {
        if (verbose) fprintf(stderr, "CudaNdarray_setitem dest and value is a python number\n");
        if(! CudaNdarray_is_c_contiguous(rval)){
            PyErr_SetString(PyExc_NotImplementedError,
                            "CudaNdarray.__setitem__: When the new value is a scalar of value 0 the part where we copy to must be c contiguous.");
            Py_XDECREF(rval);
            return -1;
        }

        long val = PyInt_AsLong(intobj);
        Py_DECREF(intobj); intobj=NULL;
        if (val == 0)
        {
            hipError_t err = hipMemset(rval->devdata, 0, CudaNdarray_SIZE(rval) * sizeof(real));
            Py_XDECREF(rval);
            if (err)
            {
                PyErr_SetString(PyExc_RuntimeError,
                                "CudaNdarray.__setitem__: hipMemset failed");
                return -1;
            }
            return 0;
        } else {
            Py_XDECREF(rval);
            PyErr_SetString(PyExc_NotImplementedError,
                                "CudaNdarray.__setitem__: we support setting only python scalar of value 0, numpy nd array and CudaNdarray.");
                return -1;
        }
    }

    PyErr_Clear(); // clear PyNumber_Int error.

    if(!CudaNdarray_Check(o) || !CudaNdarray_Check(value))
    {
        PyErr_SetString(PyExc_TypeError, "CudaNdarray.__setitem__: left must be a CudaNdarrays and right must be a CudaNdarrays, an ndarray or a python scalar of value 0.");
        return -1;
    }

    if (verbose) fprintf(stderr, "CudaNdarray_setitem dest and value are CudaNdarray\n");
    if (cnda_copy_structure_to_device(rval))
    {
        PyErr_SetString(PyExc_RuntimeError, "CudaNdarray.__setitem__: syncing structure to device failed");
        Py_DECREF(rval);
        if (verbose) fprintf(stderr, "CudaNdarray_setitem error end\n");
        return -1;
    }

    PyObject *baseSavedForComparison = rval->base;

    if(CudaNdarray_CopyFromCudaNdarray(rval, (CudaNdarray*)value, true))
    {
        Py_DECREF((PyObject*)rval);
        if (verbose) fprintf(stderr, "CudaNdarray_setitem error end\n");
        return -1;
    }

    assert (rval->base == baseSavedForComparison);
    assert (rval->dev_structure_fresh);

    // Clean up locally-created references
    Py_DECREF(rval);

    return 0;
}


PyMappingMethods CudaNdarrayMappingMethods = {
    CudaNdarray_len, //lenfunc mp_length;               __len__
    CudaNdarray_Subscript, //binaryfunc mp_subscript;   __getitem__
    CudaNdarray_setitem //objobjargproc mp_ass_subscript;                __setitem__
};

////////////////////
//
////////////////////

static PyObject *
CudaNdarray_get_shape(CudaNdarray *self, void *closure)
{
    if (self->nd < 0)
    {
        PyErr_SetString(PyExc_ValueError, "CudaNdarray not initialized");
        return NULL;
    }
    PyObject * rval = PyTuple_New(self->nd);
    for (int i = 0; i < self->nd; ++i)
    {
        if (!rval || PyTuple_SetItem(rval, i, PyInt_FromLong(CudaNdarray_HOST_DIMS(self)[i])))
        {
            Py_XDECREF(rval);
            return NULL;
        }

    }
    return rval;
}

static int
CudaNdarray_set_shape(CudaNdarray *self, PyObject *value, void *closure)
{
    PyErr_SetString(PyExc_NotImplementedError, "TODO: call reshape");
    return -1;
}

static PyObject *
CudaNdarray_get_strides(CudaNdarray *self, void *closure)
{
    if (self->nd < 0)
    {
        PyErr_SetString(PyExc_ValueError, "CudaNdarray not initialized");
        return NULL;
    }
    PyObject * rval = PyTuple_New(self->nd);
    for (int i = 0; i < self->nd; ++i)
    {
        if (!rval || PyTuple_SetItem(rval, i, PyInt_FromLong(CudaNdarray_HOST_STRIDES(self)[i])))
        {
            Py_XDECREF(rval);
            return NULL;
        }

    }
    return rval;
}

static int
CudaNdarray_set_strides(CudaNdarray *self, PyObject *value, void *closure)
{
    PyErr_SetString(PyExc_NotImplementedError, "");
    return -1;
}

static PyObject *
CudaNdarray_get_dev_data(CudaNdarray *self, void *closure)
{
    float * p =  CudaNdarray_DEV_DATA(self);
    //printf("get_dev_data %p %li \n", p, (long int)p );
    return PyInt_FromLong((long int) CudaNdarray_DEV_DATA(self));
}

static int
CudaNdarray_set_dev_data(CudaNdarray *self, PyObject *value, void *closure)
{
    long int newdevdata = PyInt_AsLong(value);
    //printf("set_dev_data %p %li \n",(float*)newdevdata ,newdevdata);
    if (PyErr_Occurred())
    {
        return -1;
    }
    return  CudaNdarray_set_device_data(self, (float*)newdevdata, (CudaNdarray*)self->base);
}

static PyObject *
CudaNdarray_get_dtype(CudaNdarray *self, void *closure)
{
    return PyString_FromString("float32");
}

static PyObject *
CudaNdarray_get_ndim(CudaNdarray *self, void *closure)
{
    return PyInt_FromLong(self->nd);
}

static PyObject *
CudaNdarray_get_base(CudaNdarray *self, void *closure)
{
    PyObject * base = self->base;
    if (!base)
    {
        // We cannot return a NULL pointer, use None instead
        base = Py_None;
    }
    Py_INCREF(base);
    return base;
}

void put_in_dict(PyObject * dict, const char * key, int val)
{
  PyObject * k = PyString_FromString(key);
  PyObject * v = PyInt_FromLong(val);
  PyDict_SetItem(dict, k, v);
  Py_DECREF(k);
  Py_DECREF(v);
}

PyObject *
GetDeviceProperties(PyObject* _unused, PyObject* args)
{
  int dev_id = -1;
  if (! PyArg_ParseTuple(args, "i", &dev_id))
    return NULL;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev_id);

  PyObject * dict = PyDict_New();
  PyObject * str= PyString_FromString("name");
  PyObject * i = PyString_FromString(deviceProp.name);
  PyDict_SetItem(dict, str, i);
  Py_DECREF(str);
  Py_DECREF(i);

  put_in_dict(dict, "major", deviceProp.major);
  put_in_dict(dict, "minor", deviceProp.minor);
#if CUDART_VERSION >= 2020
  int driverVersion = 0, runtimeVersion = 0;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  put_in_dict(dict, "driverVersion", driverVersion);
  put_in_dict(dict, "runtimeVersion", runtimeVersion);
#endif
#if CUDART_VERSION >= 2000

  put_in_dict(dict, "multiProcessorCount", deviceProp.multiProcessorCount);
  //if ConvertSMVer2Cores is not defined in hip/hip_runtime_api.h, the run time is too old.
  int sm_cores = -1;
  if(deviceProp.major==1)
    sm_cores = 32;
  else if(deviceProp.major==2 && deviceProp.minor==0)
    sm_cores = 32;
  else if(deviceProp.major==2 && deviceProp.minor==1)
    sm_cores = 48;
  put_in_dict(dict, "coresCount", sm_cores * deviceProp.multiProcessorCount);
#endif
  put_in_dict(dict, "totalConstMem", deviceProp.totalConstMem);
  put_in_dict(dict, "sharedMemPerBlock", deviceProp.sharedMemPerBlock);
  put_in_dict(dict, "regsPerBlock", deviceProp.regsPerBlock);
  put_in_dict(dict, "warpSize", deviceProp.warpSize);
  put_in_dict(dict, "maxThreadsPerBlock", deviceProp.maxThreadsPerBlock);
  put_in_dict(dict, "maxThreadsDim0", deviceProp.maxThreadsDim[0]);
  put_in_dict(dict, "maxThreadsDim1", deviceProp.maxThreadsDim[1]);
  put_in_dict(dict, "maxThreadsDim2", deviceProp.maxThreadsDim[2]);
  put_in_dict(dict, "maxGridSize0", deviceProp.maxGridSize[0]);
  put_in_dict(dict, "maxGridSize1", deviceProp.maxGridSize[1]);
  put_in_dict(dict, "maxGridSize2", deviceProp.maxGridSize[2]);
  put_in_dict(dict, "memPitch", deviceProp.memPitch);
  put_in_dict(dict, "textureAlignment", deviceProp.textureAlignment);
  put_in_dict(dict, "clockRate", deviceProp.clockRate);
#if CUDART_VERSION >= 2000
  put_in_dict(dict, "deviceOverlap", deviceProp.deviceOverlap);
#endif
#if CUDART_VERSION >= 2020
  put_in_dict(dict, "kernelExecTimeoutEnabled", deviceProp.kernelExecTimeoutEnabled);
  put_in_dict(dict, "integrated", deviceProp.integrated);
  put_in_dict(dict, "canMapHostMemory", deviceProp.canMapHostMemory);
  put_in_dict(dict, "computeMode", deviceProp.computeMode);
  //in the doc of this fct tell that 0 - Normal mode, 1 - only 1 context, 2 - no context
#endif
#if CUDART_VERSION >= 3000
  put_in_dict(dict, "concurrentKernels", deviceProp.concurrentKernels);
#endif
#if CUDART_VERSION >= 3010
  put_in_dict(dict, "ECCEnabled", deviceProp.ECCEnabled);
#endif
#if CUDART_VERSION >= 3020
  put_in_dict(dict, "tccDriver", deviceProp.tccDriver);
#endif

  return dict;
}

/*
 * Returns in *free and *total respectively, the free and total amount of memory available for allocation by the device in bytes.
 */
PyObject *
GetDeviceMemInfo(PyObject* _unused, PyObject* dummy)
{
    size_t free = 0, total = 0;
    if(g_gpu_context_active == 0){
        PyErr_Format(PyExc_RuntimeError, "No gpu device selected yet. Please make sure the gpu device was initialized by Theano before.");
        return NULL;
    }

    hipError_t err = hipMemGetInfo(&free, &total);
    if (err != hipSuccess){
        PyErr_Format(PyExc_RuntimeError, "Error while getting memory info about the gpu %d");
        return NULL;
    }
    return PyTuple_Pack(2, PyLong_FromLong(free), PyLong_FromLong(total));
}

#if COMPUTE_GPU_MEM_USED
/*
 * Return the size in bytes that Theano currently have allocated on the gpu.
 */
PyObject *
GetTheanoAllocInfo(PyObject* _unused, PyObject* dummy)
{
    return PyLong_FromLong(_allocated_size);
}
#endif

static PyGetSetDef CudaNdarray_getset[] = {
    {"shape",
        (getter)CudaNdarray_get_shape,
        (setter)CudaNdarray_set_shape,
        "shape of this ndarray (tuple)",
        NULL},
    {"_strides",
        (getter)CudaNdarray_get_strides,
        (setter)CudaNdarray_set_strides,
        "data pointer strides (in elements)",
        NULL},
    //gpudata is needed to allow calling pycuda fct with CudaNdarray input.
    {"gpudata",
        (getter)CudaNdarray_get_dev_data,
        NULL,
        "device data pointer",
        NULL},
    {"_dev_data",
        (getter)CudaNdarray_get_dev_data,
        (setter)CudaNdarray_set_dev_data,
        "device data pointer",
        NULL},
    {"dtype",
        (getter)CudaNdarray_get_dtype,
        NULL,
        "The dtype of the element. Now always float32",
        NULL},
    {"size",
        (getter)CudaNdarray_SIZE_Object,
        NULL,
        "The number of elements in this object.",
        NULL},
    //mem_size is neede for pycuda.elementwise.ElementwiseKernel Why do they use size and mem_size of the same value?
    {"mem_size",
        (getter)CudaNdarray_SIZE_Object,
        NULL,
        "The number of elements in this object.",
        NULL},
    {"ndim",
        (getter)CudaNdarray_get_ndim,
        NULL,
        "The number of dimensions in this object.",
        NULL},
    {"base",
        (getter)CudaNdarray_get_base,
        NULL,
        "If this ndarray is a view, base is the original ndarray.",
        NULL},

    {NULL, NULL, NULL, NULL}  /* Sentinel */
};



static PyTypeObject CudaNdarrayType =
{
    PyObject_HEAD_INIT(NULL)
    0,                         /*ob_size*/
    "CudaNdarray",             /*tp_name*/
    sizeof(CudaNdarray),       /*tp_basicsize*/
    0,                         /*tp_itemsize*/
    (destructor)CudaNdarray_dealloc, /*tp_dealloc*/
    0,                         /*tp_print*/
    0,                         /*tp_getattr*/
    0,                         /*tp_setattr*/
    0,                         /*tp_compare*/
    0,                         /*tp_repr*/
    &CudaNdarrayNumberMethods, /*tp_as_number*/
    0,                         /*tp_as_sequence*/
    &CudaNdarrayMappingMethods,/*tp_as_mapping*/
    0,                         /*tp_hash */
    0,                         /*tp_call*/
    0,                         /*tp_str*/
    0,                         /*tp_getattro*/
    0,                         /*tp_setattro*/
    0,                         /*tp_as_buffer*/
    Py_TPFLAGS_DEFAULT | Py_TPFLAGS_BASETYPE | Py_TPFLAGS_CHECKTYPES, /*tp_flags*/
    "CudaNdarray objects",     /* tp_doc */
    0,                         /* tp_traverse */
    0,                         /* tp_clear */
    0,                         /* tp_richcompare */
    0,                         /* tp_weaklistoffset */
    0,                         /* tp_iter */
    0,                         /* tp_iternext */
    CudaNdarray_methods,       /* tp_methods */
    CudaNdarray_members,       /* tp_members */
    CudaNdarray_getset,        /* tp_getset */
    0,                         /* tp_base */
    0,                         /* tp_dict */
    0,                         /* tp_descr_get */
    0,                         /* tp_descr_set */
    0,                         /* tp_dictoffset */
    (initproc)CudaNdarray_init,/* tp_init */
    0,                         /* tp_alloc */
    CudaNdarray_new,           /* tp_new */
};

static __global__ void get_gpu_ptr_size(int* dst)
{
    dst[0] = sizeof(float*);
    dst[1] = sizeof(int);
}

PyObject *
CudaNdarray_ptr_int_size(PyObject* _unused, PyObject* args)
{
    int *gpu_data = (int*)device_malloc(sizeof(int)*2);
    if(gpu_data == NULL){
        return PyErr_Format(PyExc_MemoryError,
                            "CudaNdarray_ptr_int_size: Can't allocate memory on the gpu.");
    }
    get_gpu_ptr_size<<<1,1>>>(gpu_data);
    if (hipSuccess != cublasGetError()){

        device_free(gpu_data);
        return PyErr_Format(PyExc_RuntimeError,
                            "CudaNdarray_ptr_int_size: error when calling the gpu code.");
    }

    // Transfer the result to cpu
    int gpu_sizes[] = {-1,-1};
    hipblasGetVector(2, sizeof(int), gpu_data, 1, gpu_sizes, 1);
    device_free(gpu_data);

    if (HIPBLAS_STATUS_SUCCESS != cublasGetError()){
        PyErr_SetString(PyExc_RuntimeError, "error copying data to from memory");
        return NULL;
    }
    return Py_BuildValue("iiii", gpu_sizes[0], sizeof(float*), sizeof(int), gpu_sizes[1]);
}

// Initialize the gpu.
// Takes one optional parameter, the device number.
// If provided, it sets that device to be the active device.
// If not provided (usually just to test whether the gpu is available at all),
// it does not set an active device.
// Raises EnvironmentError or ValueError (as appropriate) if the initialization failed.
PyObject *
CudaNdarray_gpu_init(PyObject* _unused, PyObject* args)
{
    int card_nb = 0;
    int card_number_provided = 1;

    PyArg_ParseTuple(args, "|i", &card_nb); // if we're given something wildly invalid, this will throw a TypeError

    if(PyTuple_Size(args) == 0) {
        card_number_provided = 0;
        card_nb = 0;
    }

    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if(hipSuccess != err) {
        return PyErr_Format(PyExc_EnvironmentError,
                            "Unable to get the number of gpus available: %s",
                            hipGetErrorString(hipGetLastError()));
    }

    // as soon as the first successful call to a cuda* function is made, a
    // gpu context has been created
    g_gpu_context_active = 1;

    if(deviceCount <= 0) {
        return PyErr_Format(PyExc_EnvironmentError,
                            "Can't use the GPU, no devices support CUDA");
    }
    if(card_number_provided && (card_nb < 0 || card_nb > (deviceCount - 1))) {
        return PyErr_Format(PyExc_ValueError,
                            "Bad device number %d. Only %d devices available.",
                            card_nb,
                            deviceCount);
    }

    hipDeviceProp_t deviceProp;
    err = hipGetDeviceProperties(&deviceProp, card_nb);
    if(hipSuccess != err) {
        return PyErr_Format(PyExc_EnvironmentError,
                            "Unable to get properties of gpu %i: %s",
                            card_nb,
                            hipGetErrorString(hipGetLastError()));
    }

    if(deviceProp.major == 9999 && deviceProp.minor == 9999 ){
        return PyErr_Format(PyExc_EnvironmentError,
                            "There is no device that supports CUDA");
    }

    if(card_number_provided) {
        err = hipSetDevice(card_nb);
        if(hipSuccess != err) {
            return PyErr_Format(PyExc_EnvironmentError,
                                "Unable to set device %i: %s",
                                card_nb,
                                hipGetErrorString(hipGetLastError()));
        }
    }

    Py_INCREF(Py_None);
    return Py_None;
}

PyObject *
CudaNdarray_active_device_number(PyObject* _unused, PyObject* _unused_args) {
    // NB: No cuda error checking here; keeps things simple, and it's not
    // really necessary.
    int currentDevice;
    hipGetDevice(&currentDevice);
    return PyInt_FromLong(currentDevice);
}

PyObject *
CudaNdarray_active_device_name(PyObject* _unused, PyObject* _unused_args) {
    // NB: No cuda error checking here; keeps things simple, and it's not
    // really necessary.
    int currentDevice;
    hipGetDevice(&currentDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, currentDevice);
    return PyString_FromString(deviceProp.name);
}

PyObject *
CudaNdarray_gpu_shutdown(PyObject* _unused, PyObject* _unused_args) {
    hipDeviceReset();
    g_gpu_context_active = 0; // context has now been closed down
    Py_INCREF(Py_None);
    return Py_None;
}

/*
 * This function is tested in theano/misc/test_pycuda_theano_simple.py
 */
PyObject *
CudaNdarray_from_gpu_pointer(PyObject* _unused, PyObject* args)
{
    int verbose = 0;
    PyObject *gpu_ptr = NULL;
    PyObject *shapes = NULL;
    PyObject *strides = NULL;
    PyObject *base = NULL;
    PyObject *rval = NULL;

    //args should consist of 3 python objects
    //The first is the gpu ptr
    //The second if the shape
    //The third if the strides
    if (! PyArg_ParseTuple(args, "OOOO", &gpu_ptr, &shapes, &strides, &base))
        return NULL;

    if (verbose) printf("In CudaNdarray_from_gpu_pointer\n");
    if (!PyLong_Check(gpu_ptr))
    {
        PyErr_Format(PyExc_Exception, "CudaNdarray_from_gpu_pointer: The gpu pointor is not an long");
        return NULL;
    }

    Py_ssize_t nd =  PyObject_Length(shapes);
    if (nd < 0)
    {
        PyErr_SetString(PyExc_TypeError, "CudaNdarray_from_gpu_pointer: Couldn't get length of second argument");
        return NULL;
    }
    Py_ssize_t nd_stride =  PyObject_Length(strides);
    if (nd_stride < 0)
    {
        PyErr_SetString(PyExc_TypeError, "CudaNdarray_from_gpu_pointer: Couldn't get length of third argument");
        return NULL;
    }

    if (nd != nd_stride)
    {
        PyErr_SetString(PyExc_TypeError, "CudaNdarray_from_gpu_pointer: We need the same number of shapes and strides");
        return NULL;
    }

    rval = CudaNdarray_New();

    if (CudaNdarray_set_nd((CudaNdarray *)rval, nd))
    {
        //CudaNdarray_set_nd set the error msg
        return NULL;
    }
    // set gpu pointeur
    assert(((CudaNdarray *)rval)->data_allocated == 0);
    if (CudaNdarray_set_device_data((CudaNdarray *)rval, (float *)PyInt_AsLong(gpu_ptr), base))
    {
        PyErr_SetString(PyExc_TypeError, "CudaNdarray_from_gpu_pointer: Error while setting the gpu pointor");
        return NULL;

    }

    // Set dims and strides
    for (int i = nd-1; i >= 0; --i)
    {
        PyObject * idx = PyLong_FromLong(i);
        if (idx == NULL)
        {
            PyErr_SetString(PyExc_Exception, "CudaNdarray_from_gpu_pointer: Couldn't make long object to loop over list/tuple");
            return NULL;
        }
        PyObject* dim_ = PyObject_GetItem(shapes, idx);
        PyObject* strd_ = PyObject_GetItem(strides, idx);
        if (!PyInt_Check(dim_))
        {
            PyErr_Format(PyExc_Exception, "CudaNdarray_from_gpu_pointer: shapes[%d] is not an int", i);
            return NULL;
        }
        if (!PyInt_Check(strd_))
        {
            PyErr_Format(PyExc_Exception, "CudaNdarray_from_gpu_pointer: strides[%d] is not an int", i);
            return NULL;
        }
        int dim = PyInt_AsLong(dim_);
        int strd = PyInt_AsLong(strd_);
        CudaNdarray_set_stride((CudaNdarray *)rval, i, strd);
        CudaNdarray_set_dim((CudaNdarray *)rval, i, dim);
        Py_DECREF(idx);
        Py_DECREF(dim_);
        Py_DECREF(strd_);
    }
    if (verbose) printf("CudaNdarray_from_gpu_pointer normal return\n");
    return rval;
}

PyObject *
CudaNdarray_Dot(PyObject* _unused, PyObject* args)
{
    PyObject *l=NULL;
    PyObject *r=NULL;
    PyObject * rval = NULL;

    //args should consist of two python objects ("OO")
    if (! PyArg_ParseTuple(args, "OO", &l, &r))
        return NULL;

    if (!CudaNdarray_Check(l) || !CudaNdarray_Check(r))
    {
        PyErr_SetString(PyExc_TypeError, "CudaNdarray arguments required ");
        goto CudaNdarray_dot_fail;
    }
    if (((CudaNdarray*)l)->nd != 2)
    {
        PyErr_SetString(PyExc_TypeError, "need 2d CudaNdarray arg for now");
        goto CudaNdarray_dot_fail;
    }
    if (((CudaNdarray*)r)->nd != 2)
    {
        PyErr_SetString(PyExc_TypeError, "need 2d CudaNdarray arg for now");
        goto CudaNdarray_dot_fail;
    }
    rval = CudaNdarray_New();
    if (!rval)
    {
        goto CudaNdarray_dot_fail;
    }
    int dims[2];
    dims[0] = CudaNdarray_HOST_DIMS((CudaNdarray*)l)[0];
    dims[1] = CudaNdarray_HOST_DIMS((CudaNdarray*)r)[1];
    if (CudaNdarray_alloc_contiguous((CudaNdarray*)rval, 2, dims))
    {
        goto CudaNdarray_dot_fail;
    }
    if (CudaNdarray_gemm(1.0, (CudaNdarray*)l, (CudaNdarray*)r, 0.0, (CudaNdarray*)rval))
    {
        goto CudaNdarray_dot_fail;
    }

    return rval;

    CudaNdarray_dot_fail:
    Py_XDECREF(rval);
    return NULL;
}

static PyObject *
filter(PyObject* __unsed_self, PyObject *args) // args = (data, broadcastable, strict, storage)
{
    /*
     * TODO: DOC what this function should do in the various cases of
     * What is 'strict' supposed to mean in the context of this function?
     * What do we do with input that could be interpreted as matching the broadcastable pattern in strict vs. non-strict cases?
     *
     */
    PyObject *py_data=NULL;
    PyArrayObject * data = NULL;
    int strict = 0;
    PyObject * broadcastable=NULL;
    PyObject * storage=NULL;
    CudaNdarray * rval=NULL;

    //Python object references which are provided to the caller are borrowed references
    if (!PyArg_ParseTuple(args, "OOiO", &py_data, &broadcastable, &strict, &storage)) return NULL;

    if (!PyTuple_Check(broadcastable)){
        PyErr_SetString(PyExc_TypeError, "broadcastable arg should be a tuple of int.");
        return NULL;
    }
    Py_INCREF(py_data);
    Py_INCREF(broadcastable);

    CudaNdarray * cnda = (CudaNdarray*)py_data;

    if (strict || CudaNdarray_Check(py_data))
    {
        //TODO: support non-strict "casting" from a vt to the broadcastable/type/size that we need.
        if (!CudaNdarray_Check(py_data))
        {
            Py_DECREF(py_data);
            Py_DECREF(broadcastable);
            PyErr_SetString(PyExc_TypeError, "strict mode requires CudaNdarray");
            return NULL;
        }
        if (cnda->nd != PyTuple_Size(broadcastable))
        {
            Py_DECREF(py_data);
            Py_DECREF(broadcastable);
            PyErr_Format(PyExc_TypeError, "Wrong rank: %i vs %li", cnda->nd, (long)PyTuple_Size(broadcastable));
            return NULL;
        }
        for (int i = 0; i < cnda->nd; ++i)
        {
            if ((CudaNdarray_HOST_DIMS(cnda)[i] > 1) && PyInt_AsLong(PyTuple_GetItem(broadcastable, Py_ssize_t(i))))
            {
                PyErr_Format(PyExc_TypeError, "Non-unit size in broadcastable vt dimension %i", i);
                Py_DECREF(py_data);
                Py_DECREF(broadcastable);
                return NULL;
            }
        }
        Py_DECREF(broadcastable);
        return py_data;
    }
    else
    {
        data = (PyArrayObject*)PyArray_FromObject(py_data, REAL_TYPENUM, PyTuple_Size(broadcastable), PyTuple_Size(broadcastable));
        if (!data)
        {
            //err message already defined
            Py_DECREF(py_data);
            Py_DECREF(broadcastable);
            return NULL;
        }
        for (int i = 0; i < data->nd; ++i)
        {
            if ((data->dimensions[i] > 1) && PyInt_AsLong(PyTuple_GetItem(broadcastable, Py_ssize_t(i))))
            {
                PyErr_Format(PyExc_TypeError, "Non-unit size in broadcastable dimension %i", i);
                Py_DECREF(data);
                Py_DECREF(py_data);
                Py_DECREF(broadcastable);
                return NULL;
            }
        }
        if (storage && CudaNdarray_Check(storage))
        {
            rval = (CudaNdarray*) storage;
            Py_INCREF(rval);
        }
        else
        {
            rval = (CudaNdarray*) CudaNdarray_New();
        }
        if (rval)
        {
            if (CudaNdarray_CopyFromArray(rval, data))
            {
                Py_DECREF(rval);
                rval = NULL;
            }
        }
        Py_DECREF(data);
        Py_DECREF(py_data);
        Py_DECREF(broadcastable);
        return (PyObject*)rval;
    }
}

//TODO-- CudaNdarray_Dot and CudaNdarray_active_device_name are following different capitalization conventions.
//       Pick one and standardize it, this file is already annoying enough to grep through
static PyMethodDef module_methods[] = {
    {"dimshuffle", CudaNdarray_Dimshuffle, METH_VARARGS, "Returns the dimshuffle of a CudaNdarray."},
    {"dot", CudaNdarray_Dot, METH_VARARGS, "Returns the matrix product of two CudaNdarray arguments."},
    {"gpu_init", CudaNdarray_gpu_init, METH_VARARGS, "Select the gpu card to use; also usable to test whether CUDA is available."},
    {"active_device_name", CudaNdarray_active_device_name, METH_VARARGS, "Get the name of the active device."},
    {"active_device_number", CudaNdarray_active_device_number, METH_VARARGS, "Get the number of the active device."},
    {"gpu_shutdown", CudaNdarray_gpu_shutdown, METH_VARARGS, "Shut down the gpu."},
    {"device_properties", GetDeviceProperties, METH_VARARGS, "Return a dictionary with the device properties."},
    {"mem_info", GetDeviceMemInfo, METH_NOARGS, "Return a tuple with the free and total memory on the gpu in bytes."},
#if COMPUTE_GPU_MEM_USED
    {"theano_allocated", GetTheanoAllocInfo, METH_NOARGS, "Return the size in bytes of memory Theano currently have allocated on the gpu."},
#endif
    {"ptr_int_size", CudaNdarray_ptr_int_size, METH_VARARGS, "Return a tuple with the size of gpu pointer, cpu pointer and int in bytes."},
    {"filter", filter, METH_VARARGS, "filter(obj, broadcastable, strict, storage) returns a CudaNdarray initialized to obj if it matches the constraints of broadcastable.  strict=True prevents any numeric casting. If storage is a CudaNdarray it may be overwritten and used as the return value."},
    {"outstanding_mallocs", outstanding_mallocs, METH_VARARGS, "how many more mallocs have been called than free's"},
    {"from_gpu_pointer", CudaNdarray_from_gpu_pointer, METH_VARARGS, "Used to create a CudaNdarray from already allocated memory on the gpu.(example by pycuda)"},
    {NULL, NULL, NULL, NULL}  /* Sentinel */
};

#ifndef PyMODINIT_FUNC  /* declarations for DLL import/export */
#define PyMODINIT_FUNC void
#endif
PyMODINIT_FUNC
initcuda_ndarray(void)
{
    import_array();

    PyObject* m;

    if (PyType_Ready(&CudaNdarrayType) < 0)
        return;

    m = Py_InitModule3("cuda_ndarray", module_methods,
                       "Example module that creates an extension type.");

    if (m == NULL)
        return;

    Py_INCREF(&CudaNdarrayType);
    PyModule_AddObject(m, "CudaNdarray", (PyObject *)&CudaNdarrayType);
#if COMPUTE_GPU_MEM_USED
    for(int i=0;i<TABLE_SIZE;i++){
        _alloc_size_table[i].ptr=NULL;
        _alloc_size_table[i].size=0;
    }
#endif
    //    cublasInit();
    //if (0&&HIPBLAS_STATUS_SUCCESS != cublasGetError())
    //{
        //std::cerr << "WARNING: initcuda_ndarray: error initializing device\n";
    //}
    if (0) //TODO: is this necessary?
    {
        int deviceId = 0; // TODO: what number goes here?
        hipSetDevice(deviceId);
        hipError_t err = hipGetLastError();
        if( hipSuccess != err)
        {
            std::cerr << "Error in SetDevice:" << hipGetErrorString(err) << "\n";
        }
    }
}


//////////////////////////////////////
//
// C API FOR CudaNdarray
//
//////////////////////////////////////

int
CudaNdarray_Check(const PyObject * ob)
{
    //TODO: doesn't work with inheritance
    return CudaNdarray_CheckExact(ob);
}
int
CudaNdarray_CheckExact(const PyObject * ob)
{
    return ((ob->ob_type == &CudaNdarrayType) ? 1 : 0);
}

PyObject *
CudaNdarray_New(int nd)
{
    CudaNdarray *self = (CudaNdarray *)CudaNdarrayType.tp_alloc(&CudaNdarrayType, 0);
    if (self == NULL)
    {
        PyErr_SetString(PyExc_RuntimeError, "CudaNdarray_New failed to allocate self");
        return NULL;
    }
    CudaNdarray_null_init(self);

    if (nd == 0)
    {
        self->nd = 0;
    }
    else if (nd > 0)
    {
        if (CudaNdarray_set_nd(self, nd))
        {
            Py_DECREF(self);
            return NULL;
        }
    }
    ++_outstanding_mallocs[1];
    return (PyObject *)self;
}



//////////////////////////////
//
// Published helper functions
//
//////////////////////////////

int
cublas_init()
{
    cublasInit();
    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "error initializing device");
        return -1;
    }
    return 0;
}
int
cublas_shutdown()
{
    cublasShutdown();
    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "error shutting down device");
        return -1;
    }
    return 0;
}

int
CudaNdarray_CopyFromArray(CudaNdarray * self, PyArrayObject*obj)
{
    int err = CudaNdarray_alloc_contiguous(self, obj->nd, obj->dimensions);
    if (err) {
        return err;
    }

    int typenum = PyArray_TYPE(obj);
    if (typenum != REAL_TYPENUM)
    {
        PyErr_SetString(PyExc_TypeError, "can only copy from float arrays");
        return -1;
    }
    assert( 4 ==  PyArray_ITEMSIZE(obj));
    PyObject * py_src = PyArray_ContiguousFromAny((PyObject*)obj, typenum, self->nd, self->nd);
    if (!py_src) {
        return -1;
    }
    hipblasSetVector(PyArray_SIZE(py_src),
            sizeof(real),
            PyArray_DATA(py_src), 1,
            self->devdata, 1);
    CNDA_THREAD_SYNC;
    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "error copying data to device memory");
        Py_DECREF(py_src);
        return -1;
    }
    Py_DECREF(py_src);
    return 0;
}

bool
CudaNdarray_is_c_contiguous(const CudaNdarray * self)
{
    bool c_contiguous = true;
    int size = 1;
    for (int i = self->nd-1; (i >= 0) && c_contiguous; --i)
    {
        if (CudaNdarray_HOST_DIMS(self)[i] == 1)
            continue;
        if (CudaNdarray_HOST_STRIDES(self)[i] != size)
        {
            c_contiguous = false;
        }
        size = size * CudaNdarray_HOST_DIMS(self)[i];
    }
    return c_contiguous;
}

PyObject *
CudaNdarray_new_nd(int nd)
{
    CudaNdarray * rval = (CudaNdarray*) CudaNdarray_New();
    if (!rval || CudaNdarray_set_nd(rval, nd))
    {
        Py_XDECREF(rval);
        rval = NULL;
    }
    return (PyObject *) rval;
}


/**
 * Initialize 'self' as a view of 'base', with memory storage 'data'
 */

int CudaNdarray_set_device_data(CudaNdarray * self, float * data, PyObject * base)
{
    if (self->data_allocated)
    {
        assert(self->devdata);
        if (device_free(self->devdata))
        {
            self->devdata = NULL;
            self->data_allocated = 0;
            return -1;
        }
    }
    // Get the original base object (base.base.base...)
    PyObject * orig_base = base;
    // base is not always a CudaNdarray. It can be a GpuArray from pycuda, ...
    while (orig_base && CudaNdarray_Check(orig_base) && ((CudaNdarray*) orig_base)->base)
    {
        // base_base is itself a view
        orig_base = ((CudaNdarray*) orig_base)->base;
    }
    //N.B. XDECREF and XINCREF are no-ops for NULL pointers
    if (self->base != orig_base)
    {
        Py_XDECREF(self->base);
        self->base = orig_base;
        Py_XINCREF(self->base);
    }
    self->data_allocated = 0;
    self->devdata = data;
    return 0;
}

static __global__ void k_copy_1d(const int N, const float * x, const int sx, float * y, const int sy)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x*blockDim.x)
    {
        y[i*sy] = x[i*sx];
    }
}

//copy from other into self
int CudaNdarray_CopyFromCudaNdarray(CudaNdarray * self, const CudaNdarray * other, bool unbroadcast)
{
    int verbose = 0;
    if (verbose>1) fprintf(stderr, "CudaNdarray_CopyFromCudaNdarray\n");

    //standard elemwise size checks
    if (self->nd == -1)
    {
        PyErr_SetString(PyExc_TypeError, "can't copy into un-initialized CudaNdarray");
        return -1;
    }
    if (self->nd != other->nd)
    {
        PyErr_Format(PyExc_NotImplementedError, "CudaNdarray_CopyFromCudaNdarray: need same number of dims. destination nd=%d, source nd=%d. No broadcasting implemented.", self->nd, other->nd);
        return -1;
    }
    //standard elemwise dim checks (also compute total size)
    unsigned int size = 1;
    unsigned int size_source = 1;
    for (int i = 0; i< self->nd; ++i)
    {
        if ((CudaNdarray_HOST_DIMS(self)[i] != CudaNdarray_HOST_DIMS(other)[i])
            && (1!=CudaNdarray_HOST_DIMS(other)[i] || !unbroadcast) )
        {
          PyErr_Format(PyExc_ValueError, "need same dimensions for dim %d, destination=%d, source=%d",
                       i, CudaNdarray_HOST_DIMS(self)[i], CudaNdarray_HOST_DIMS(other)[i]);
            return -1;
        }
        size *= (unsigned int) CudaNdarray_HOST_DIMS(self)[i];
        size_source *= (unsigned int) CudaNdarray_HOST_DIMS(other)[i];
    }
    if (0 == size)
    {
        return 0; //nothing to copy, we're done.
    }
    if (CudaNdarray_is_c_contiguous(self) && CudaNdarray_is_c_contiguous(other) && size == size_source)
    {
        hipblasScopy(size, CudaNdarray_DEV_DATA(other), 1, CudaNdarray_DEV_DATA(self), 1);
        CNDA_THREAD_SYNC;
        if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
        {
            PyErr_SetString(PyExc_RuntimeError, "Error copying memory");
            return -1;
        }
        return 0;
    }
    //TODO: rewrite these copy operations to be more efficient
    //      See, for example the transpose example in the cuda_sdk.
    switch (self->nd)
    {
        case 0: // scalar
            {
                // THIS CASE SHOULD NEVER HAPPEN BECAUSE SCALARS ARE ALWAYS C CONTIGUOUS
                assert(0);
            }; break;
        case 1: // vector
            {
                if (verbose) fprintf(stderr, "Copying non-contiguous vector\n");
                if (verbose) fprint_CudaNdarray(stderr, other);
                unsigned int n_blocks = std::min(size, (unsigned int)NUM_VECTOR_OP_BLOCKS);
                unsigned int n_threads = std::min(ceil_intdiv(size, n_blocks), (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
                k_copy_1d<<<n_blocks, n_threads>>>(size,
                        CudaNdarray_DEV_DATA(other), CudaNdarray_HOST_STRIDES(other)[0],
                        CudaNdarray_DEV_DATA(self), CudaNdarray_HOST_STRIDES(self)[0]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err)
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s. (n_blocks=%i, n_threads_per_block=%i)\n", "k_copy_1d", hipGetErrorString(err), n_blocks, n_threads);
                    return -1;
                }
            }; break;
        default:
            {
                assert (hipSuccess == hipGetLastError());
                if (verbose) fprintf(stderr, "Copying with default version unbroadcast=%d\n", unbroadcast);
                // call worker routine
                unsigned int n_blocks = std::min(size, (unsigned int)NUM_VECTOR_OP_BLOCKS);
                unsigned int threads_per_block = std::min(ceil_intdiv(size, n_blocks), (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
                const CudaNdarray * cuda_dims = other;
                if(unbroadcast)
                    cuda_dims = self;
                //copy from other into self
                k_elemwise_unary_rowmajor_copy<<<n_blocks, threads_per_block>>>(
                        size,
                        (unsigned int)other->nd,
                        (const int *)CudaNdarray_DEV_DIMS(cuda_dims),
                        (const float*)CudaNdarray_DEV_DATA(other), (const int *)CudaNdarray_DEV_STRIDES(other),
                        CudaNdarray_DEV_DATA(self),  (const int *)CudaNdarray_DEV_STRIDES(self));
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err)
                {
                    //fprint_CudaNdarray(stderr, self);
                    //fprint_CudaNdarray(stderr, other);
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s. (n_blocks=%i, n_threads_per_block=%i)\n", "k_elemwise_unary_rowmajor_copy", hipGetErrorString(err), n_blocks, threads_per_block);
                    return -1;
                }
            }
    };
    return 0;
}

int CudaNdarray_gemm(float alpha, const CudaNdarray * A, const CudaNdarray * B, float beta, CudaNdarray * C)
{
    if (A->nd != 2)
    {
        PyErr_SetString(PyExc_ValueError, "non-matrix arg A to gemm");
        return -1;
    }
    if (B->nd != 2)
    {
        PyErr_SetString(PyExc_ValueError, "non-matrix arg B to gemm");
        return -1;
    }
    if (C->nd != 2)
    {
        PyErr_SetString(PyExc_ValueError, "non-matrix arg C to gemm");
        return -1;
    }

    // We must allow dimensions to be zeros.
    if ((CudaNdarray_HOST_DIMS(A)[1] != CudaNdarray_HOST_DIMS(B)[0])
            || (CudaNdarray_HOST_DIMS(A)[0] != CudaNdarray_HOST_DIMS(C)[0])
            || (CudaNdarray_HOST_DIMS(B)[1] != CudaNdarray_HOST_DIMS(C)[1]))
    {
        PyErr_Format(PyExc_ValueError, "dimension mismatch in args to gemm (%i,%i)x(%i,%i)->(%i,%i)",
                CudaNdarray_HOST_DIMS(A)[0],
                CudaNdarray_HOST_DIMS(A)[1],
                CudaNdarray_HOST_DIMS(B)[0],
                CudaNdarray_HOST_DIMS(B)[1],
                CudaNdarray_HOST_DIMS(C)[0],
                CudaNdarray_HOST_DIMS(C)[1]);
        return -1;
    }

    // If matrix A or B has non-unit size and non-unit stride in both
    // dimensions, we can make a copy.
    if (((CudaNdarray_HOST_DIMS(A)[0] > 1)
         && (CudaNdarray_HOST_STRIDES(A)[0] != 1)
         && (CudaNdarray_HOST_DIMS(A)[1] > 1)
         && (CudaNdarray_HOST_STRIDES(A)[1] != 1))
        || (CudaNdarray_HOST_STRIDES(A)[0] < 0)
        || (CudaNdarray_HOST_STRIDES(A)[1] < 0))
    {
        const CudaNdarray* A_new = (CudaNdarray*) CudaNdarray_Copy(A);
        if (!A_new)
            return -1;
        A = A_new;
    }
    else
    {
        // In the case above, we will need to decref A_new at the end.
        // To make things simpler, we incref A here, so we can always
        // decref A.
        Py_INCREF(A);
    }

    if (((CudaNdarray_HOST_DIMS(B)[0] > 1)
         && (CudaNdarray_HOST_STRIDES(B)[0] != 1)
         && (CudaNdarray_HOST_DIMS(B)[1] > 1)
         && (CudaNdarray_HOST_STRIDES(B)[1] != 1))
        || (CudaNdarray_HOST_STRIDES(B)[0] < 0)
        || (CudaNdarray_HOST_STRIDES(B)[1] < 0))
    {
        const CudaNdarray* B_new = (CudaNdarray*) CudaNdarray_Copy(B);
        if (!B_new)
        {
            Py_XDECREF(A);
            return -1;
        }
        B = B_new;
    }
    else
    {
        // In the case above, we will need to decref B_new at the end.
        // To make things simpler, we incref B here, so we can always
        // decref B.
        Py_INCREF(B);
    }

    // If matrix C has non-unit size and non-unit stride in both
    // dimensions, or negative strides, we can't operate. We cannot copy
    // C either, because the calling code will expect the result to be
    // in the original C container.
    if (((CudaNdarray_HOST_DIMS(C)[0] > 1)
         && (CudaNdarray_HOST_STRIDES(C)[0] != 1)
         && (CudaNdarray_HOST_DIMS(C)[1] > 1)
         && (CudaNdarray_HOST_STRIDES(C)[1] != 1))
        || (CudaNdarray_HOST_STRIDES(C)[0] < 0)
        || (CudaNdarray_HOST_STRIDES(C)[1] < 0))
    {
        PyErr_Format(PyExc_AssertionError,
                     "non-unit or negative stride in gemm arg C (%i,%i) of shape (%i,%i)",
                     CudaNdarray_HOST_STRIDES(C)[0],
                     CudaNdarray_HOST_STRIDES(C)[1],
                     CudaNdarray_HOST_DIMS(C)[0],
                     CudaNdarray_HOST_DIMS(C)[1]);
        Py_XDECREF(A);
        Py_XDECREF(B);
        return -1;
    }

    // the unit integer is divided logically into three fields of 4 bits
    // the lowermost 4 bits encode the stride pattern of the output
    // the next higher 4 bits encode the B variable (or y)
    // the next higher 4 bits encode the C variable (or x)
    //
    // the stride pattern for each input is encoded as 0 for unit stride from col to col (Row major)
    //                                                 1 for unit stride from row to row (Col major)

    // a stride of 0 implies a dimension of 1 - so we can actually define
    // a stride of 0 as a 'unit' stride because gemm will never use it.
    // If a dimension is 0, its stride will not be used either, so we can
    // consider it a 'unit' stride too.
    int unit = 0;
    if (CudaNdarray_HOST_STRIDES(A)[1] == 1 || CudaNdarray_HOST_DIMS(A)[1] <= 1) {
        unit |= (0x0 << 8);
    } else if (CudaNdarray_HOST_STRIDES(A)[0] == 1 || CudaNdarray_HOST_DIMS(A)[0] <= 1) {
        unit |= (0x1 << 8);
    } else {
        unit |= (0x2 << 8);
    }
    if (CudaNdarray_HOST_STRIDES(B)[1] == 1 || CudaNdarray_HOST_DIMS(B)[1] <= 1) {
        unit |= (0x0 << 4);
    } else if (CudaNdarray_HOST_STRIDES(B)[0] == 1 || CudaNdarray_HOST_DIMS(B)[0] <= 1) {
        unit |= (0x1 << 4);
    } else {
        unit |= (0x2 << 4);
    }
    if (CudaNdarray_HOST_STRIDES(C)[1] == 1 || CudaNdarray_HOST_DIMS(C)[1] <= 1) {
        unit |= (0x0 << 0);
    } else if (CudaNdarray_HOST_STRIDES(C)[0] == 1 || CudaNdarray_HOST_DIMS(C)[0] <= 1) {
        unit |= (0x1 << 0);
    } else {
        unit |= (0x2 << 0);
    }

    /* create appropriate strides for malformed matrices that are row or column
     * vectors
     */
    int sa_0 = (CudaNdarray_HOST_DIMS(A)[0] > 1) ? CudaNdarray_HOST_STRIDES(A)[0] : CudaNdarray_HOST_DIMS(A)[1];
    int sa_1 = (CudaNdarray_HOST_DIMS(A)[1] > 1) ? CudaNdarray_HOST_STRIDES(A)[1] : CudaNdarray_HOST_DIMS(A)[0];
    int sb_0 = (CudaNdarray_HOST_DIMS(B)[0] > 1) ? CudaNdarray_HOST_STRIDES(B)[0] : CudaNdarray_HOST_DIMS(B)[1];
    int sb_1 = (CudaNdarray_HOST_DIMS(B)[1] > 1) ? CudaNdarray_HOST_STRIDES(B)[1] : CudaNdarray_HOST_DIMS(B)[0];
    int sc_0 = (CudaNdarray_HOST_DIMS(C)[0] > 1) ? CudaNdarray_HOST_STRIDES(C)[0] : CudaNdarray_HOST_DIMS(C)[1];
    int sc_1 = (CudaNdarray_HOST_DIMS(C)[1] > 1) ? CudaNdarray_HOST_STRIDES(C)[1] : CudaNdarray_HOST_DIMS(C)[0];

    float* a = CudaNdarray_DEV_DATA(A);
    float* b = CudaNdarray_DEV_DATA(B);
    float* c = CudaNdarray_DEV_DATA(C);
    char N = 'N';
    char T = 'T';
    //std::cerr << (unit/256) MOD 16 << (unit / 16) MOD 16 << unit MOD 16<< '\\n';
    // There should be no negative stride at that point
#define CHK_STRIDE_SGEMM(T0, T1, D0, D1, D2, a, x, sx, y, sy, b, z, sz) \
    if (sx == 0){sx = 1;}\
    if (sy == 0){sy = 1;}\
    if (sz == 0){sz = 1;}\
    if ((sx > 0) && (sy > 0) && (sz > 0)) { \
        hipblasSgemm(T0, T1, D0, D1, D2, a, x, sx, y, sy, b, z, sz); \
    } else { \
        PyErr_SetString(PyExc_AssertionError, "negative stride to sGemm");\
        Py_XDECREF(A);\
        Py_XDECREF(B);\
        return -1; \
    }

    switch(unit)
    {
        case 0x000: CHK_STRIDE_SGEMM(N, N, CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(A)[1], alpha, b, sb_0, a, sa_0, beta, c, sc_0); break;
        case 0x100: CHK_STRIDE_SGEMM(N, T, CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(A)[1], alpha, b, sb_0, a, sa_1, beta, c, sc_0); break;
        case 0x010: CHK_STRIDE_SGEMM(T, N, CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(A)[1], alpha, b, sb_1, a, sa_0, beta, c, sc_0); break;
        case 0x110: CHK_STRIDE_SGEMM(T, T, CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(A)[1], alpha, b, sb_1, a, sa_1, beta, c, sc_0); break;
        case 0x001: CHK_STRIDE_SGEMM(T, T, CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(A)[1], alpha, a, sa_0, b, sb_0, beta, c, sc_1); break;
        case 0x101: CHK_STRIDE_SGEMM(N, T, CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(A)[1], alpha, a, sa_1, b, sb_0, beta, c, sc_1); break;
        case 0x011: CHK_STRIDE_SGEMM(T, N, CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(A)[1], alpha, a, sa_0, b, sb_1, beta, c, sc_1); break;
        case 0x111: CHK_STRIDE_SGEMM(N, N, CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(A)[1], alpha, a, sa_1, b, sb_1, beta, c, sc_1); break;
        default: PyErr_Format(PyExc_ValueError, "some matrix has no unit stride (unit=%x)", unit);
                 return -1;
    };
    CNDA_THREAD_SYNC;
    Py_XDECREF(A);
    Py_XDECREF(B);

    hipblasStatus_t err = cublasGetError();
    if (HIPBLAS_STATUS_SUCCESS != err)
    {
        PyErr_Format(PyExc_RuntimeError,
                     "hipblasSgemm failed (%i)",
                     err);
        return -1;
    }
    return 0;
}

int CudaNdarray_sgemv(float alpha, const CudaNdarray * A, const CudaNdarray * B, float beta, CudaNdarray * C)
{
    /**
    * C <- alpha A B + beta C
    *    A : matrix
    *    B, C: vector
    *    alpha, beta: scalars
    */
    if (A->nd != 2) { PyErr_SetString(PyExc_ValueError, "non-matrix arg to gemv"); return -1; }
    if (B->nd != 1) { PyErr_SetString(PyExc_ValueError, "non-vector arg to gemv"); return -1; }
    if (C->nd != 1) { PyErr_SetString(PyExc_ValueError, "non-vector arg to gemv"); return -1; }

    // We must allow dimensions to be zeros.
    if ((CudaNdarray_HOST_DIMS(A)[1] != CudaNdarray_HOST_DIMS(B)[0])
            || (CudaNdarray_HOST_DIMS(A)[0] != CudaNdarray_HOST_DIMS(C)[0]))
    {
        PyErr_Format(PyExc_ValueError, "dimension mismatch in args to gemv (%i,%i)x(%i)->(%i)",
                CudaNdarray_HOST_DIMS(A)[0],
                CudaNdarray_HOST_DIMS(A)[1],
                CudaNdarray_HOST_DIMS(B)[0],
                CudaNdarray_HOST_DIMS(C)[0]);
        return -1;
    }

    // If matrix A has non-unit size and non-unit stride in both
    // dimensions, or negative strides, we cannot operate, but we can
    // make a copy.
    if (((CudaNdarray_HOST_DIMS(A)[0] > 1)
         && (CudaNdarray_HOST_STRIDES(A)[0] != 1)
         && (CudaNdarray_HOST_DIMS(A)[1] > 1)
         && (CudaNdarray_HOST_STRIDES(A)[1] != 1))
        || (CudaNdarray_HOST_STRIDES(A)[0] < 0)
        || (CudaNdarray_HOST_STRIDES(A)[1] < 0))
    {
        const CudaNdarray* A_new = (CudaNdarray*) CudaNdarray_Copy(A);
        if (!A_new)
            return -1;
        A = A_new;
    }
    else
    {
        // Incref A, so we can decref it at the end in all cases
        Py_INCREF(A);
    }

    // If vector B as a negative stride, we also have to make a copy.
    if (CudaNdarray_HOST_STRIDES(B)[0] < 0)
    {
        const CudaNdarray* B_new = (CudaNdarray*) CudaNdarray_Copy(B);
        if (!B_new)
        {
            Py_XDECREF(A);
            return -1;
        }
        B = B_new;
    }
    else
    {
        // Incref B, so we can decref it at the end in all cases
        Py_INCREF(B);
    }

    // cudablas does not handle negative strides as expected
    if (   (CudaNdarray_HOST_STRIDES(A)[0] < 0)
        || (CudaNdarray_HOST_STRIDES(A)[1] < 0))
    {
        PyErr_Format(PyExc_ValueError, "illegal strides in args to gemv (%i,%i)",
                CudaNdarray_HOST_STRIDES(A)[0],
                CudaNdarray_HOST_STRIDES(A)[1]);
        Py_XDECREF(A);
        Py_XDECREF(B);
        return -1;
    }

    /* create appropriate strides for malformed matrices that are row or column
     * vectors
     */
    int sa_0 = (CudaNdarray_HOST_DIMS(A)[0] > 1) ? CudaNdarray_HOST_STRIDES(A)[0] : CudaNdarray_HOST_DIMS(A)[1];
    int sa_1 = (CudaNdarray_HOST_DIMS(A)[1] > 1) ? CudaNdarray_HOST_STRIDES(A)[1] : CudaNdarray_HOST_DIMS(A)[0];
    int sb_0 = (CudaNdarray_HOST_DIMS(B)[0] > 1) ? CudaNdarray_HOST_STRIDES(B)[0] : 1;
    int sc_0 = (CudaNdarray_HOST_DIMS(C)[0] > 1) ? CudaNdarray_HOST_STRIDES(C)[0] : 1;

    if (sa_0 == 0)
        sa_0 = 1;
    if (sa_1 == 0)
        sa_1 = 1;

    if (CudaNdarray_SIZE(C)) {
        if ((CudaNdarray_HOST_DIMS(A)[0] <= 1)
            || ((CudaNdarray_HOST_STRIDES(A)[0] == 1)
                && (CudaNdarray_HOST_STRIDES(A)[1] > 0)))
        {
            hipblasSgemv('N',
                    CudaNdarray_HOST_DIMS(A)[0], CudaNdarray_HOST_DIMS(A)[1],
                    alpha,
                    CudaNdarray_DEV_DATA(A), sa_1,
                    CudaNdarray_DEV_DATA(B), sb_0,
                    beta,
                    CudaNdarray_DEV_DATA(C), sc_0);
        }
        else if ((CudaNdarray_HOST_DIMS(A)[1] <= 1)
                || ((CudaNdarray_HOST_STRIDES(A)[1] == 1)
                    && (CudaNdarray_HOST_STRIDES(A)[0] > 0)))
        {
            hipblasSgemv('T',
                    CudaNdarray_HOST_DIMS(A)[1], CudaNdarray_HOST_DIMS(A)[0],
                    alpha,
                    CudaNdarray_DEV_DATA(A), sa_0,
                    CudaNdarray_DEV_DATA(B), sb_0,
                    beta,
                    CudaNdarray_DEV_DATA(C), sc_0);
        }
        else
        {
            PyErr_Format(PyExc_AssertionError,
                         "Unexpected stride pattern in gemv: (%i, %i) x %i -> %i.\n"
                         "Shapes are: (%i, %i) x %i -> %i\n",
                         CudaNdarray_HOST_STRIDES(A)[0],
                         CudaNdarray_HOST_STRIDES(A)[1],
                         CudaNdarray_HOST_STRIDES(B)[0],
                         CudaNdarray_HOST_STRIDES(C)[0],
                         CudaNdarray_HOST_DIMS(A)[0],
                         CudaNdarray_HOST_DIMS(A)[1],
                         CudaNdarray_HOST_DIMS(B)[0],
                         CudaNdarray_HOST_DIMS(C)[0]);
            Py_XDECREF(A);
            Py_XDECREF(B);
            return -1;
        }
    }

    CNDA_THREAD_SYNC;
    Py_XDECREF(A);
    Py_XDECREF(B);

    hipblasStatus_t err = cublasGetError();
    if (HIPBLAS_STATUS_SUCCESS != err)
    {
        PyErr_Format(PyExc_RuntimeError,
                     "hipblasSgemv failed (%i)",
                     err);
        return -1;
    }
    return 0;
}

int CudaNdarray_sger(float alpha, const CudaNdarray * x, const CudaNdarray * y, CudaNdarray * A) {
    if (x->nd != 1) { PyErr_SetString(PyExc_ValueError, "non-vector arg x to sger"); return -1; }
    if (y->nd != 1) { PyErr_SetString(PyExc_ValueError, "non-vector arg y to sger"); return -1; }
    if (A->nd != 2) { PyErr_SetString(PyExc_ValueError, "non-matrix arg A to sger"); return -1; }

    if ((CudaNdarray_HOST_DIMS(A)[0] != CudaNdarray_HOST_DIMS(x)[0])
        || (CudaNdarray_HOST_DIMS(A)[1] != CudaNdarray_HOST_DIMS(y)[0])) {
        PyErr_Format(PyExc_ValueError,
                     "dimension mismatch in args to sger (%i)x(%i)->(%i,%i)",
                     CudaNdarray_HOST_DIMS(x)[0],
                     CudaNdarray_HOST_DIMS(y)[0],
                     CudaNdarray_HOST_DIMS(A)[0],
                     CudaNdarray_HOST_DIMS(A)[1]);
        return -1;
    }

    int x_strides = CudaNdarray_HOST_STRIDES(x)[0];
    const CudaNdarray * x_ = x;
    if(x_strides == 0){
        if(CudaNdarray_HOST_DIMS(x)[0] != 1){
            PyErr_Format(PyExc_RuntimeError,
                         "CudaNdarray_sger: Invalid input x(should not happen)."
                         " We received an CudaNdarray vector with a stride of 0"
                         " that have more then 1 elements!");
            return -1;
        }
        x_strides = 1;
    } else if(x_strides < 0){
        x_ = (CudaNdarray*)CudaNdarray_Copy(x);
        x_strides = CudaNdarray_HOST_STRIDES(x_)[0];
    }

    int y_strides = CudaNdarray_HOST_STRIDES(y)[0];
    const CudaNdarray * y_ = y;
    if(y_strides == 0){
        if(CudaNdarray_HOST_DIMS(y)[0] != 1){
            PyErr_Format(PyExc_RuntimeError,
                         "CudaNdarray_sger: Invalid input y(should not happen)."
                         " We received an CudaNdarray vector with a stride of 0"
                         " that have more then 1 elements!");
            return -1;
        }
        y_strides = 1;
    } else if(y_strides < 0){
        y_ = (CudaNdarray*)CudaNdarray_Copy(y);
        y_strides = CudaNdarray_HOST_STRIDES(y_)[0];
    }

    // Create appropriate strides if A is a row or column vector
    int sa_0 = (CudaNdarray_HOST_DIMS(A)[0] > 1) ? CudaNdarray_HOST_STRIDES(A)[0]
                                                 : CudaNdarray_HOST_DIMS(A)[1];
    int sa_1 = (CudaNdarray_HOST_DIMS(A)[1] > 1) ? CudaNdarray_HOST_STRIDES(A)[1]
                                                 : CudaNdarray_HOST_DIMS(A)[0];

    if(CudaNdarray_SIZE(A)){
        // If A is in col-major
        if ((CudaNdarray_HOST_DIMS(A)[0] <= 1)
            || ((CudaNdarray_HOST_STRIDES(A)[0] == 1)
                && (CudaNdarray_HOST_STRIDES(A)[1] > 0)))
        {
            hipblasSger(CudaNdarray_HOST_DIMS(x)[0], CudaNdarray_HOST_DIMS(y)[0], alpha,
                       CudaNdarray_DEV_DATA(x_), x_strides,
                       CudaNdarray_DEV_DATA(y_), y_strides,
                       CudaNdarray_DEV_DATA(A), sa_1);
        }
        // Since Sger expects A in col-major, we invert x and y to fake this.
        else if ((CudaNdarray_HOST_DIMS(A)[1] <= 1)
                || ((CudaNdarray_HOST_STRIDES(A)[1] == 1)
                    && (CudaNdarray_HOST_STRIDES(A)[0] > 0)))
        {
            hipblasSger(CudaNdarray_HOST_DIMS(y)[0], CudaNdarray_HOST_DIMS(x)[0], alpha,
                       CudaNdarray_DEV_DATA(y_), y_strides,
                       CudaNdarray_DEV_DATA(x_), x_strides,
                       CudaNdarray_DEV_DATA(A), sa_0);
        }
        // A has to be either c- or f-contiguous, with no negative strides
        else
        {
            PyErr_SetString(PyExc_NotImplementedError,
                            "non-contiguous A, or negative strides, in sger");
            return -1;
        }
    }
    CNDA_THREAD_SYNC;
    if(x_ != x)
        Py_DECREF(x_);
    if(y_ != y)
        Py_DECREF(y_);

    hipblasStatus_t err = cublasGetError();
    if (HIPBLAS_STATUS_SUCCESS != err)
    {
        PyErr_Format(PyExc_RuntimeError,
                     "hipblasSger failed (%i)",
                     err);
        return -1;
    }

    return 0;
}

/**
 *
 * Precondition:
 *  a->dim[d] == (dims_a[d]==0) ? (1 << log2_dims_a[d]) : dims_a[d]
 *  z->dim[d] == (z_str[d]==0) ? 1 : dims_a[d];
 *
 *  TODO: templatize this function to support other reductions.
 *  All that needs to change is the initial value for sum, and the reduction operator.
 */

static __global__ void kernel_reduce_sum(const unsigned int size_z,
        const unsigned int nd,
        const int * dims_a,
        const int * log2_dims_a,
        const int * a_str,
        const float * a_data,
        const int * z_str,
        float * z_data)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    //structure data contains the strides and dimensions of both a and z
    // a_dim[0], a_dim[1], ... a_dim[nd-1],
    // a_log2dim[0], a_log2dim[1], ... a_log2dim[nd-1],
    // a_str[0], ... a_str[nd-1],
    // z_str[0], ... z_str[nd-1]
    extern __shared__ int structure_data[];
    for (unsigned int i = threadIdx.x; i < nd; i += blockDim.x)
    {
        structure_data[i+0*nd] = dims_a[i];
        structure_data[i+1*nd] = log2_dims_a[i];
        structure_data[i+2*nd] = a_str[i];
        structure_data[i+3*nd] = z_str[i];
    }
    dims_a = structure_data;
    log2_dims_a = structure_data + nd;
    a_str = structure_data + 2*nd;
    z_str = structure_data + 3*nd;

    __syncthreads(); //wait for all the shared structure to be loaded

    for (unsigned int i = idx; i < size_z; i += numThreads)
    {
        unsigned int ii = i;
        const float * a_data_i = a_data;
        float * z_data_i = z_data;
        unsigned int n_reduce_elements = 1;
        unsigned int n_reduce_dims = 0;
        unsigned int reduce_dim0 = nd-1;


        //In this loop, we locate the initial element of the slice that we'd like to reduce with this thread
        //  At the same time, we [re]calculate the size of that slice (n_reduce_elements)
        for (unsigned int d = 0; d < nd; ++d)
        {
            if (a_str[d] && (!z_str[d])) // this means 'd' is a dimension we are reducing over
            {
                n_reduce_elements *= dims_a[d];
                n_reduce_dims += 1;
                reduce_dim0 = (d < reduce_dim0) ? d : reduce_dim0;
            }
            else //'d' is not a dimension that we are reducing over
            {
                unsigned int pos_d;
                if (log2_dims_a[d]==-1) //TODO: when things are working, use this switch
                {
                    // this branch is not preferred,
                    // because the manual said that integer mod and div operations are slow on gpu
                    pos_d = (ii % dims_a[d]);
                    ii = (ii / dims_a[d]);
                }
                else
                {
                    pos_d = (ii & ((1 << log2_dims_a[d])-1)); //take the lower log2_dims bits
                    ii = (ii >> log2_dims_a[d]);  //shift those lower log2_dims bits off of ii
                }
                a_data_i += pos_d * a_str[d];
                z_data_i += pos_d * z_str[d];
            }
        }
        // now we've got pointers a_data_i and z_data_i into element 0 of the slice over which we are reducing
        // do a similar loop

        float sum = 0.0f;
        switch(n_reduce_dims)
        {
            case 0:
                {
                    sum = a_data_i[0];
                }
                break;
            case 1:
                {
                    const int stride = a_str[reduce_dim0];
                    const float * a_data_i_max = a_data_i + dims_a[reduce_dim0] * stride;
                    while (a_data_i != a_data_i_max)
                    {
                        sum += a_data_i[0];
                        a_data_i += stride;
                    }
                }
                break;
            case 2:
                {
                    int rd = reduce_dim0+1;
                    for (; rd < nd; ++rd)
                    {
                        if (a_str[rd] && (!z_str[rd])) // this means 'rd' is a dimension we are reducing over
                            break;
                    }
                    const int stride0 = a_str[reduce_dim0];
                    const int stride1 = a_str[rd];
                    for (int ii = 0; ii < dims_a[rd]; ++ii)
                    {
                        const float * a_data_ri = a_data_i + ii * stride1;
                        const float * a_data_ri_max = a_data_ri + dims_a[reduce_dim0] * stride0;
                        while (a_data_ri != a_data_ri_max)
                        {
                            sum += a_data_ri[0];
                            a_data_ri += stride0;
                        }
                    }
                };
                break;
            default:
                {
                    for (unsigned int reduce_i = 0; reduce_i < n_reduce_elements; ++reduce_i)
                    {
                        //TODO: optimize this loop to work more like theano's Elemwise.  It's serial code.
                        unsigned int reduce_ii = reduce_i;
                        const float * a_data_ri = a_data_i;

                        //This loop finds the element in the a slice to add.
                        for (unsigned int rd = reduce_dim0; rd < nd; ++rd)
                        {
                            unsigned int pos_d;
                            if (a_str[rd] && (!z_str[rd])) // this means 'd' is a dimension we are reducing over
                            {
                                if (log2_dims_a[rd]==-1)
                                {
                                    // this branch is not preferred,
                                    // because the manual said that integer mod and div operations are slow on gpu
                                    pos_d = (reduce_ii % dims_a[rd]);
                                    reduce_ii = (reduce_ii / dims_a[rd]);
                                }
                                else
                                {
                                    pos_d = (reduce_ii & ((1 << log2_dims_a[rd])-1)); //take the lower log2_dims bits
                                    reduce_ii = (reduce_ii >> log2_dims_a[rd]);  //shift those lower log2_dims bits off of ii
                                }
                                a_data_ri += pos_d * a_str[rd];
                            }
                        }
                        sum += a_data_ri[0];
                    }
                }
        }
        z_data_i[0] = sum;
    }
}

static __global__ void kernel_reduce_sum_1011(
        const unsigned int d0,
        const unsigned int d1,
        const unsigned int d2,
        const unsigned int d3,
        const float *A, const int sA0, const int sA1, const int sA2, const int sA3,
        float * Z, const int sZ0)
{
    const int threadCount = blockDim.x * blockDim.y * blockDim.z;
    const int threadNum = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    extern __shared__ float buf[];
    float mysum = 0.0f;

    if (warpSize != 32)
    {
        return;  //TODO: set error code
    }

    for (int i0 = threadIdx.z; i0 < d0; i0 += blockDim.z)
    {
        float Ai = A[i0 * sA0 + blockIdx.x * sA1 + threadIdx.y * sA2 + threadIdx.x * sA3];
        mysum += Ai;
    }
    buf[threadNum] = mysum;
    __syncthreads();

    // rest of function is handled by one warp
    if (threadNum < warpSize)
    {
        for (int i = threadNum + warpSize; i < threadCount; i += warpSize)
        {
            mysum += buf[i];
        }
        buf[threadNum] = mysum;
        if (threadNum < 16)
        {
            //reduce so that threadNum 0 has the sum of everything
            if(threadNum + 16 < threadCount) buf[threadNum] += buf[threadNum+16];
            if(threadNum + 8 < threadCount) buf[threadNum] += buf[threadNum+8];
            if(threadNum + 4 < threadCount) buf[threadNum] += buf[threadNum+4];
            if(threadNum + 2 < threadCount) buf[threadNum] += buf[threadNum+2];
            if(threadNum + 1 < threadCount) buf[threadNum] += buf[threadNum+1];
            if (threadNum == 0)
            {
                Z[blockIdx.x*sZ0] = buf[0];
            }
        }
    }
}
/**
 * Dimensions in which the self has size 1 and A has size > 1 are considered summing dimensions
 * Dimensions in which self has size > 1 and A has size > 1 are considered non-summing dimensions, and in this case their sizes must be equal.
 */
int
CudaNdarray_reduce_sum(CudaNdarray * self, CudaNdarray * A)
{
    int verbose = 0;
    //check input rank
    if (self->nd != A->nd)
    {
        PyErr_Format(PyExc_TypeError, "Rank mismatch in CudaNdarray_sum: %i vs %i", self->nd, A->nd);
        return -1;
    }
    for (int i = 0; i < self->nd; ++i)
    {
        if ((CudaNdarray_HOST_DIMS(self)[i] > 1) && (CudaNdarray_HOST_DIMS(self)[i] != CudaNdarray_HOST_DIMS(A)[i]))
        {
            PyErr_Format(PyExc_TypeError, "Dimension mismatch in CudaNdarray_sum: self->dim[%i] == %i , A->dim[%i] = %i",
                    i, CudaNdarray_HOST_DIMS(self)[i], i, CudaNdarray_HOST_DIMS(A)[i]);
            return -1;
        }
    }

    int n_summations = (unsigned int)CudaNdarray_SIZE(self);
    if (verbose)
    {
        std::cerr << "reduce_sum n_summations " << n_summations  << '\n';
        std::cerr << "reduce_sum nd " << self->nd  << '\n';
        fprint_CudaNdarray(stderr, A);
        fprint_CudaNdarray(stderr, self);
    }
    if (0 && (A->nd == 4) //check to see if kernel_reduce_sum_1011 applies
            && (CudaNdarray_HOST_DIMS(self)[0] == 1)
            && (CudaNdarray_HOST_DIMS(self)[2] == 1)
            && (CudaNdarray_HOST_DIMS(self)[3] == 1)
       )
    {
        dim3 n_threads(CudaNdarray_HOST_DIMS(A)[3], CudaNdarray_HOST_DIMS(A)[2]);
        dim3 n_blocks(CudaNdarray_HOST_DIMS(A)[1]);
        while (n_threads.x * n_threads.y * n_threads.z < NUM_VECTOR_OP_THREADS_PER_BLOCK) ++n_threads.z;
        n_threads.z -= 1;
        if (n_threads.z > 64) n_threads.z = 64;
        if (n_threads.z)
        {
            if (verbose) printf("trying kernel_reduce_sum_1011\n");
            int n_shared = sizeof(float) * n_threads.x * n_threads.y * n_threads.z;
            kernel_reduce_sum_1011<<<n_blocks, n_threads, n_shared>>>(
                    CudaNdarray_HOST_DIMS(A)[0],
                    CudaNdarray_HOST_DIMS(A)[1],
                    CudaNdarray_HOST_DIMS(A)[2],
                    CudaNdarray_HOST_DIMS(A)[3],
                    CudaNdarray_DEV_DATA(A),
                    CudaNdarray_HOST_STRIDES(A)[0],
                    CudaNdarray_HOST_STRIDES(A)[1],
                    CudaNdarray_HOST_STRIDES(A)[2],
                    CudaNdarray_HOST_STRIDES(A)[3],
                    CudaNdarray_DEV_DATA(self),
                    CudaNdarray_HOST_STRIDES(self)[1]);
            CNDA_THREAD_SYNC;
            if (hipSuccess == hipGetLastError()) return 0;
            if (verbose) printf("failed, falling back to kernel_reduce_sum\n");
        }
    }

    int n_threads_per_block = std::min(n_summations,
            NUM_VECTOR_OP_THREADS_PER_BLOCK);
    int n_blocks = std::min(ceil_intdiv(n_summations,n_threads_per_block),
            NUM_VECTOR_OP_BLOCKS);
    int n_structure_cache = self->nd * 4 * sizeof(int);

    if (verbose)
    {
        std::cerr << "n_blocks, n_threads_per_block " << n_blocks << ' ' << n_threads_per_block  << '\n';
    }
    assert (self->nd > 0);
    assert (self->nd == A->nd);
    kernel_reduce_sum<<<n_blocks, n_threads_per_block, n_structure_cache>>>(
            n_summations,
            self->nd,
            CudaNdarray_DEV_DIMS(A),
            CudaNdarray_DEV_LOG2DIMS(A),
            CudaNdarray_DEV_STRIDES(A),
            CudaNdarray_DEV_DATA(A),
            CudaNdarray_DEV_STRIDES(self),
            CudaNdarray_DEV_DATA(self));
    CNDA_THREAD_SYNC;
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "kernel_reduce_sum", hipGetErrorString(err));
        return -1;
    }
    return 0;
}
int
CudaNdarray_reduce_prod(CudaNdarray * self, const CudaNdarray * A)
{
    PyErr_SetString(PyExc_NotImplementedError, "");
    return -1;
}
int
CudaNdarray_reduce_min(CudaNdarray * self, const CudaNdarray * A)
{
    PyErr_SetString(PyExc_NotImplementedError, "");
    return -1;
}
int
CudaNdarray_reduce_max(CudaNdarray * self, const CudaNdarray * A)
{
    PyErr_SetString(PyExc_NotImplementedError, "");
    return -1;
}


/**
 *
 *  pattern is a permutation of [0, 1, ... self->nd-1] with the following twists:
 *  - an element 'd' of the permutation can be dropped if CudaNdarray_HOST_DIMS(self)[d] == 1
 *  - any number of '-1' elements can be in the pattern, and they will cause new ranks (with dim==1) to be inserted.
 *
 *  For example, if CudaNdarray_HOST_DIMS(self) == [4, 5, 1, 6], and pattern = [0,3,-1,-1, 1], then CudaNdarray_HOST_DIMS(self) would be modified to become:
 *     [4, 6, 1, 1, 5] (we dropped the original dim[2]==1, and inserted two singleton dimensions with the -1s.
 */
int
CudaNdarray_dimshuffle(CudaNdarray * self, unsigned int len, const int * pattern)
{
    //TODO: pass a workspace pointer to avoid the internal malloc
    int * newdims = (int *)malloc(sizeof(int) * (len + len + self->nd)); //we tack on the taken buffer here for speed of not having to malloc twice.
    int * newstrides = newdims + len;
    int * dims_taken = newstrides + len;
    if (!newdims)
    {
        PyErr_SetString(PyExc_MemoryError, "CudaNdarray_dimshuffle: Failed to allocate temporary space");
        return -1;
    }
    for (int i = 0; i < self->nd; ++i)
    {
        dims_taken[i] = 0;
    }
    for (int i = 0; i < len; ++i)
    {
        if (pattern[i] < 0)
        {
            newdims[i] = 1;
            newstrides[i] = 0;
        }
        else if(dims_taken[pattern[i]])
        {
            PyErr_Format(PyExc_ValueError, "Cudandarray_dimshuffle: invalid pattern for Cudandarray_dimshuffle. You used the dimensions %d multiple time",
                         pattern[i]);
            free(newdims);
            return -1;
        }
        else if (pattern[i]>= self->nd)
        {
            PyErr_Format(PyExc_ValueError, "Cudandarray_dimshuffle: invalid pattern for Cudandarray_dimshuffle. You asked for a dimensions that don't exist %d for a %d dims CudaNdarray",
                         pattern[i], self->nd);
            free(newdims);
            return -1;
        }
        else
        {
            newdims[i] = CudaNdarray_HOST_DIMS(self)[pattern[i]];
            newstrides[i] = CudaNdarray_HOST_STRIDES(self)[pattern[i]];
            dims_taken[pattern[i]] = 1;
        }
    }
    //Check if we dropped not broadcastable dims
    for (int i = 0; i < self->nd; ++i)
    {
        if (dims_taken[i]==0 && CudaNdarray_HOST_DIMS(self)[i]!=1)
        {
            PyErr_SetString(PyExc_ValueError, "Cudandarray_dimshuffle: You cannot drop a non-broadcastable dimension.");
            free(newdims);
            return -1;
        }
    }
    //swap this structure in for the one in self, and sync to the card
    if (CudaNdarray_set_nd(self, len))
    {
        free(newdims);
        return -1;
    }
    for (int i = 0; i < len; ++i)
    {
        CudaNdarray_set_dim(self, i, newdims[i]);
        CudaNdarray_set_stride(self, i, newstrides[i]);
    }
    if (cnda_copy_structure_to_device(self))
    {
        free(newdims);
        return -1;
    }
    free(newdims);
    return 0;
}



/**
 *
 *  This is the function that bind to python.
 *  See CudaNdarray_dimshuffle to call from C.
 *  We use -1 to mean 'x' as in Tensor Dimshuffle.
 */
PyObject *
CudaNdarray_Dimshuffle(PyObject* _unused, PyObject* args)
{
    PyObject * self = NULL;
    PyObject * pattern_object = NULL;
    int * pattern = NULL;
    PyObject * rval = NULL;
    int success = -1;
    //const int * dims = NULL;

    //args should consist of two python objects ("OO")
    if (! PyArg_ParseTuple(args, "OO", &self, &pattern_object))
        return NULL;

    if (!CudaNdarray_Check(self) )
    {
        PyErr_SetString(PyExc_TypeError, "First argument to cuda_ndarray.dimshuffle must be a CudaNdarray");
        return NULL;
    }

    //parse pattern_object into int * pattern

    Py_ssize_t pattern_dim =  PyObject_Length(pattern_object);

    if (pattern_dim < 0)
    {
        PyErr_SetString(PyExc_TypeError, "Couldn't get length of third argument to cuda_ndarray.dimshuffle");
        return NULL;
    }

    pattern = (int *) malloc( pattern_dim * sizeof(int));

    for (Py_ssize_t i = 0; i < pattern_dim; i++)
    {
        PyObject * idx = PyLong_FromLong(i);

        if (idx == NULL)
        {
            PyErr_SetString(PyExc_Exception, "Couldn't make long object to loop over list/tuple");
            goto CudaNdarray_dimshuffle_fail;
        }

        long elem_value = 0;

        PyObject * elem = PyObject_GetItem(pattern_object, idx);

        if (elem == NULL)
        {
            Py_XDECREF( elem);
            PyErr_SetString(PyExc_ValueError, "Third argument to dimshuffle must be list or tuple of integers");
            goto CudaNdarray_dimshuffle_fail;
        }

        elem_value = PyInt_AsLong(elem);

        if (elem_value == -1 && PyErr_Occurred() )
        {
            Py_XDECREF(elem);
            PyErr_SetString(PyExc_ValueError, "Third argument to dimshuffle must be list or tuple of integers");
            goto CudaNdarray_dimshuffle_fail;
        }

        pattern[i] = elem_value;

        Py_XDECREF( elem );
        Py_XDECREF( idx );
    }

    //allocate rval
    rval =  (PyObject *) CudaNdarray_View((CudaNdarray *) self);

    if (rval == NULL)
    {
        //CudaNdarray_New should have set the exception string
        goto CudaNdarray_dimshuffle_fail;
    }


    //printf("pattern_dim: %d\n",pattern_dim);
    //printf("pattern: %d %d\n",pattern[0],pattern[1]);
    //dims = CudaNdarray_HOST_DIMS( (CudaNdarray *) self);
    //printf("dims before: %d %d\n",dims[0],dims[1]);

    success = CudaNdarray_dimshuffle((CudaNdarray *) rval, pattern_dim, pattern);

    if (success != 0)
    {
        //Exception string should already be set by CudaNdarray_dimshuffle
        goto CudaNdarray_dimshuffle_fail;
    }

    free(pattern);

    return rval;

    CudaNdarray_dimshuffle_fail:

    if (pattern != NULL)
        free(pattern);

    Py_XDECREF(rval);
    return NULL;
}


int
cnda_structure_size(int nd)
{
    // dim0, dim1, ...
    // str0, str1, ...
    // log2(dim0), log2(dim1), ...
    return nd + nd + nd;
}

const int *
CudaNdarray_HOST_DIMS(const CudaNdarray * self)
{
    return self->host_structure;
}

const int *
CudaNdarray_HOST_STRIDES(const CudaNdarray * self)
{
    return self->host_structure + self->nd;
}
const int *
CudaNdarray_HOST_LOG2DIMS(const CudaNdarray * self)
{
    return self->host_structure + 2*self->nd;
}

void
cnda_mark_dev_structure_dirty(CudaNdarray * self)
{
    self->dev_structure_fresh = 0;
}

int
CudaNdarray_EqualAndIgnore(CudaNdarray *cnda1, CudaNdarray *cnda2, int ignoreSync, int ignoreBase)
{
    int verbose = 1;

    if (!ignoreSync && cnda1->dev_structure_fresh != cnda2->dev_structure_fresh)
    {
        if(verbose) fprintf(stdout, "CUDANDARRAY_EQUAL FAILED : 1\n");
        return 0;
    }

    if (cnda1->nd != cnda2->nd)
    {
        if(verbose) fprintf(stdout, "CUDANDARRAY_EQUAL FAILED : 2\n");
        return 0;
    }

    for (int i=0; i < 2*cnda1->nd; i++)
    {
        if (cnda1->host_structure[i] != cnda2->host_structure[i])
        {
            if(verbose)
                fprintf(stdout, "CUDANDARRAY_EQUAL : host_structure : %d, %d, %d\n", i, cnda1->host_structure[i], cnda2->host_structure[i]);
            return 0;
        }
    }

    if (!ignoreBase && cnda1->base != cnda2->base)
    {
        if(verbose) fprintf(stdout, "CUDANDARRAY_EQUAL FAILED : 4");
        return 0;
    }
    else if (cnda1->data_allocated != cnda2->data_allocated)
    {
        if(verbose) fprintf(stdout, "CUDANDARRAY_EQUAL FAILED : 5");
        return 0;
    }
    else if (cnda1->data_allocated && cnda1->devdata != cnda2->devdata)
    {
        if(verbose) fprintf(stdout, "CUDANDARRAY_EQUAL FAILED : 6");
        // no need to check devdata if data is not allocated
        return 0;
    }

    return 1;
}


int
CudaNdarray_Equal(CudaNdarray *cnda1, CudaNdarray *cnda2)
{
    return CudaNdarray_EqualAndIgnore(cnda1, cnda2, 0, 0);
}

void
CudaNdarray_set_dim(CudaNdarray * self, int idx, int d)
{
    if ((idx >= self->nd) || (idx < 0) || (d < 0))
    {
        fprintf(stderr, "WARNING: probably bad CudaNdarray_set_dim arguments: %i %i\n", idx, d);
    }

    if (d != self->host_structure[idx])
    {
        self->host_structure[idx] = d;
        int log2d = (int)log2((double)d);
        self->host_structure[idx + 2*self->nd] = (d == (1 << log2d)) ? log2d : -1;
        cnda_mark_dev_structure_dirty(self);
    }
}

void
CudaNdarray_set_stride(CudaNdarray * self, int idx, int s)
{
    if ((idx >= self->nd) || (idx < 0))
    {
        fprintf(stderr, "WARNING: probably bad CudaNdarray_set_stride arguments: %i %i\n", idx, s);
    }

    if (s != CudaNdarray_HOST_STRIDES(self)[idx])
    {
        self->host_structure[idx+self->nd] = s;
        cnda_mark_dev_structure_dirty(self);
    }
}


int
cnda_copy_structure_to_device(const CudaNdarray * self)
{
    hipblasSetVector(cnda_structure_size(self->nd), sizeof(int), self->host_structure, 1, self->dev_structure, 1);
    CNDA_THREAD_SYNC;
    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "error copying structure to device memory");
        return -1;
    }
    self->dev_structure_fresh = 1;
    return 0;
}

const int *
CudaNdarray_DEV_DIMS(const CudaNdarray * self)
{
    if (!self->dev_structure_fresh)
    {
        if (cnda_copy_structure_to_device(self))
            return NULL;
    }
    return self->dev_structure;
}
const int *
CudaNdarray_DEV_STRIDES(const CudaNdarray * self)
{
    if (!self->dev_structure_fresh)
    {
        if (cnda_copy_structure_to_device(self))
            return NULL;
    }
    return self->dev_structure + self->nd;
}
const int *
CudaNdarray_DEV_LOG2DIMS(const CudaNdarray * self)
{
    if (!self->dev_structure_fresh)
    {
        if (cnda_copy_structure_to_device(self))
            return NULL;
    }
    return self->dev_structure + 2*self->nd;
}
float *
CudaNdarray_DEV_DATA(const CudaNdarray * self)
{
    return self->devdata;
}

/**
 * Return the number of elements in the ndarray (product of the dimensions)
 */
int
CudaNdarray_SIZE(const CudaNdarray *self)
{
    if (self->nd == -1) return 0;
    int size = 1;
    for (int i = 0; i < self->nd; ++i)
    {
        size *= CudaNdarray_HOST_DIMS(self)[i];
    }
    return size;
}

PyObject *
CudaNdarray_SIZE_Object(const CudaNdarray *self, void *closure)
{
    return PyInt_FromLong(CudaNdarray_SIZE(self));
}

int CudaNdarray_set_nd(CudaNdarray * self, const int nd)
{
    if (nd != self->nd)
    {
        if (self->dev_structure)
        {
            if (device_free(self->dev_structure))
            {
                return -1;
            }
            self->dev_structure = NULL;
        }
        if (self->host_structure)
        {
            free(self->host_structure);
            self->host_structure = NULL;
            self->nd = -1;
        }
        if (nd == -1) return 0;

        self->host_structure = (int*)malloc(cnda_structure_size(nd)*sizeof(int));
        if (NULL == self->host_structure)
        {
            PyErr_SetString(PyExc_MemoryError, "Failed to allocate dim or str");
            return -1;
        }
        //initialize all dimensions and strides to 0
        for (int i = 0; i < cnda_structure_size(nd); ++i)
        {
            self->host_structure[i] = 0;
        }

        int struct_size = cnda_structure_size(nd);
        if (struct_size)
        {
            self->dev_structure = (int*)device_malloc(struct_size* sizeof(int));
            if (NULL == self->dev_structure)
            {
                free(self->host_structure);
                self->host_structure = NULL;
                self->dev_structure = NULL;
                return -1;
            }
        }
        self->nd = nd;
        self->dev_structure_fresh = 0;
    }
    return 0;
}

int CudaNdarray_set_device_data(CudaNdarray * self, float * data, CudaNdarray * base)
{
    return CudaNdarray_set_device_data(self, data, (PyObject *) base);
}

PyObject * CudaNdarray_IS_C_Contiguous(CudaNdarray * self)
{
    return PyBool_FromLong(CudaNdarray_is_c_contiguous(self));
}

void fprint_CudaNdarray(FILE * fd, const CudaNdarray *self)
{
    fprintf(fd, "CudaNdarray <%p, %p> nd=%i dev_structure_fresh=%d data_allocated=%d\n",
            self, self->devdata, self->nd, self->dev_structure_fresh, self->data_allocated);
    fprintf(fd, "\tHOST_DIMS:      ");
    for (int i = 0; i < self->nd; ++i)
    {
        fprintf(fd, "%i\t", CudaNdarray_HOST_DIMS(self)[i]);
    }
    fprintf(fd, "\n\tHOST_STRIDES: ");
    for (int i = 0; i < self->nd; ++i)
    {
        fprintf(fd, "%i\t", CudaNdarray_HOST_STRIDES(self)[i]);
    }

    int data=0;
    fprintf(fd, "\n\tDEV_DIMS:      ");
    for (int i = 0; i < self->nd; ++i)
    {
        hipblasGetVector(1, sizeof(int),
                        self->dev_structure+i, 1,
                        &data, 1);
        fprintf(fd, "%i\t", data);
    }
    fprintf(fd, "\n\tDEV_STRIDES: ");
    for (int i = 0; i < self->nd; ++i)
    {
        hipblasGetVector(1, sizeof(int),
                        self->dev_structure + self->nd+i, 1,
                        &data, 1);
        fprintf(fd, "%i \t", data);
    }
    fprintf(fd, "\n");
}

/*
  Local Variables:
  mode:c++
  c-basic-offset:4
  c-file-style:"stroustrup"
  c-file-offsets:((innamespace . 0)(inline-open . 0))
  indent-tabs-mode:nil
  fill-column:79
  End:
*/
// vim: filetype=cpp:expandtab:shiftwidth=4:tabstop=8:softtabstop=4:textwidth=79 :
