#include "hip/hip_runtime.h"
// This uses a lot of code from Caffe (http://caffe.berkeleyvision.org/);
// sources are clearly marked. Below we reproduce the original license of
// the Caffe software.
/*
Copyright (c) 2014, The Regents of the University of California (Regents)
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#undef _GLIBCXX_ATOMIC_BUILTINS


// (borrowed from Caffe: https://github.com/BVLC/caffe/blob/master/src/caffe/caffe_common.hpp)
// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n)                        \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n);                                       \
       i += blockDim.x * gridDim.x)

// CUDA: thread number configuration.
// Use 1024 threads per block, which requires cuda sm_2x or above,
// or fall back to attempt compatibility (best of luck to you).
#if __CUDA_ARCH__ >= 200
    const int CUDA_NUM_THREADS = 1024;
#else
    const int CUDA_NUM_THREADS = 512;
#endif

// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}


// (Adapted from Caffe: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu)
// Kernels for fast unfold + copy
// CUDA kernel for the case of dilation
__global__ void dilated_im3d2col_kernel(const int n, const float* data_im,
    const int height, const int width, const int depth,
    const int kernel_h, const int kernel_w, const int kernel_d,
    const int dilation_h, const int dilation_w, const int dilation_d,
    const int pad_h, const int pad_w, const int pad_d,
    const int stride_h, const int stride_w, const int stride_d,
    const int height_col, const int width_col, const int depth_col,
    float* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    const int w_index = index / depth_col;
    const int h_index = w_index / width_col;
    const int d_col = index % depth_col;
    const int h_col = h_index % height_col;
    const int w_col = w_index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w * kernel_d;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    const int d_offset = d_col * stride_d - pad_d;
    float* data_col_ptr = data_col;
    data_col_ptr += c_col * (height_col * width_col * depth_col) +
      h_col * (width_col * depth_col) + w_col * depth_col + d_col;
    const float* data_im_ptr = data_im;
    data_im_ptr += c_im * (height * width * depth) +
      h_offset * (width * depth) + w_offset * depth + d_offset;

    for (int i = 0; i < kernel_h; ++i)
    {
      int h_im = h_offset + i * dilation_h;
      for (int j = 0; j < kernel_w; ++j)
      {
        int w_im = w_offset + j * dilation_w;
        for (int k = 0; k < kernel_d; ++k)
        {
          int d_im = d_offset + k * dilation_d;
          *data_col_ptr = (h_im >= 0 && w_im >= 0 && d_im >= 0 &&
                           h_im < height && w_im < width && d_im < depth) ?
                           data_im_ptr[i * dilation_h * (width * depth) +
                                       j * dilation_w * depth +
                                       k * dilation_d] : 0;
          data_col_ptr += height_col * width_col * depth_col;
        }
      }
    }
  }
}

__global__ void im3d2col_kernel(const int n, const float* data_im,
                                const int height, const int width, const int depth,
                                const int kernel_h, const int kernel_w, const int kernel_d,
                                const int pad_h, const int pad_w, const int pad_d,
                                const int stride_h, const int stride_w, const int stride_d,
                                const int height_col, const int width_col, const int depth_col,
                                float* data_col)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    const int w_index = index / depth_col;
    const int h_index = w_index / width_col;
    const int d_col = index % depth_col;
    const int h_col = h_index % height_col;
    const int w_col = w_index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w * kernel_d;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    const int d_offset = d_col * stride_d - pad_d;
    float* data_col_ptr = data_col;
    data_col_ptr += c_col * (height_col * width_col * depth_col) +
      h_col * (width_col * depth_col) + w_col * depth_col + d_col;
    const float* data_im_ptr = data_im;
    data_im_ptr += c_im * (height * width * depth) +
      h_offset * (width * depth) + w_offset * depth + d_offset;

    for (int i = 0; i < kernel_h; ++i)
    {
      int h_im = h_offset + i;
      for (int j = 0; j < kernel_w; ++j)
      {
        int w_im = w_offset + j;
        for (int k = 0; k < kernel_d; ++k)
        {
          int d_im = d_offset + k;
          *data_col_ptr = (h_im >= 0 && w_im >= 0 && d_im >= 0 &&
                           h_im < height && w_im < width && d_im < depth) ?
                           data_im_ptr[i * (width * depth) + j * depth + k] : 0;
          data_col_ptr += height_col * width_col * depth_col;
        }
      }
    }
  }
}

void im3d2col(const float* data_im, const int channels,
              const int height, const int width, const int depth,
              const int kernel_h, const int kernel_w, const int kernel_d,
              const int dilation_h, const int dilation_w, const int dilation_d,
              const int pad_h, const int pad_w, const int pad_d,
              const int stride_h, const int stride_w, const int stride_d,
              float* data_col)
{
  // We are going to launch channels * height_col * width_col * depth_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int dil_kernel_h = (kernel_h - 1) * dilation_h + 1;
  int dil_kernel_w = (kernel_w - 1) * dilation_w + 1;
  int dil_kernel_d = (kernel_d - 1) * dilation_d + 1;
  int height_col = (height + 2 * pad_h - dil_kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - dil_kernel_w) / stride_w + 1;
  int depth_col = (depth + 2 * pad_d - dil_kernel_d) / stride_d + 1;
  int num_kernels = channels * height_col * width_col * depth_col;
  if(dilation_h != 1 || dilation_w != 1 || dilation_d != 1){
    dilated_im3d2col_kernel<<<GET_BLOCKS(num_kernels),
                      CUDA_NUM_THREADS>>>(num_kernels, data_im,
                                          height, width, depth,
                                          kernel_h, kernel_w, kernel_d,
                                          dilation_h, dilation_w, dilation_d,
                                          pad_h, pad_w, pad_d,
                                          stride_h, stride_w, stride_d,
                                          height_col, width_col, depth_col,
                                          data_col);
  }
  else{
    im3d2col_kernel<<<GET_BLOCKS(num_kernels),
                      CUDA_NUM_THREADS>>>(num_kernels, data_im,
                                          height, width, depth,
                                          kernel_h, kernel_w, kernel_d,
                                          pad_h, pad_w, pad_d,
                                          stride_h, stride_w, stride_d,
                                          height_col, width_col, depth_col,
                                          data_col);
  }
}

// CUDA kernel for the case of dilation
__global__ void dilated_col2im3d_kernel(
    const int n, const float* data_col,
    const int height, const int width, const int depth,
    const int channels,
    const int kernel_h, const int kernel_w, const int kernel_d,
    const int dilation_h, const int dilation_w, const int dilation_d,
    const int pad_h, const int pad_w, const int pad_d,
    const int stride_h, const int stride_w, const int stride_d,
    const int height_col, const int width_col, const int depth_col,
    float* data_im)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    float val = 0;
    const int d_im = index % depth + pad_d;
    const int w_index = index / depth;
    const int w_im = w_index % width + pad_w;
    const int h_index = w_index / width;
    const int h_im = h_index % height + pad_h;
    const int c_im = h_index / height;
    int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
    int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
    int kernel_extent_d = (kernel_d - 1) * dilation_d + 1;
    // compute the start and end of the output
    const int d_col_start = (d_im < kernel_extent_d) ? 0 : (d_im - kernel_extent_d) / stride_d + 1;
    const int d_col_end = min(d_im / stride_d + 1, depth_col);
    const int w_col_start = (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start = (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);
    // TODO: use LCM of stride and dilation to avoid unnecessary loops
    for (int d_col = d_col_start; d_col < d_col_end; ++d_col) {
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          int h_k = (h_im - h_col * stride_h);
          int w_k = (w_im - w_col * stride_w);
          int d_k = (d_im - d_col * stride_d);
          if (h_k % dilation_h == 0 && w_k % dilation_w == 0 && d_k % dilation_d == 0) {
            h_k /= dilation_h;
            w_k /= dilation_w;
            d_k /= dilation_d;
            int data_col_index = c_im * kernel_h * kernel_w * kernel_d * height_col * width_col * depth_col +
                                 h_k             * kernel_w * kernel_d * height_col * width_col * depth_col +
                                 w_k                        * kernel_d * height_col * width_col * depth_col +
                                 d_k                                   * height_col * width_col * depth_col +
                                 h_col                                              * width_col * depth_col +
                                 w_col                                                          * depth_col +
                                 d_col;
            val += data_col[data_col_index];
          }
        }
      }
   }
   data_im[index] = val;
  }
}

__global__ void col2im3d_kernel(const int n, const float* data_col,
                                const int height, const int width, const int depth,
                                const int channels,
                                const int kernel_h, const int kernel_w, const int kernel_d,
                                const int pad_h, const int pad_w, const int pad_d,
                                const int stride_h, const int stride_w, const int stride_d,
                                const int height_col, const int width_col, const int depth_col,
                                float* data_im)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    float val = 0;
    const int d_im = index % depth + pad_d;
    const int w_index = index / depth;
    const int w_im = w_index % width + pad_w;
    const int h_index = w_index / width;
    const int h_im = h_index % height + pad_h;
    const int c_im = h_index / height;

    // compute the start and end of the output
    const int d_col_start = (d_im < kernel_d) ? 0 : (d_im - kernel_d) / stride_d + 1;
    const int d_col_end = min(d_im / stride_d + 1, depth_col);
    const int w_col_start = (w_im < kernel_w) ? 0 : (w_im - kernel_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start = (h_im < kernel_h) ? 0 : (h_im - kernel_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);

    int offset =
      (c_im * kernel_h * kernel_w * kernel_d + h_im * kernel_w * kernel_d +
       w_im * kernel_d + d_im) * height_col * width_col * depth_col;

    int coeff_h_col = (1 - stride_h * kernel_w * kernel_d * height_col) * width_col * depth_col;
    int coeff_w_col = (1 - stride_w * kernel_d * height_col * width_col) * depth_col;
    int coeff_d_col = (1 - stride_d * height_col * width_col * depth_col);
    for (int d_col = d_col_start; d_col < d_col_end; ++d_col) {
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col + d_col * coeff_d_col];
        }
      }
   }
   data_im[index] = val;
  }
}

void col2im3d(const float* data_col, const int channels,
              const int height, const int width, const int depth,
              const int patch_h, const int patch_w, const int patch_d,
              const int dilation_h, const int dilation_w, const int dilation_d,
              const int pad_h, const int pad_w, const int pad_d,
              const int stride_h, const int stride_w, const int stride_d,
              float* data_im)
{
  int dil_patch_h = (patch_h - 1) * dilation_h + 1;
  int dil_patch_w = (patch_w - 1) * dilation_w + 1;
  int dil_patch_d = (patch_d - 1) * dilation_d + 1;
  int height_col = (height + 2 * pad_h - dil_patch_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - dil_patch_w) / stride_w + 1;
  int depth_col = (depth + 2 * pad_d - dil_patch_d) / stride_d + 1;
  int num_kernels = channels * height * width * depth;

  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  if(dilation_h != 1 || dilation_w != 1 || dilation_d != 1){
    dilated_col2im3d_kernel<<<GET_BLOCKS(num_kernels),
                              CUDA_NUM_THREADS>>>(num_kernels, data_col,
                                                  height, width, depth, channels,
                                                  patch_h, patch_w, patch_d,
                                                  dilation_h, dilation_w, dilation_d,
                                                  pad_h, pad_w, pad_d,
                                                  stride_h, stride_w, stride_d,
                                                  height_col, width_col, depth_col,
                                                  data_im);
  }
  else{
    col2im3d_kernel<<<GET_BLOCKS(num_kernels),
                      CUDA_NUM_THREADS>>>(num_kernels, data_col,
                                          height, width, depth, channels,
                                          patch_h, patch_w, patch_d,
                                          pad_h, pad_w, pad_d,
                                          stride_h, stride_w, stride_d,
                                          height_col, width_col, depth_col,
                                          data_im);
  }
}




// Theano op code
// Authors: Arjun Jain, Frederic Bastien, Jan Schluter, Nicolas Ballas
// Reference code: https://github.com/BVLC/caffe/blob/master/src/caffe/layers/conv_layer.cu
//   and https://github.com/torch/cunn/blob/master/SpatialConvolutionMM.cu
// Adaptation for 3d
CudaNdarray* corr3dMM(CudaNdarray *const bottom,
                      CudaNdarray *const weight,
                      CudaNdarray *const top,
                      const int direction,
                      const int dH = 1,
                      const int dW = 1,
                      const int dD = 1,
                      const int dilH = 1,
                      const int dilW = 1,
                      const int dilD = 1,
                      const int padH = 0,
                      const int padW = 0,
                      const int padD = 0)
{
    if (bottom->nd != 5)
    {
      PyErr_SetString(PyExc_ValueError, "GpuCorr3dMM requires bottom of 5D");
      return NULL;
    }
    if (!CudaNdarray_is_c_contiguous(bottom))
    {
      PyErr_Format(PyExc_ValueError,
                   "GpuCorr3dMM requires bottom to be C-contiguous, "
                   "but strides are: %d %d %d %d %d\n",
                   CudaNdarray_HOST_STRIDES(bottom)[0],
                   CudaNdarray_HOST_STRIDES(bottom)[1],
                   CudaNdarray_HOST_STRIDES(bottom)[2],
                   CudaNdarray_HOST_STRIDES(bottom)[3],
                   CudaNdarray_HOST_STRIDES(bottom)[4]);
      return 0;
    }
    if (weight->nd != 5)
    {
      PyErr_SetString(PyExc_ValueError, "GpuCorr3dMM requires weight of 5D");
      return 0;
    }
    if (!CudaNdarray_is_c_contiguous(weight))
    {
      PyErr_Format(PyExc_ValueError,
                   "GpuCorr3dMM requires weight to be C-contiguous, "
                   "but strides are: %d %d %d %d %d\n",
                   CudaNdarray_HOST_STRIDES(weight)[0],
                   CudaNdarray_HOST_STRIDES(weight)[1],
                   CudaNdarray_HOST_STRIDES(weight)[2],
                   CudaNdarray_HOST_STRIDES(weight)[3],
                   CudaNdarray_HOST_STRIDES(weight)[4]);
      return 0;
    }

    if (top->nd != 5)
    {
      PyErr_SetString(PyExc_ValueError, "GpuCorr3dMM requires top of 5D");
      return 0;
    }
    if (!CudaNdarray_is_c_contiguous(top))
    {
      PyErr_Format(PyExc_ValueError,
                   "GpuCorr3dMM requires top to be C-contiguous, "
                   "but strides are: %d %d %d %d %d\n",
                   CudaNdarray_HOST_STRIDES(top)[0],
                   CudaNdarray_HOST_STRIDES(top)[1],
                   CudaNdarray_HOST_STRIDES(top)[2],
                   CudaNdarray_HOST_STRIDES(top)[3],
                   CudaNdarray_HOST_STRIDES(top)[4]);
      return 0;
    }


    // Extract some shape information for later and check shape consistency
    // bottom: (batchSize, nChannels, bottomHeight, bottomWidth, bottomDepth)
    const int batchSize = CudaNdarray_HOST_DIMS(bottom)[0];
    const int nChannels = CudaNdarray_HOST_DIMS(bottom)[1];
    const int bottomHeight = CudaNdarray_HOST_DIMS(bottom)[2];
    const int bottomWidth = CudaNdarray_HOST_DIMS(bottom)[3];
    const int bottomDepth = CudaNdarray_HOST_DIMS(bottom)[4];
    // weights: (nFilters, nChannels, rows, columns, depth)
    const int nFilters = CudaNdarray_HOST_DIMS(weight)[0];
    const int kH = CudaNdarray_HOST_DIMS(weight)[2];
    const int kW = CudaNdarray_HOST_DIMS(weight)[3];
    const int kD = CudaNdarray_HOST_DIMS(weight)[4];
    if (nChannels != CudaNdarray_HOST_DIMS(weight)[1])
    {
      PyErr_SetString(PyExc_ValueError,
                      "GpuCorr3dMM images and kernel must have the same stack size\n");
      return 0;
    }
    // implicit dilated filter
    const int dil_kH = (kH - 1) * dilH + 1;
    const int dil_kW = (kW - 1) * dilW + 1;
    const int dil_kD = (kD - 1) * dilD + 1;
    // top: (batchSize, nFilters, topHeight, topWidth, topDepth)
    const int topHeightNoDH = (bottomHeight + 2*padH - dil_kH);
    const int topWidthNoDW  = (bottomWidth + 2*padW - dil_kW);
    const int topDepthNoDD  = (bottomDepth + 2*padD - dil_kD);
    // the above values might be negative so we need to use Python-like
    // flooring integer division to be compatible with get_conv_output.
    // note: this macro implements Python's // for negative x only
#define _CONV_FLOORDIV_X(x,y) ((x < 0) ? (- ((-x) / y) - (((-x) % y) == 0 ? 0 : 1)) : (x / y))
    const int topHeight = _CONV_FLOORDIV_X(topHeightNoDH, dH) + 1;
    const int topWidth  = _CONV_FLOORDIV_X(topWidthNoDW, dW) + 1;
    const int topDepth  = _CONV_FLOORDIV_X(topDepthNoDD, dD) + 1;
#undef _CONV_FLOORDIV
    if (batchSize != CudaNdarray_HOST_DIMS(top)[0] ||
        nFilters != CudaNdarray_HOST_DIMS(top)[1] ||
        topHeight != CudaNdarray_HOST_DIMS(top)[2] ||
        topWidth != CudaNdarray_HOST_DIMS(top)[3] ||
        topDepth != CudaNdarray_HOST_DIMS(top)[4])
   {
     PyErr_Format(PyExc_ValueError,
                  "GpuCorr3dMM shape inconsistency:\n"
                  "  bottom shape: %d %d %d %d %d\n"
                  "  weight shape: %d %d %d %d %d\n"
                  "  top shape: %d %d %d %d %d (expected %d %d %d %d %d)\n",
                  batchSize, nChannels, bottomHeight, bottomWidth, bottomDepth,
                  nFilters, nChannels, kH, kW, kD,
                  CudaNdarray_HOST_DIMS(top)[0], CudaNdarray_HOST_DIMS(top)[1],
                  CudaNdarray_HOST_DIMS(top)[2], CudaNdarray_HOST_DIMS(top)[3],
                  CudaNdarray_HOST_DIMS(top)[4],
                  batchSize, nFilters, topHeight, topWidth, topDepth);
        return 0;
    }

    // Create temporary columns
    int col_dim[2];
    col_dim[0] = nChannels * kW * kH * kD;
    col_dim[1] = topHeight * topWidth * topDepth;
    CudaNdarray* col = (CudaNdarray*) CudaNdarray_NewDims(2, col_dim);
    if (0 == col)
    {
      PyErr_Format(PyExc_RuntimeError,
                   "GpuCorr3dMM failed to allocate working memory of %d x %d\n",
                   col_dim[0], col_dim[1]);
        return 0;
    }

    // Define some useful variables
    const int bottom_stride = CudaNdarray_HOST_STRIDES(bottom)[0];
    const int top_stride = CudaNdarray_HOST_STRIDES(top)[0];
    const int K_ = col_dim[0];
    const int N_ = col_dim[1];
    const int M_ = nFilters;
    const float one = 1.0f;
    const float zero = 0.0f;

    CudaNdarray *output;
    if (direction == 0)
    { // forward pass
      output = top;
      if (batchSize == 0 || nChannels == 0 || nFilters == 0) {
          hipError_t err = hipMemset(output->devdata, 0,
                                       CudaNdarray_SIZE(output) * sizeof(real));
          if (err != hipSuccess) {
            PyErr_Format(PyExc_RuntimeError,
                         "GpuCorr3dMM could not fill the output with zeros: %s",
                         hipGetErrorString(err));
            Py_DECREF(col);
            return NULL;
          }
          Py_DECREF(col);
          return output;
      }
      // valid correlation: im2col, then gemm
      // Iterate over batch
      for (int n = 0; n < batchSize; n++)
      {
        // First, im3d2col
        im3d2col(bottom->devdata + n * bottom_stride,
                 nChannels,
                 bottomHeight, bottomWidth, bottomDepth,
                 kH, kW, kD,
                 dilH, dilW, dilD,
                 padH, padW, padD,
                 dH, dW, dD,
                 col->devdata);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
          PyErr_Format(PyExc_RuntimeError,
                       "GpuCorr3dMM encountered a CUDA error in im2col: %s\n"
                       "This could be a known bug in CUDA, please see the "
                       "GpuCorr3dMM() documentation.\n",
                       hipGetErrorString(err));
          Py_DECREF(col);
          return 0;
        }
        // Second, gemm
        hipblasStatus_t status = hipblasSgemm(handle,
                                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                                            N_, M_, K_,
                                            &one,
                                            col->devdata, N_,
                                            weight->devdata, K_,
                                            &zero,
                                            top->devdata + n * top_stride, N_);
        if (status != HIPBLAS_STATUS_SUCCESS)
        {
          PyErr_Format(PyExc_RuntimeError,
                       "GpuCorr3dMM encountered a CUBLAS error: %s\n"
                       "This could be a known bug in CUDA, please see the "
                       "GpuCorr3dMM() documentation.\n",
                       cublasGetErrorString(status));
          Py_DECREF(col);
          return 0;
        }
      }
    }
    else if (direction == 1)
    {
      // backprop wrt. weights
      output = weight;
      if (batchSize == 0 || nChannels == 0 || nFilters == 0) {
          hipError_t err = hipMemset(output->devdata, 0,
                                       CudaNdarray_SIZE(output) * sizeof(real));
          if (err != hipSuccess) {
            PyErr_Format(PyExc_RuntimeError,
                         "GpuCorr3dMM grad wrt. weights could not fill the output with zeros: %s",
                         hipGetErrorString(err));
            Py_DECREF(col);
            return NULL;
          }
          Py_DECREF(col);
          return output;
      }
      // valid convolution: im2col, then gemm
      // Iterate over batch
      for (int n = 0; n < batchSize; n++)
      {
        // First, im2col
        im3d2col(bottom->devdata + n * bottom_stride, nChannels,
                 bottomHeight, bottomWidth, bottomDepth,
                 kH, kW, kD,
                 dilH, dilW, dilD,
                 padH, padW, padD,
                 dH, dW, dD,
                 col->devdata);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
       {
         PyErr_Format(PyExc_RuntimeError,
                      "GpuCorr3dMM encountered a CUDA error in im2col: %s\n"
                      "This could be a known bug in CUDA, please see the "
                      "GpuCorr3dMM() documentation.\n",
                      hipGetErrorString(err));
         Py_DECREF(col);
         return 0;
       }
        // Second, gemm
        // Note that we accumulate into weight. We do so by setting beta = 0
        // for the first iteration and beta = 1 for subsequent ones. (This
        // is faster than setting weight to all zeros before the loop.)
        hipblasStatus_t status = hipblasSgemm(handle,
                                            HIPBLAS_OP_T, HIPBLAS_OP_N,
                                            K_, M_, N_,
                                            &one,
                                            col->devdata, N_,
                                            top->devdata + n * top_stride, N_,
                                            (n == 0) ? &zero : &one,
                                            weight->devdata, K_);
        if (status != HIPBLAS_STATUS_SUCCESS)
        {
          PyErr_Format(PyExc_RuntimeError,
                       "GpuCorr3dMM encountered a CUBLAS error: %s\n"
                       "This could be a known bug in CUDA, please see the "
                       "GpuCorr3dMM() documentation.\n",
                       cublasGetErrorString(status));
          Py_DECREF(col);
          return 0;
        }
      }
    }
    else if (direction == 2)
    {
      // backprop wrt. inputs
      output = bottom;
      if (batchSize == 0 || nChannels == 0 || nFilters == 0) {
          hipError_t err = hipMemset(output->devdata, 0,
                                       CudaNdarray_SIZE(output) * sizeof(real));
          if (err != hipSuccess) {
            PyErr_Format(PyExc_RuntimeError,
                         "GpuCorr3dMM grad wrt. inputs could not fill the output with zeros: %s",
                         hipGetErrorString(err));
            Py_DECREF(col);
            return NULL;
          }
          Py_DECREF(col);
          return output;
      }
      // full convolution: gemm, then col2im3d
      // Iterate over batch
      for (int n = 0; n < batchSize; n++)
      {
        // gemm into columns
        hipblasStatus_t status = hipblasSgemm(handle,
                                            HIPBLAS_OP_N, HIPBLAS_OP_T,
                                            N_, K_, M_,
                                            &one,
                                            top->devdata + n * top_stride, N_,
                                            weight->devdata, K_,
                                            &zero,
                                            col->devdata, N_);
        if (status != HIPBLAS_STATUS_SUCCESS)
        {
          PyErr_Format(PyExc_RuntimeError,
                       "GpuCorr3dMM encountered a CUBLAS error: %s\n"
                       "This could be a known bug in CUDA, please see the "
                       "GpuCorr3dMM() documentation.\n",
                       cublasGetErrorString(status));
          Py_DECREF(col);
          return 0;
        }
        // col2im3d back to the data
        col2im3d(col->devdata, nChannels,
                 bottomHeight, bottomWidth, bottomDepth,
                 kH, kW, kD,
                 dilH, dilW, dilD,
                 padH, padW, padD,
                 dH, dW, dD, bottom->devdata + n * bottom_stride);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
          PyErr_Format(PyExc_RuntimeError,
                       "GpuCorr3dMM encountered a CUDA error in col2im: %s\n"
                       "This could be a known bug in CUDA, please see the "
                       "GpuCorr3dMM() documentation.\n",
                       hipGetErrorString(err));
          Py_DECREF(col);
          return 0;
        }
      }
    }
    // Free temporary columns
    Py_DECREF(col);

    // Note that we don't change the refcount of the output matrix here. Output
    // allocation and refcounting is done in BaseGpuCorr3dMM.c_code_helper();
    // in here output is just aliased to one of bottom, weights, or top.
    return output;
}
